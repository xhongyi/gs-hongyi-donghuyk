#include "hip/hip_runtime.h"
/*
 * edit_distance.cpp
 *
 *  Created on: Oct 29, 2011
 *      Author: mac
 */

/*
 *
 */
#define		_MAIN_PATH_		0
#define		_DELETE_PATH_	1
#define		_INSERT_PATH_	2
#define		_UN_FILLED_		1000

#include "edit_distance.h"
#include <algorithm>
#include <cstdlib>
//#include <cstring>
#include <iostream>
#include <assert.h>
/*
 *          reference string
 * 		  a p p l i c a t i o n
 *      0 1 2
 * t  a 1 0 1 2
 * e  p 2 1 0 1 2    <----delete_path
 * s  p   2 1 0 1 2
 * t  p     2 1 1 2 3
 *    l       2 1 2 3
 * s  o     ^   2 2 3 <------main_path
 * t  c     |     3 2
 * i  a     |         2
 * n  t     |           2
 * g  i     |             2
 *    o     |               2
 *	        |
 *	    insert_path
 *
 *
 *          reference string
 *        a p p l i c a t i o n
 *      0 1 2 3
 * t  a 1 0 1 2 3
 * e  p 2 1 0 1 2 3  <----bondary_path
 * s  p 3 2 1 0 1 2 3
 * t  p   3 2 1 1 2 3 4
 *    l     3 2 1 2 3 4 5
 * s  o     ^ 3 2 2 3 4 5 <----delete_path
 * t  c     |   3 3 2 3 4
 * i  a     |       3 2 3 4
 * n  i     |         3 3 3 4
 * g  o     |             4 3 4
 *    n     |             ^ 4 3 <------main_path
 *	        |             |
 *	   bondary_path  insert_path
 *      |
 *      | | | 3 3 3 3 4 5
 *      | | 2 2 2 2 3 4 5
 *      | 1 1 1 1 2 3 4 4 4 4 4
 *      0 0 0 0 1 2 3 3 3 3 3 3
 *      1 1 1 1 1 2 2 2 3 4 4 |
 *      2 2 2 2 2 3 3 3   | | |
 *      3 3 3 3 3       | | | |
 *
 */
__device__ void initializePath(ED_path* path, int main_lane, int max_indel_num);

__device__ ED_result editDistanceCalFWD(char* test_read, char* ref_read,
		int key_num, ED_path* path, int main_lane, int max_indel_num,
		int max_diff_num);

__device__ ED_result editDistanceCalBWD(char* test_read, char* ref_read,
		int key_num, ED_path* path, int main_lane, int max_indel_num,
		int max_diff_num);

__device__ void initializeFWDFront(int key_num, ED_path* path, int main_lane,
		int max_indel_num);

__device__ void initializeBWDFront(int key_num, ED_path* path, int main_lane,
		int max_indel_num);

// initializePath only fills the path elements now.
__device__ void initializePath(ED_path* path, int main_lane, int max_indel_num) {
	for (int i = 0; i < max_indel_num * 2 + 3; i++) {
		for (int j = 0; j <= READ_LENGTH; j++) {
			path[i].path_cost[j] = _UN_FILLED_;
		}
	}
}

__device__ void initializeFWDFront(int key_num, ED_path* path, int main_lane,
		int max_indel_num) {
	for (int i = 0; i < max_indel_num * 2 + 3; i++)
		//For the insertion lanes, the front point is shift right.
		path[i].front_idx = (i < main_lane) ? key_num * KEY_LENGTH + main_lane
				- i : key_num * KEY_LENGTH;
}

__device__ void initializeBWDFront(int key_num, ED_path* path, int main_lane,
		int max_indel_num) {
	for (int i = 0; i < max_indel_num * 2 + 3; i++)
		//For the insertion lanes, the front point is shift right.
		path[i].front_idx = (i > main_lane) ? key_num * KEY_LENGTH + main_lane
				- i : key_num * KEY_LENGTH;
}

__device__ ED_result editDistanceCal(char* test_read, char* ref_read,
		int key_num, ED_path* path, int main_lane, int max_indel_num,
		int max_diff_num) {

	DEBUG_PRINT2("Inside ED test 1  threadId: %i\n", threadIdx.x);

	//Initialize path
	ED_result result;
	ED_result FWD_result, BWD_result;
	initializePath(path, main_lane, max_indel_num);
	/*----------------------------------------------------------------------
	for (int i = 0; i < max_indel_num * 2 + 3; i++) {
		for (int j = 0; j <= READ_LENGTH; j++) {
			path[i].path_cost[j] = _UN_FILLED_;
		}
	}
	----------------------------------------------------------------------*/
	DEBUG_PRINT2("FWDeditDistanceCal started  threadId: %i\n", threadIdx.x); //Idealy, We should first see all threads printing a since they are at the same warp
	FWD_result = editDistanceCalFWD(test_read, ref_read, key_num, path,
			main_lane, max_indel_num, max_diff_num);
/*
	for (int i = 0; i < 12; i++) {
		//printf("Now path i: %i\n", i);
		for (int j = 0; j <= 108; j++) {
			//printf("path[%i].path_cost[%i]: %i\n", i, j, path[i].path_cost[j]);
		}
	}
*/
	//__syncthreads(); //Force all threads synchronize.

	DEBUG_PRINT2("BWDeditDistanceCal started  threadId: %i\n", threadIdx.x); //We should see all threads print b. However, it is not doing that.
	//DEBUG_PRINT2("***---Befor BWD key_num : %i\n", key_num);
	BWD_result = editDistanceCalBWD(test_read, ref_read, key_num, path,
			main_lane, max_indel_num, max_diff_num);
	DEBUG_PRINT2("2 Directions have finished  threadId: %i\n", threadIdx.x);

	if (BWD_result.correct)
		//printf("BWD is correct!!!  threadId: %i\n", threadIdx.x);

	result.diff_num = FWD_result.diff_num + BWD_result.diff_num;

	//This is the result index used to probe the result error queue.
	int error_idx = 0;

	if (FWD_result.correct && BWD_result.correct && result.diff_num
			<= max_diff_num) {

		result.correct = true;

		for (int i = 0; i < BWD_result.diff_num; i++) {
			result.error[error_idx] = BWD_result.error[i];
			error_idx++;
		}

		for (int i = FWD_result.diff_num - 1; i >= 0; i--) {
			result.error[error_idx] = FWD_result.error[i];
			error_idx++;
		}

	} else
		result.correct = false;

	return result;
}

/*
 * Find the min edit distance givin the location.
 */
__device__ unsigned char minDistance(unsigned char const** path, int row, int column) {
	unsigned char result;
	if (path[row - 1][column] < path[row - 1][column + 1])
		result = path[row - 1][column];
	else
		result = path[row - 1][column + 1];
	
	if (path[row][column - 1] < result)
		result = path[row][column - 1];
	
	return result;
}

/*
 * Going Forward, starting from the key_num * KEY_LENGTH position and going forward until end.
 */
__device__ void fillPath(char const* test_read, char const* ref_read,
		int key_num, unsigned char** path, ED_constant const* cons) {

	//i for rows and j for columns


	//Fill up the first default row (just fill the columns)
	//For the default columns we fill 0
	for (int j = 1; j <= cons->max_indel_num; j++) {
		path[0][j] = 0;
	}
	//Starting from max_indel_num we fill numbers.
	for (int j = cons->max_indel_num + 1; j <= 2 * cons->max_indel_num + 1; j++) {
		path[0][j] = j - cons->max_indel_num - 1;
	}


	//Fill up the beginning rows
	for (int i = 1; i <= 0 + cons->max_indel_num; i++) {
		//fill 0 for imaginary columns
		for (j = 1; j <= error_num + 1 - i; j++) {
			unsigned char distance = minDistance(Path, i, j);
			path[i][j] = distance + 1;
		}

		//compare and fill the real rows
		for (j = error_num; j <= 2 * cons->max_indel_num + 1; j++) {
			unsigned char distance = minDistance(path, i, j);
			if (test_read[i - 1] == ref_read[j + i - cons->max_indel_num - 1])
				path[i][j] = distance;
			else
				path[i][j] = distance + 1;
		}
	}

	//Normal fill
	for (i = 0 + cons->max_indel_num + 1; i <= READ_LENGTH - cons->max_indel_num) {
		for (j = 1; j <= 2 * cons->max_indel_num + 1; j++) {
			unsigned char distance = minDistance(path, i, j);
			if (test_read[i - 1] == ref_read[j + i - cons->max_indel_num - 2])
				path[i][j] = distance;
			else
				path[i][j] = distance + 1;
		}
	}

	//End process. Cleaning up process
	for (i = READ_LENGTH - cons->max_indel_num + 1; i <= READ_LENGTH; i++) {
		for (j = 1; j <= READ_LENGTH + 2 + cons->max_indel_num - i; j++) {
			unsigned char distance = minDistance(path, i, j);
			if (test_read[i - 1] == ref_read[j + i - cons->max_indel_num - 2])
				path[i][j] = distance;
			else
				path[i][j] = distance + 1;
		}

		for (j = READ_LENGTH + 2 + cons->max_indel_num - i + 1;
				j <= 2 * cons->max_indel_num + 1; j++) {
			unsigned char distance = minDistance(path, i, j);
			path[i][j] = distance + 1;
		}
	}
	return;
}

//Filling backward.

__device__ void fillPath(char const* test_read, char const* ref_read,
		int key_num, unsigned char** path, ED_constant const* cons) {

	int i, j;//i for rows and j for columns

	//Fill up the first default row (i = key_num * KEY_LENGTH)
	for (j = 1; j <= cons->max_indel_num; j++) {
		path[key_num * KEY_LENGTH][j] = 0;
	}

	for (j = cons->max_indel_num + 1; j <= 2 * cons->max_indel_num + 1; j++) {
		path[key_num * KEY_LENGTH][j] = j - cons->max_indel_num;
	}

	//Fill up the beginning rows
	for (i = key_num * KEY_LENGTH + 1; i <= key_num * KEY_LENGTH + cons->max_indel_num; i++) {
		//fill 0 for imaginary columns
		for (j = 1; j <= error_num - i; j++) {
			path[i][i] = 0;
		}

		//compare and fill the real rows
		for (j = error_num; j <= 2 * cons->max_indel_num + 1; j++) {
			unsigned char distance = minDistance(path, i, j);
			if (test_read[i - 1] == ref_read[j + i - cons->max_indel_num - 1])
				path[i][j] = distance;
			else
				path[i][j] = distance + 1;
		}
	}

	//Normal fill
	for (i = key_num * KEY_LENGTH + cons->max_indel_num + 1; i <= READ_LENGTH - cons->max_indel_num) {
		for (j = 1; j <= 2 * cons->max_indel_num + 1; j++) {
			unsigned char distance = minDistance(path, i, j)
			if (test_read[i - 1] == ref_read[j + i - cons->max_indel_num - 1])
				path[i][j] = distance;
			else
				path[i][j] = distance + 1;
		}
	}

	//
	return;
}

__device__ void fillPath(char const* test_read, char const* ref_read,
		int key_num, unsigned char** path, ED_constant const* cons) {

	//i for rows and j for columns

	//Fill up the first default row (i = key_num * KEY_LENGTH)
	for (int j = 2 * cons->max_indel_num + 1; j >= cons->max_indel_num; j--) {
		path[(key_num  + 1)* KEY_LENGTH][j] = 0;
	}

	for (int j = cons->max_indel_num i - 1; j >= 1; j--) {
		path[key_num * KEY_LENGTH][j] = cons->max_indel_num - j;
	}

	//Fill up the beginning rows
	for (int i = key_num * KEY_LENGTH + 1; i <= cons->max_indel_num; i--) {
		//fill 0 for imaginary columns
		for (j = 1; j <= error_num - i; j++) {
			path[i][i] = 0;
		}

		//compare and fill the real rows
		for (j = error_num; j <= 2 * cons->max_indel_num + 1; j++) {
			unsigned char distance = minDistance(path, i, j);
			if (test_read[i - 1] == ref_read[j + i - cons->max_indel_num - 1])
				path[i][j] = distance;
			else
				path[i][j] = distance + 1;
		}
	}
	return;
}

	//Current distance pointer set to 0
	int cur_dist = 0;
	//Start at the main lane
	int cur_lane = main_lane;
	//Set the first cost of the main lane to 0
	path[cur_lane].path_cost[path[cur_lane].front_idx] = 0;

	bool ED_finished = false;
//String pointers. Used in comparing characters int test_idx;
	int ref_idx;

	//Do string Comparison.
	while (!ED_finished) {
		//First Pick a path.
		/*
		 * This Portion is the divergence portion, where each thread will pick a path to go through.
		 * The path pick may be different. But after a path is picked, there will be no difference.
		 * The picked path will be recorded in cur_lane.
		 */
		while (path[cur_lane].path_cost[path[cur_lane].front_idx] != cur_dist) {
			if (cur_lane == max_indel_num * 2 + 1) { //Next is boundary
				if (cur_dist >= max_diff_num) { //Check if we exceeds the max diff tolarence
					//TODO: BREAK AND END.
					ED_finished = true;
					result.correct = false;
					break;
				} else {
					//Go to next tier
					cur_lane = 1;
					cur_dist++;
				}
			} else {
				cur_lane++;
			}
		}

		//Indicate if the lane should be stopped
		int slide_stop = 0;

		//Slide down the lane.
		/*
		 * If the 2 string does not differ much, which we assume will be the common case after passing the filtering,
		 * then they would loop like 20~100 times. That's why we want to strip out the divergent portion and put it in
		 * a different path picking loop.
		 */
		for (int m = 0; m <= READ_LENGTH - key_num * KEY_LENGTH; m++) { 
		//while (!ED_finished) {
			//Conservative test, speed up common case
			if (path[cur_lane].front_idx >= READ_LENGTH - max_indel_num) {
				//Test if it's the last element
				if ((cur_lane <= main_lane && path[cur_lane].front_idx
						== READ_LENGTH) //Insertion lane
						|| (cur_lane > main_lane && path[cur_lane].front_idx //Deletion Lane
								== READ_LENGTH + main_lane - cur_lane)) {
					ED_finished = true;
					result.correct = true;
					break;
				}
			}

			//Compute string idx:
			test_idx = path[cur_lane].front_idx + cur_lane - main_lane;
			ref_idx = path[cur_lane].front_idx;

			//test if can slide down
			if (!(path[cur_lane].path_cost[path[cur_lane].front_idx + 1]
					== cur_dist //If can just slide
					|| test_read[test_idx] == ref_read[ref_idx]))
				slide_stop = 1;

			//Check neighbor lanes and update them. Modify -> increment -> modify
			//Check lower neighbor
			if (path[cur_lane + 1].path_cost[path[cur_lane].front_idx]
					> cur_dist + slide_stop + 1)
				path[cur_lane + 1].path_cost[path[cur_lane].front_idx]
						= cur_dist + 1;
			//Increment
			path[cur_lane].front_idx++;
			//Check itself
			if (path[cur_lane].path_cost[path[cur_lane].front_idx] > cur_dist
					+ slide_stop)
				path[cur_lane].path_cost[path[cur_lane].front_idx] = cur_dist
						+ slide_stop;
			//Check upper (right) neighbor
			if (path[cur_lane - 1].path_cost[path[cur_lane].front_idx]
					> cur_dist + slide_stop + 1)
				path[cur_lane - 1].path_cost[path[cur_lane].front_idx]
						= cur_dist + 1;

			//stop if can't slide anymore
			if (slide_stop == 1)
				break;
		}
	}
	
	DEBUG_PRINT2("inside editCalFWD, after path cost generation a  threadId: %i\n", threadIdx.x);
	DEBUG_PRINT2("inside editCalFWD, after path cost generation b  threadId: %i\n", threadIdx.x);

	//Tracing back period
	/*
	 * After you got the matrix, and all distance number, you have to go back and find out which path is the
	 * "least divergent path" and also find out where are those insertions and deletions and mismatches.
	 * The code here should not divergent much since each element has to compare it's left, up and up-left neighbor.
	 * The only divergent part is when it see an insertion or mismatch, they will have to go inside the "if block"
	 * and update next iteration position. Otherwise, it's just 3 if tests. Since the "differences" between 2 strings
	 * should be fairly small (at most 5 out of 108 comparisons). I think it's acceptiable.
	 */
	if (result.correct == false)
		return result;
	else { //If pass the test, trace back

		//char temp_result[30]; //Temp string. Used for appending.
		int cur_idx = (cur_lane <= main_lane) ? READ_LENGTH : READ_LENGTH
				+ main_lane - cur_lane;

		result.diff_num = path[cur_lane].path_cost[cur_idx];

		cur_dist = path[cur_lane].path_cost[cur_idx];

		int error_ptr = 0;

		while (cur_lane != main_lane || cur_idx != key_num * KEY_LENGTH) {

			//If we should have an insertion
			if (cur_idx == key_num * KEY_LENGTH
					|| path[cur_lane - 1].path_cost[cur_idx]
							< path[cur_lane].path_cost[cur_idx - 1]) {

DEBUG_PRINT5("1xx6: threadIdx %i / error_ptr: %i / cur_lane: %i main_lane: %i", 
							threadIdx.x, error_ptr, cur_lane, main_lane);
				result.error[error_ptr].diff = INSERTION;
				result.error[error_ptr].location = cur_idx + cur_lane
						- main_lane - 1;
				result.error[error_ptr].diff_char
						= test_read[result.error[error_ptr].location];

				error_ptr++;
				cur_lane--;
				continue;
			}

			//If we should have a deletion
			if (path[cur_lane + 1].path_cost[cur_idx - 1]
					< path[cur_lane].path_cost[cur_idx - 1]) {

				result.error[error_ptr].diff = DELETION;
				result.error[error_ptr].location = cur_idx + cur_lane
						- main_lane - 1;
				result.error[error_ptr].diff_char = ref_read[cur_idx];

				error_ptr++;
				cur_lane++;
				cur_idx--;
				continue;
			}

			//Check if we have a mismatch
			if (path[cur_lane].path_cost[cur_idx - 1]
					< path[cur_lane].path_cost[cur_idx]) {

				result.error[error_ptr].diff = MISMATCH;
				result.error[error_ptr].location = cur_idx + cur_lane
						- main_lane - 1;
				result.error[error_ptr].diff_char
						= test_read[result.error[error_ptr].location];

				error_ptr++;
				cur_idx--;
				continue;
			}

			//Move to the next element
			cur_idx--;
		}
	}
	DEBUG_PRINT2("inside editCalFWD, after tracing back generation a  threadId: %i\n", threadIdx.x);
	DEBUG_PRINT2("inside editCalFWD, after tracing back generation b  threadId: %i\n", threadIdx.x);
	return result;
}

