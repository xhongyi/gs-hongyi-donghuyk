#include "hip/hip_runtime.h"
/*
 * edit_distance.cpp
 *
 *  Created on: Oct 29, 2011
 *      Author: mac
 */

/*
 *
 */
#define		_MAIN_PATH_		0
#define		_DELETE_PATH_	1
#define		_INSERT_PATH_	2
#define		_UN_FILLED_		1000

#include "edit_distance.h"
#include <algorithm>
#include <cstdlib>
//#include <cstring>
#include <iostream>
#include <assert.h>
/*
 *          reference string
 * 		  a p p l i c a t i o n
 *      0 1 2
 * t  a 1 0 1 2
 * e  p 2 1 0 1 2    <----delete_path
 * s  p   2 1 0 1 2
 * t  p     2 1 1 2 3
 *    l       2 1 2 3
 * s  o     ^   2 2 3 <------main_path
 * t  c     |     3 2
 * i  a     |         2
 * n  t     |           2
 * g  i     |             2
 *    o     |               2
 *	        |
 *	    insert_path
 *
 *
 *          reference string
 *        a p p l i c a t i o n
 *      0 1 2 3
 * t  a 1 0 1 2 3
 * e  p 2 1 0 1 2 3  <----bondary_path
 * s  p 3 2 1 0 1 2 3
 * t  p   3 2 1 1 2 3 4
 *    l     3 2 1 2 3 4 5
 * s  o     ^ 3 2 2 3 4 5 <----delete_path
 * t  c     |   3 3 2 3 4
 * i  a     |       3 2 3 4
 * n  i     |         3 3 3 4
 * g  o     |             4 3 4
 *    n     |             ^ 4 3 <------main_path
 *	        |             |
 *	   bondary_path  insert_path
 *      |
 *      | | | 3 3 3 3 4 5
 *      | | 2 2 2 2 3 4 5
 *      | 1 1 1 1 2 3 4 4 4 4 4
 *      0 0 0 0 1 2 3 3 3 3 3 3
 *      1 1 1 1 1 2 2 2 3 4 4 |
 *      2 2 2 2 2 3 3 3   | | |
 *      3 3 3 3 3       | | | |
 *
 */

/*
 * Find the min edit distance givin the location.
 */
__device__ unsigned char minDistance(unsigned char const** path, int row, int column) {
	unsigned char result;
	if (path[row - 1][column] < path[row - 1][column + 1])
		result = path[row - 1][column];
	else
		result = path[row - 1][column + 1];
	
	if (path[row][column - 1] < result)
		result = path[row][column - 1];
	
	return result;
}

/*
 * Going Forward, starting from the key_num * KEY_LENGTH position and going forward until end.
 */
__device__ void fillPath(char const* test_read, char const* ref_read,
		int key_num, unsigned char** path, ED_constant const* cons) {

	//i for rows and j for columns


	//Fill up the first default row (just fill the columns)
	//For the default columns we fill 0
	for (int j = 1; j <= cons->max_indel_num; j++) {
		path[0][j] = 0;
	}
	//Starting from max_indel_num we fill numbers.
	for (int j = cons->max_indel_num + 1; j <= 2 * cons->max_indel_num + 1; j++) {
		path[0][j] = j - cons->max_indel_num - 1;
	}


	//Fill up the beginning rows
	for (int i = 1; i <= 0 + cons->max_indel_num; i++) {
		//fill 0 for imaginary columns
		for (j = 1; j <= error_num + 1 - i; j++) {
			unsigned char distance = minDistance(Path, i, j);
			path[i][j] = distance + 1;
		}

		//compare and fill the real rows
		for (j = error_num; j <= 2 * cons->max_indel_num + 1; j++) {
			unsigned char distance = minDistance(path, i, j);
			if (test_read[i - 1] == ref_read[j + i - cons->max_indel_num - 1])
				path[i][j] = distance;
			else
				path[i][j] = distance + 1;
		}
	}

	//Normal fill
	for (i = 0 + cons->max_indel_num + 1; i <= READ_LENGTH - cons->max_indel_num) {
		for (j = 1; j <= 2 * cons->max_indel_num + 1; j++) {
			unsigned char distance = minDistance(path, i, j);
			if (test_read[i - 1] == ref_read[j + i - cons->max_indel_num - 2])
				path[i][j] = distance;
			else
				path[i][j] = distance + 1;
		}
	}

	//End process. Cleaning up process
	for (i = READ_LENGTH - cons->max_indel_num + 1; i <= READ_LENGTH; i++) {
		for (j = 1; j <= READ_LENGTH + 2 + cons->max_indel_num - i; j++) {
			unsigned char distance = minDistance(path, i, j);
			if (test_read[i - 1] == ref_read[j + i - cons->max_indel_num - 2])
				path[i][j] = distance;
			else
				path[i][j] = distance + 1;
		}

		for (j = READ_LENGTH + 2 + cons->max_indel_num - i + 1;
				j <= 2 * cons->max_indel_num + 1; j++) {
			unsigned char distance = minDistance(path, i, j);
			path[i][j] = distance + 1;
		}
	}
	return;
}

//Filling backward.
/*
__device__ void fillPath(char const* test_read, char const* ref_read,
		int key_num, unsigned char** path, ED_constant const* cons) {

	int i, j;//i for rows and j for columns

	//Fill up the first default row (i = key_num * KEY_LENGTH)
	for (j = 1; j <= cons->max_indel_num; j++) {
		path[key_num * KEY_LENGTH][j] = 0;
	}

	for (j = cons->max_indel_num + 1; j <= 2 * cons->max_indel_num + 1; j++) {
		path[key_num * KEY_LENGTH][j] = j - cons->max_indel_num;
	}

	//Fill up the beginning rows
	for (i = key_num * KEY_LENGTH + 1; i <= key_num * KEY_LENGTH + cons->max_indel_num; i++) {
		//fill 0 for imaginary columns
		for (j = 1; j <= error_num - i; j++) {
			path[i][i] = 0;
		}

		//compare and fill the real rows
		for (j = error_num; j <= 2 * cons->max_indel_num + 1; j++) {
			unsigned char distance = minDistance(path, i, j);
			if (test_read[i - 1] == ref_read[j + i - cons->max_indel_num - 1])
				path[i][j] = distance;
			else
				path[i][j] = distance + 1;
		}
	}

	//Normal fill
	for (i = key_num * KEY_LENGTH + cons->max_indel_num + 1; i <= READ_LENGTH - cons->max_indel_num) {
		for (j = 1; j <= 2 * cons->max_indel_num + 1; j++) {
			unsigned char distance = minDistance(path, i, j)
			if (test_read[i - 1] == ref_read[j + i - cons->max_indel_num - 1])
				path[i][j] = distance;
			else
				path[i][j] = distance + 1;
		}
	}

	//
	return;
}

__device__ void fillPath(char const* test_read, char const* ref_read,
		int key_num, unsigned char** path, ED_constant const* cons) {

	//i for rows and j for columns

	//Fill up the first default row (i = key_num * KEY_LENGTH)
	for (int j = 2 * cons->max_indel_num + 1; j >= cons->max_indel_num; j--) {
		path[(key_num  + 1)* KEY_LENGTH][j] = 0;
	}

	for (int j = cons->max_indel_num i - 1; j >= 1; j--) {
		path[key_num * KEY_LENGTH][j] = cons->max_indel_num - j;
	}

	//Fill up the beginning rows
	for (int i = key_num * KEY_LENGTH + 1; i <= cons->max_indel_num; i--) {
		//fill 0 for imaginary columns
		for (j = 1; j <= error_num - i; j++) {
			path[i][i] = 0;
		}

		//compare and fill the real rows
		for (j = error_num; j <= 2 * cons->max_indel_num + 1; j++) {
			unsigned char distance = minDistance(path, i, j);
			if (test_read[i - 1] == ref_read[j + i - cons->max_indel_num - 1])
				path[i][j] = distance;
			else
				path[i][j] = distance + 1;
		}
	}
	return;
}

