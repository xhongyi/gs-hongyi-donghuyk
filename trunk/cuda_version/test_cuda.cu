/*
 * test_search.cpp
 *
 *  Created on: Oct 19, 2011
 *	  Author: mac
 */
#include <iostream>
#include "universal_header.h"

using namespace std;

int test_cuda(void) {

	long long coord_num = loadHash("toy_hash_0");
    long long ref_num = loadRef("toy_ref_0");

	GPU_fragment test_fragment;
	strcpy(
			test_fragment.fragment,
			"GGGTGGTAGGTGCAGAGACGGGAGGGGCAGAGCCGCAGGCACAGCCAAGAGGGCTGAAGAAATGGTAGAACGGAGCAGCTGGTGATGTGTGGGCCCACCGGCCCCAGG");
	//Getting the sort key.
	key_struct sort_input[KEY_NUMBER];
	for (int i = 0; i < KEY_NUMBER; i++) {
		char key[KEY_LENGTH];
		for (int j = 0; j < KEY_LENGTH; j++) {
			key[j] = test_fragment.fragment[j + KEY_LENGTH * i];
			//key = fragment.substr(KEY_LENGTH * i, KEY_LENGTH);
		}
		int key_hash = hashVal(key);
		int key_entry = hash_table[key_hash];
		int key_entry_size = coordinate[key_entry];
		sort_input[i].order = 0;
		sort_input[i].key_number = i;
		sort_input[i].key_entry = key_entry;
		sort_input[i].key_entry_size = key_entry_size;
	}

	sortPrefilter(test_fragment.sorted_keys, sort_input);
	for (int i = 0; i < KEY_NUMBER; i++) {
		cout << "key_number:" << test_fragment.sorted_keys[i].key_number << " key_entry: " << test_fragment.sorted_keys[i].key_entry << " key_entry_size: " << test_fragment.sorted_keys[i].key_entry_size << " base: " << test_fragment.sorted_keys[i].base << endl;
	}

	cout << "Search Fragment: " << test_fragment.fragment << endl;

	GPU_fragment* dev_fragment;
	final_result* dev_result;
	final_result* test_result = (final_result*) malloc(sizeof(final_result));

	hipMalloc((void**) &dev_fragment, sizeof(GPU_fragment));
	hipMalloc((void**) &dev_result, sizeof(final_result));
	hipMemcpy(dev_fragment, &test_fragment, sizeof(GPU_fragment),
			hipMemcpyHostToDevice);

	searchFragment <<<1, 2>>> (dev_fragment, 1, dev_ref_string, dev_hash_table, dev_coordinate, 5, 5, dev_result);

	hipMemcpy(test_result, dev_result, sizeof(final_result),
			hipMemcpyDeviceToHost);
	cout << "Fragment	 : " << (*test_result).fragment << endl;
	cout << "Spilled	  : " << (*test_result).spilled << endl;
	cout << "Result Size  : " << (*test_result).size << endl;
	for (int i = 0; i < (*test_result).size; i++) {
		cout << "Result coor  : " << (*test_result).coor_results[i].coordiante
				<< endl;
		cout << "Result diff  : " << (*test_result).coor_results[i].diff_num
				<< endl;
	}

	free(test_result);
	hipFree(dev_fragment);
	hipFree(dev_result);
	freeHash();
	freeRef();
	return 0;
}

int main() {
	test_cuda();
	return 0;
}
