/*
 * test_search.cpp
 *
 *  Created on: Oct 19, 2011
 *      Author: mac
 */
#include <list>
#include <iostream>
//#include "book.h"
#include "common.h"
#include "hash_transfer.h"

using namespace std;

int ref_test(void) {
	char* tmp_ref_string;
	long long ref_num = loadRef("toy_ref_0");
	fprintf(stdout, "reference number: %lli  \n", ref_num);
	char* str_ref_string = (char*)malloc(sizeof(char)*ref_num);
	hipMalloc((void**)&tmp_ref_string, ref_num*sizeof(char));
	loadRefTest <<<1, 1>>> (tmp_ref_string, ref_num);
	hipMemcpy(str_ref_string, tmp_ref_string, ref_num*sizeof(char), hipMemcpyDeviceToHost );
	bool error_ref_string = false;
	for(int i = 0; i<ref_num; i++){
		if(ref_string[i] != str_ref_string[i])
			error_ref_string = true;
	}
	if (error_ref_string == true)
		fprintf(stdout, "Ref String ERROR!! \n");
	else
		fprintf(stdout, "Ref String PASSED! \n");

	hipFree(tmp_ref_string);
	freeRef();
	return 0;
}

int main() {
	ref_test();
	hash_test();
	return 0;
}
