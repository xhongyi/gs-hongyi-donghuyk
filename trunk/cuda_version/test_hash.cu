/*
 * test_search.cpp
 *
 *  Created on: Oct 19, 2011
 *      Author: mac
 */
#include <list>
#include <iostream>
//#include "ref_db_generator.h"
#include "fragment_match.h"

using namespace std;

int main() {
	set_max_indel_num(1);
	set_max_diff_num(2);
	loadHash("toy_hash_0");
//-----------------------------------------------------------------------------------------------------------------------------------------------------------------------------------
	int  *total_num = (int*) malloc(sizeof(int));
	long *index_num = (int*) malloc(sizeof(long)*INDEX_NUM); 
	char *gen_distribution_file = (char*) malloc(sizeof(hash_table_name)+20);
	for (int l = 0 ; l < INDEX_NUM ; l++) {
		index_num[l] = 0;
	}

	int  *dev_total_num;
	long *dev_index_num;
	HANDLE_ERROR( hipMalloc( (void**)&dev_total_num, sizeof(int) ));
	HANDLE_ERROR( hipMalloc( (void**)&dev_index_num, sizeof(int)*INDEX_NUM ));

	HANDLE_ERROR( hipMemcpy(total_num, dev_total_num, sizeof(int), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy(index_num, dev_index_num, sizeof(int)*INDEX_NUM, hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipFree( dev_total_num ) );
	HANDLE_ERROR( hipFree( dev_index_num ) );
	freeHash();
//-----------------------------------------------------------------------------------------------------------------------------------------------------------------------------------
	FILE * pFileR;
	FILE * pFileOut;
	int    sum_number;
	sprintf(gen_distribution_file, "%s%s\0", hash_table_name, "_distribution");
	pFileOut = fopen (gen_distribution_file, "w");
	fprintf(pFileOut, "total_number : %i \n", total_number);
	for (int i = 0 ; i < INDEX_NUM ; i++) {
		if (index_num[i] != 0) {
			fprintf(pFileOut, "%i : %li \n", i, index_num[i]);
			sum_number = sum_number + index_num[i];
		}
	}
	fprintf(pFileOut, "sum_number : %i \n", sum_number);
	fclose(pFileOut);
	free(gen_distribution_file);
	return 0;
}
