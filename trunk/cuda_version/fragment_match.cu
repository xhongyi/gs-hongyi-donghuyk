#include "hip/hip_runtime.h"
/*
 * fragment_match.cpp
 *
 *  Created on: Oct 8, 2011
 *      Author: mac
 */
#include <cstdlib>
#include <sstream>
#include <iostream>
#include <assert.h>
#include "common.h"
#include "edit_distance.h"
#include "fragment_match.h"

bool select_cheapest;

__device__ bool searchKey(int target_coor, int entry_coor, int entry_size,
		int* coordinate, int max_indel_num);

__device__ bool searchKey(int target_coor, int entry_coor, int entry_size,
		int* coordinate, int max_indel_num) {
	if (entry_size == 0)
		return false;
	int lower_bound = entry_coor + 1;
	int upper_bound = entry_coor + entry_size;
	int mid = lower_bound + entry_size / 2;
	while (lower_bound < upper_bound) {
		if (coordinate[mid] == target_coor)
			break;
		else if (coordinate[mid] < target_coor)
			lower_bound = mid + 1;
		else
			upper_bound = mid - 1;
		mid = lower_bound + (upper_bound - lower_bound) / 2;
	}

	if (coordinate[mid] <= target_coor + max_indel_num && coordinate[mid]
			>= target_coor - max_indel_num) {
		return true;
	} else
		return false;
}

bool sortPrefilter(key_struct* sort_result, key_struct* sort_input) {
	if (select_cheapest == false) {
		for (int i = 0; i < KEY_NUMBER; i++) {
			sort_result[i].key_number = sort_input[i].key_number;
			sort_result[i].key_entry = sort_input[i].key_entry;
			sort_result[i].key_entry_size = sort_input[i].key_entry_size;
		}
		return true;
	}
	for (int i = 0; i < KEY_NUMBER; i++) {
		for (int j = 0; j < KEY_NUMBER; j++) {
			if (sort_input[i].key_entry_size > sort_input[j].key_entry_size) {
				sort_input[i].order = sort_input[i].order + 1;
			}
		}
	}
	int loop_index = 0;
	for (int i = 0; i < KEY_NUMBER; i++) {
		for (int j = 0; j < KEY_NUMBER; j++) {
			if (sort_input[j].order == i) {
				sort_result[loop_index].key_entry = sort_input[j].key_entry;
				sort_result[loop_index].key_number = sort_input[j].key_number;
				sort_result[loop_index].key_entry_size
						= sort_input[j].key_entry_size;
				loop_index = loop_index + 1;
			}
			//if (loop_index == max_diff_num + 1)
			//	return true;
		}
	}
	return true;//false;
}

__global__ void searchFragment(GPU_fragment* fragment, int fragment_size, char* ref,
		int* hash_table, int* coordinate, int max_diff_num, int max_indel_num,
		final_result* result) {
	//This will be used in edit_distance Calculation.
	int main_lane = max_indel_num + 1;
	//Each thread will have a path array for edit_distance calculation.
	ED_path path[MAX_ERROR_NUM];

	//Fragment_counter: get cooresponding fragment
	__shared__ int fragment_count;
	__shared__ int size;
	if (threadIdx.x == 0) {
		fragment_count = blockIdx.x;
	}

	while (blockIdx.x < fragment_size) {
		//get the corresponding key_num and it's coordinate.
		int coor_count = threadIdx.x;
		int cur_key = 0;


		int target_coor =
				fragment[fragment_count].sorted_keys[cur_key].key_entry + 1
						+ coor_count
						- fragment[fragment_count].sorted_keys[cur_key].base;
		if (threadIdx.x == 0) {
			size = -1;
		}

		do {
			//Do adjacency filtering
			int diff_num = 0;

			for (int i = 0; i < KEY_NUMBER; i++) { //for each segment
				if (i - diff_num > KEY_NUMBER - max_diff_num)
					break;

				if (!searchKey(
						target_coor
								+ (fragment[fragment_count].sorted_keys[i].key_number
										- fragment[fragment_count].sorted_keys[cur_key].key_number)
										* KEY_LENGTH,
						fragment[fragment_count].sorted_keys[i].key_entry,
						fragment[fragment_count].sorted_keys[i].key_entry_size, coordinate, max_indel_num)) {
					diff_num++;
					if (diff_num > max_diff_num)
						break;
				}
			}

			//Edit_distance Calculation
			if (diff_num <= max_diff_num) {
				char ref_str[READ_LENGTH];
				for (int i = 0; i < READ_LENGTH; i++) //Get reference string
					ref_str[i]
							= ref[target_coor
									- fragment[fragment_count].sorted_keys[cur_key].key_number
											* KEY_LENGTH + i];

				ED_result
						edit_result =
								editDistanceCal(
										fragment[fragment_count].fragment,
										ref_str,
										fragment[fragment_count].sorted_keys[cur_key].key_number,
										path, main_lane, max_indel_num, max_diff_num);

				if (edit_result.correct) {
					atomicAdd(&size, 1);
					result[fragment_count].coor_results[size].coordiante
							= target_coor
									- fragment[fragment_count].sorted_keys[cur_key].key_number
											* KEY_LENGTH;
					result[fragment_count].coor_results[size].diff_num
							= edit_result.diff_num;
					for (int i = 0; i < edit_result.diff_num; i++)
						result[fragment_count].coor_results[size].error[i]
								= edit_result.error[i];
				}
			}

			//move to the next coordinate by incrementing coor_count
			coor_count += blockDim.x;
			//Move to the next key
			while (cur_key <= max_diff_num
					&& coor_count
							>= fragment[fragment_count].sorted_keys[coor_count].base
									+ fragment[fragment_count].sorted_keys[coor_count].key_entry_size) {
				cur_key++;
			}
		} while (size <= MAX_COOR_RESULT_NUM && cur_key < max_indel_num);

		__syncthreads();

		if (threadIdx.x == 0) {
			for (int i = 0; i < READ_LENGTH; i++)
				result[fragment_count].fragment[i] = fragment[fragment_count].fragment[i];
			if (size > MAX_COOR_RESULT_NUM )
				result[fragment_count].spilled = true;
			else {
				result[fragment_count].spilled = false;
				result[fragment_count].size = size;
			}
			fragment_count += gridDim.x;
		}

	}
}

/*
 final_result searchFragment(char* fragment, char* ref, int* hash_table,
 int* coordinate) {
 key_struct sort_input[KEY_NUMBER];
 for (int i = 0; i < KEY_NUMBER; i++) {
 char key[KEY_LENGTH];
 for (int j = 0; j < KEY_NUMBER; j++) {
 key[j] = fragment[j + KEY_LENGTH * i];
 //key = fragment.substr(KEY_LENGTH * i, KEY_LENGTH);
 }
 int key_hash = hashVal(key);
 int key_entry = hash_table[key_hash];
 int key_entry_size = coordinate[key_entry];
 sort_input[i].order = 0;
 sort_input[i].key_number = i;
 sort_input[i].key_entry = key_entry;
 sort_input[i].key_entry_size = key_entry_size;
 }

 key_struct keys_input[KEY_NUMBER];
 sortPrefilter(keys_input, sort_input);
 previous_result.size = 0;
 final_result return_result;
 return_result.total_binary_search = 0;
 return_result.total_edit_perform = 0;
 return_result.total_correct_num = 0;

 for (int k = 0; k < max_diff_num + 1; k++) { //Getting the seed
 for (int i = keys_input[k].key_entry + 1; i <= keys_input[k].key_entry
 + keys_input[k].key_entry_size; i++) { //Going through each entry in the seed coordinate list
 int coor_value = coordinate[i];
 int diff_num = 0;
 if (!searchPrevious(coor_value, keys_input[k].key_number,
 previous_result)) { //Pre-filtering
 return_result.total_binary_search++;
 for (int j = 0; j < KEY_NUMBER; j++) { //for each segment
 if (j - diff_num > KEY_NUMBER - max_diff_num)
 break;
 if (!searchKey(
 coor_value + (keys_input[j].key_number
 - keys_input[k].key_number) * KEY_LENGTH,
 keys_input[j].key_entry,
 keys_input[j].key_entry_size)) {
 diff_num++;
 if (diff_num > max_diff_num)
 break;
 }
 }
 if (diff_num <= max_diff_num) {
 if (previous_result.size <= PREFILTER_SIZE) {
 previous_result.coor[previous_result.size] = coor_value
 - keys_input[k].key_number * KEY_LENGTH; //start_coor;
 previous_result.size++;
 }
 return_result.total_edit_perform++;
 string ref_str(FRAGMENT_LENGTH, 'A');
 ref_str = (*ref).substr(
 coor_value - keys_input[k].key_number * KEY_LENGTH,
 FRAGMENT_LENGTH); //start_coor;

 /////////////////////Just For Testing
 char test_char[READ_LENGTH];
 char ref_char[READ_LENGTH];
 strcpy(test_char, fragment.c_str());
 strcpy(ref_char, ref_str.c_str());
 cout << "ref__read: " << ref_char << endl;
 cout << "test_read: " << test_char << endl;
 cout << "key_num__: " << keys_input[k].key_number << endl;
 /////////////////////Testing END

 ED_result edit_result = editDistanceCal(test_char,
 ref_char, keys_input[k].key_number);
 if (edit_result.correct) {
 return_result.total_correct_num++;
 //cout << "ref_read      : " << ref_str << "  coordinate: "<< (*it_result).coordinate << "  Key_number: "<< (*it_result).key_number;
 //cout << "  result: correct " << endl;
 } else {
 //cout << "ref_read      : " << ref_str << "  coordinate: "<< (*it_result).coordinate << "  Key_number: "<< (*it_result).key_number;
 //cout << "  result: not correct" <<endl;
 }
 }
 }
 }
 }
 return return_result;
 }
 */

