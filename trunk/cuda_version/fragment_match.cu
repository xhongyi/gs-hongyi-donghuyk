/*
 * fragment_match.cpp
 *
 *  Created on: Oct 8, 2011
 *      Author: mac
 */
#include <cstdlib>
#include <sstream>
#include <iostream>
#include <assert.h>
#include "common.h"
#include "fragment_match.h"
#include "book.h"

int* hash_table;
int* coordinate;
int* dev_coordinate;
int* dev_hash_table;
texture<int> tex_hash_table;
texture<int> tex_coordinate;

void loadHash(string hash_name) {
	long long coord_num = hashReconstructor(&hash_table, &coordinate, hash_name.c_str());
	HANDLE_ERROR(hipMalloc((void**)&dev_hash_table, INDEX_NUM*sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_coordinate, coord_num*sizeof(int)));
	HANDLE_ERROR(hipBindTexture(NULL, tex_hash_table, dev_hash_table, INDEX_NUM*sizeof(int)));
	HANDLE_ERROR(hipBindTexture(NULL, tex_coordinate, dev_coordinate, coord_num*sizeof(int)));
	HANDLE_ERROR(hipMemcpy(dev_coordinate, coordinate, coord_num*sizeof(int), hipMemcpyHostToDevice ));
	HANDLE_ERROR(hipMemcpy(dev_hash_table, hash_table, INDEX_NUM*sizeof(int), hipMemcpyHostToDevice ));
}

void freeHash(void) {
	HANDLE_ERROR( hipUnbindTexture ( tex_hash_table) );
	HANDLE_ERROR( hipUnbindTexture ( tex_coordinate) );
	HANDLE_ERROR( hipFree ( dev_hash_table) );
	HANDLE_ERROR( hipFree ( dev_coordinate) );
}
