#include "hip/hip_runtime.h"
/*
 * fragment_match.cpp
 *
 *  Created on: Oct 8, 2011
 *      Author: mac
 */
#include <cstdlib>
#include <sstream>
#include <iostream>
#include <assert.h>
#include "common.h"
#include "edit_distance.h"
#include "fragment_match.h"

bool select_cheapest;

__device__ bool searchKey(int target_coor, int entry_coor, int entry_size,
		int* coordinate, int max_indel_num);

__device__ bool searchKey(int target_coor, int entry_coor, int entry_size,
		int* coordinate, int max_indel_num) {
	printf("1\n");
	if (entry_size == 0)
		return false;
	int lower_bound = entry_coor + 1;
	int upper_bound = entry_coor + entry_size;
	int mid = lower_bound + entry_size / 2;
	while (lower_bound < upper_bound) {
		if (coordinate[mid] == target_coor)
			break;
		else if (coordinate[mid] < target_coor)
			lower_bound = mid + 1;
		else
			upper_bound = mid - 1;
		mid = lower_bound + (upper_bound - lower_bound) / 2;
	}
	printf("2\n");

	printf("mid: %i\n", mid);
	printf("coordinate[mid]: %i\n", coordinate[mid]);

	if (coordinate[mid] <= target_coor + max_indel_num && coordinate[mid]
			>= target_coor - max_indel_num) {
		printf("3 true\n");
		return true;
	} else {
		printf("3 false\n");
		return false;
	}
}

bool sortPrefilter(key_struct* sort_result, key_struct* sort_input) {
	for (int i = 0; i < KEY_NUMBER; i++) {
		for (int j = 0; j < KEY_NUMBER; j++) {
			if (sort_input[i].key_entry_size > sort_input[j].key_entry_size) {
				sort_input[i].order = sort_input[i].order + 1;
			}
		}
	}
	int accumulative_base = 0;
	int loop_index = 0;
	for (int i = 0; i < KEY_NUMBER; i++) {
		for (int j = 0; j < KEY_NUMBER; j++) {
			if (sort_input[j].order == i) {
				sort_result[loop_index].key_entry = sort_input[j].key_entry;
				sort_result[loop_index].key_number = sort_input[j].key_number;
				sort_result[loop_index].key_entry_size
						= sort_input[j].key_entry_size;
				sort_result[loop_index].base = accumulative_base;
				accumulative_base += sort_result[loop_index].key_entry_size;
				loop_index = loop_index + 1;
			}
			//if (loop_index == max_diff_num + 1)
			//	return true;
		}
	}
	return true;//false;
}

__global__ void searchFragment(GPU_fragment* fragment, int fragment_size,
		char* ref, int* hash_table, int* coordinate, int max_diff_num,
		int max_indel_num, final_result* result) {
	printf("HELLO!!\n");
	printf("cuda fragment: %c\n", (*fragment).fragment[0]);
	printf("blockIdx.x: %i, threadIdx.x: %i\n", blockIdx.x, threadIdx.x);
	//This will be used in edit_distance Calculation.
	int main_lane = max_indel_num + 1;
	//Each thread will have a path array for edit_distance calculation.
	ED_path path[MAX_ERROR_NUM];

	//Fragment_counter: get cooresponding fragment
	__shared__ int fragment_count;
	__shared__ int size;
	if (threadIdx.x == 0) {
		printf("Hey Man!!! Here is threadIdx.x == 0\n");
		fragment_count = blockIdx.x;
	}

	while (fragment_count < fragment_size) {
		printf("Ever started!!\n");
		//get the corresponding key_num and it's coordinate.
		int coor_count = threadIdx.x;
		int cur_key = 0;

		if (threadIdx.x == 0) {
			size = 0;
		}

		do {
			printf("Hey doing something!\n");
			//Do adjacency filtering
			int diff_num = 0;
			int
					coor_idx =
							fragment[fragment_count].sorted_keys[cur_key].key_entry
									+ 1 + coor_count
									- fragment[fragment_count].sorted_keys[cur_key].base;
			printf("fragment[fragment_count].sorted_keys[cur_key].key_entry: %i\n", fragment[fragment_count].sorted_keys[cur_key].key_entry);
			printf("fragment[fragment_count].sorted_keys[cur_key].base: %i\n", fragment[fragment_count].sorted_keys[cur_key].base);
			printf("fragment[fragment_count].sorted_keys[cur_key].key_entry_size: %i\n", fragment[fragment_count].sorted_keys[cur_key].key_entry_size);
			printf("***fragment_count: %i, cur_key: %i, coor_count: %i, coor_idx: %i\n", fragment_count, cur_key,coor_count, coor_idx);

			for (int i = 0; i < KEY_NUMBER; i++) { //for each segment
				if (i - diff_num > KEY_NUMBER - max_diff_num)
					break;

				printf(
						"coor_idx + (fragment[fragment_count].sorted_keys[i].key_number - fragment[fragment_count].sorted_keys[cur_key].key_number) * KEY_LENGTH: %i\n",
						(coordinate[coor_idx]
								+ (fragment[fragment_count].sorted_keys[i].key_number
										- fragment[fragment_count].sorted_keys[cur_key].key_number)
										* KEY_LENGTH));
				printf(
						"fragment[fragment_count].sorted_keys[i].key_entry: %i\n",
						fragment[fragment_count].sorted_keys[i].key_entry);
				printf(
						"fragment[fragment_count].sorted_keys[i].key_entry_size: %i\n",
						fragment[fragment_count].sorted_keys[i].key_entry_size);

				if (!searchKey(
						coordinate[coor_idx]
								+ (fragment[fragment_count].sorted_keys[i].key_number
										- fragment[fragment_count].sorted_keys[cur_key].key_number)
										* KEY_LENGTH,
						fragment[fragment_count].sorted_keys[i].key_entry,
						fragment[fragment_count].sorted_keys[i].key_entry_size,
						coordinate, max_indel_num)) {
					diff_num++;
					printf("4\n");
					if (diff_num > max_diff_num)
						break;
				}
			}
			printf("5\n");
			//Edit_distance Calculation
			if (diff_num <= max_diff_num) {
				printf("Hey in ED test now\n");
				printf("cur_key: %i\n", cur_key);
				char ref_str[READ_LENGTH];
				for (int i = 0; i < READ_LENGTH; i++) { //Get reference string
					printf(
							"coordinate: %i\n",
							coordinate[coor_idx]
									- fragment[fragment_count].sorted_keys[cur_key].key_number
											* KEY_LENGTH + i);
					ref_str[i]
							= ref[coordinate[coor_idx]
									- fragment[fragment_count].sorted_keys[cur_key].key_number
											* KEY_LENGTH + i];
					printf("ref_str[%i]: %c\n", i, ref_str[i]);
				}
				printf("7\n");

				ED_result
						edit_result =
								editDistanceCal(
										fragment[fragment_count].fragment,
										ref_str,
										fragment[fragment_count].sorted_keys[cur_key].key_number,
										path, main_lane, max_indel_num,
										max_diff_num);

				if (edit_result.correct) {
					printf("find something!\n");
					atomicAdd(&size, 1);
					result[fragment_count].coor_results[size - 1].coordiante
							= coordinate[coor_idx]
									- fragment[fragment_count].sorted_keys[cur_key].key_number
											* KEY_LENGTH;
					result[fragment_count].coor_results[size - 1].diff_num
							= edit_result.diff_num;
					for (int i = 0; i < edit_result.diff_num; i++)
						result[fragment_count].coor_results[size - 1].error[i]
								= edit_result.error[i];
				}
			}
			printf("6\n");
			//move to the next coordinate by incrementing coor_count
			printf("blockDim.x: %i\n", blockDim.x);
			coor_count += blockDim.x;
			//Move to the next key
			while (cur_key <= max_diff_num
					&& coor_count
							>= fragment[fragment_count].sorted_keys[cur_key].base
									+ fragment[fragment_count].sorted_keys[cur_key].key_entry_size) {
				printf("&&&cur_key incrementing!!\n");
				cur_key++;
			}
			printf("still here!!!\n");
		} while (size < MAX_COOR_RESULT_NUM && cur_key <= max_diff_num);

		printf("before syncthreads\n");

		__syncthreads();

		printf("print Ids again!! blockIdx.x: %i, threadIdx.x: %i\n",
				blockIdx.x, threadIdx.x);

		if (threadIdx.x == 0) {
			printf("I'm Here!!!\n");
			for (int i = 0; i < READ_LENGTH; i++) {
				result[fragment_count].fragment[i]
						= fragment[fragment_count].fragment[i];
				printf("result[%i].fragment[%i]: %c\n", fragment_count, i,
						result[fragment_count].fragment[i]);
			}
			if (size >= MAX_COOR_RESULT_NUM)
				result[fragment_count].spilled = true;
			else {
				result[fragment_count].spilled = false;
				result[fragment_count].size = size;
			}
			fragment_count += gridDim.x;
		}

	}
}

/*
 final_result searchFragment(char* fragment, char* ref, int* hash_table,
 int* coordinate) {
 key_struct sort_input[KEY_NUMBER];
 for (int i = 0; i < KEY_NUMBER; i++) {
 char key[KEY_LENGTH];
 for (int j = 0; j < KEY_NUMBER; j++) {
 key[j] = fragment[j + KEY_LENGTH * i];
 //key = fragment.substr(KEY_LENGTH * i, KEY_LENGTH);
 }
 int key_hash = hashVal(key);
 int key_entry = hash_table[key_hash];
 int key_entry_size = coordinate[key_entry];
 sort_input[i].order = 0;
 sort_input[i].key_number = i;
 sort_input[i].key_entry = key_entry;
 sort_input[i].key_entry_size = key_entry_size;
 }

 key_struct keys_input[KEY_NUMBER];
 sortPrefilter(keys_input, sort_input);
 previous_result.size = 0;
 final_result return_result;
 return_result.total_binary_search = 0;
 return_result.total_edit_perform = 0;
 return_result.total_correct_num = 0;

 for (int k = 0; k < max_diff_num + 1; k++) { //Getting the seed
 for (int i = keys_input[k].key_entry + 1; i <= keys_input[k].key_entry
 + keys_input[k].key_entry_size; i++) { //Going through each entry in the seed coordinate list
 int coor_value = coordinate[i];
 int diff_num = 0;
 if (!searchPrevious(coor_value, keys_input[k].key_number,
 previous_result)) { //Pre-filtering
 return_result.total_binary_search++;
 for (int j = 0; j < KEY_NUMBER; j++) { //for each segment
 if (j - diff_num > KEY_NUMBER - max_diff_num)
 break;
 if (!searchKey(
 coor_value + (keys_input[j].key_number
 - keys_input[k].key_number) * KEY_LENGTH,
 keys_input[j].key_entry,
 keys_input[j].key_entry_size)) {
 diff_num++;
 if (diff_num > max_diff_num)
 break;
 }
 }
 if (diff_num <= max_diff_num) {
 if (previous_result.size <= PREFILTER_SIZE) {
 previous_result.coor[previous_result.size] = coor_value
 - keys_input[k].key_number * KEY_LENGTH; //start_coor;
 previous_result.size++;
 }
 return_result.total_edit_perform++;
 string ref_str(FRAGMENT_LENGTH, 'A');
 ref_str = (*ref).substr(
 coor_value - keys_input[k].key_number * KEY_LENGTH,
 FRAGMENT_LENGTH); //start_coor;

 /////////////////////Just For Testing
 char test_char[READ_LENGTH];
 char ref_char[READ_LENGTH];
 strcpy(test_char, fragment.c_str());
 strcpy(ref_char, ref_str.c_str());
 cout << "ref__read: " << ref_char << endl;
 cout << "test_read: " << test_char << endl;
 cout << "key_num__: " << keys_input[k].key_number << endl;
 /////////////////////Testing END

 ED_result edit_result = editDistanceCal(test_char,
 ref_char, keys_input[k].key_number);
 if (edit_result.correct) {
 return_result.total_correct_num++;
 //cout << "ref_read      : " << ref_str << "  coordinate: "<< (*it_result).coordinate << "  Key_number: "<< (*it_result).key_number;
 //cout << "  result: correct " << endl;
 } else {
 //cout << "ref_read      : " << ref_str << "  coordinate: "<< (*it_result).coordinate << "  Key_number: "<< (*it_result).key_number;
 //cout << "  result: not correct" <<endl;
 }
 }
 }
 }
 }
 return return_result;
 }

 */
