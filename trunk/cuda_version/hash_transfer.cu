#include "hip/hip_runtime.h"
/*
 * fragment_match.cpp
 *
 *  Created on: Oct 8, 2011
 *      Author: mac
 */
#include <cstdlib>
#include <sstream>
#include <iostream>
#include <assert.h>
#include "common.h"
#include "book.h"
#include "fragment_match.h"

int* hash_table;
int* coordinate;
int* dev_coordinate;
int* dev_hash_table;
texture<int> tex_hash_table;
texture<int> tex_coordinate;

void getHashTablePtr(int ** ptr) {
	*ptr = hash_table;
}

void getCoordinatePtr(int ** ptr) {
	*ptr = coordinate;
}

long long loadHash(string hash_name) {
	long long coord_num = hashReconstructor(&hash_table, &coordinate, hash_name.c_str());
	HANDLE_ERROR(hipMalloc((void**)&dev_hash_table, INDEX_NUM*sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_coordinate, coord_num*sizeof(int)));
	HANDLE_ERROR(hipBindTexture(NULL, tex_hash_table, dev_hash_table, INDEX_NUM*sizeof(int)));
	HANDLE_ERROR(hipBindTexture(NULL, tex_coordinate, dev_coordinate, coord_num*sizeof(int)));
	HANDLE_ERROR(hipMemcpy(dev_coordinate, coordinate, coord_num*sizeof(int), hipMemcpyHostToDevice ));
	HANDLE_ERROR(hipMemcpy(dev_hash_table, hash_table, INDEX_NUM*sizeof(int), hipMemcpyHostToDevice ));
	return coord_num;
}

void freeHash(void) {
	HANDLE_ERROR( hipUnbindTexture ( tex_hash_table) );
	HANDLE_ERROR( hipUnbindTexture ( tex_coordinate) );
	HANDLE_ERROR( hipFree ( dev_hash_table) );
	HANDLE_ERROR( hipFree ( dev_coordinate) );
}

__global__ void loadHashTest(int *tmp_coordinate, int *tmp_hash_table, long long coord_num) {
	for(int i = 0; i < INDEX_NUM; i++) {
		tmp_hash_table[i] = tex1Dfetch(tex_hash_table, i);
	}
	for(int i = 0; i < coord_num; i++) {
		tmp_coordinate[i] = tex1Dfetch(tex_coordinate, i);
	}
}

int hash_test(void) {
	int* tmp_coordinate;
	int* tmp_hash_table;
	long long coord_num = loadHash("toy_hash_0");
	fprintf(stdout, " coordinate number: %lli  \n", coord_num);
	int* str_hash_table = (int*)malloc(sizeof(int)*INDEX_NUM);
	int* str_coordinate = (int*)malloc(sizeof(int)*coord_num);
	HANDLE_ERROR(hipMalloc((void**)&tmp_hash_table, INDEX_NUM*sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&tmp_coordinate, coord_num*sizeof(int)));
	loadHashTest <<<1, 1>>> (tmp_coordinate, tmp_hash_table, coord_num);
	HANDLE_ERROR(hipMemcpy(str_hash_table, tmp_hash_table, INDEX_NUM*sizeof(int), hipMemcpyDeviceToHost ));
	HANDLE_ERROR(hipMemcpy(str_coordinate, tmp_coordinate, coord_num*sizeof(int), hipMemcpyDeviceToHost ));
	bool error_hash_table = false;
	bool error_coordinate = false;
	for(int i = 0; i<INDEX_NUM; i++){
		if(hash_table[i] != str_hash_table[i]) error_hash_table = true;
	}
	for(int i = 0; i<coord_num; i++){
		if(coordinate[i] != str_coordinate[i]) error_coordinate = true;
	}
	if (error_hash_table == true)   fprintf(stdout, " Hash table ERROR!! \n");
	else			    	fprintf(stdout, " Hash table PASSED! \n");
	if (error_coordinate == true)   fprintf(stdout, " Coordinate ERROR!! \n");
	else			    	fprintf(stdout, " Coordinate PASSED! \n");

	HANDLE_ERROR( hipFree( tmp_hash_table ) );
	HANDLE_ERROR( hipFree( tmp_coordinate ) );
	freeHash();
	return 0;
}

