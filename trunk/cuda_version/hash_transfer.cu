#include "hip/hip_runtime.h"
/*
 * hash_transfer.cpp
 *
 *  Created on: Oct 8, 2011
 *      Author: mac
 */
#include <cstdlib>
#include <sstream>
#include <iostream>
#include <assert.h>
#include "common.h"
#include "book.h"
#include "hash_transfer.h"

int* hash_table;
int* coordinate;
string ref_string;
int* dev_coordinate;
int* dev_hash_table;
char* dev_ref_string;
texture<int> tex_hash_table;
texture<int> tex_coordinate;
texture<char> tex_ref_string;

void getHashTablePtr(int ** ptr) {
	*ptr = hash_table;
}

void getCoordinatePtr(int ** ptr) {
	*ptr = coordinate;
}

long long loadHash(string hash_name) {
	long long coord_num = hashReconstructor(&hash_table, &coordinate, hash_name.c_str());
	HANDLE_ERROR(hipMalloc((void**)&dev_hash_table, INDEX_NUM*sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_coordinate, coord_num*sizeof(int)));
	HANDLE_ERROR(hipBindTexture(NULL, tex_hash_table, dev_hash_table, INDEX_NUM*sizeof(int)));
	HANDLE_ERROR(hipBindTexture(NULL, tex_coordinate, dev_coordinate, coord_num*sizeof(int)));
	HANDLE_ERROR(hipMemcpy(dev_coordinate, coordinate, coord_num*sizeof(int), hipMemcpyHostToDevice ));
	HANDLE_ERROR(hipMemcpy(dev_hash_table, hash_table, INDEX_NUM*sizeof(int), hipMemcpyHostToDevice ));
	return coord_num;
}

int loadRef(string ref_name) {
        char tmp_ref_name[50];
        strcpy(tmp_ref_name, ref_name.c_str());
        refLoader(ref_string, tmp_ref_name);
	char *tmp_ref_string = (char*)malloc( ref_string.size() * sizeof(char));
        strcpy(tmp_ref_string, ref_string.c_str());
	HANDLE_ERROR(hipMalloc((void**)&dev_ref_string, ref_string.size() * sizeof(char)));
        HANDLE_ERROR(hipBindTexture(NULL, tex_ref_string, dev_ref_string, ref_string.size() * sizeof(char)));
        HANDLE_ERROR(hipMemcpy(dev_ref_string, tmp_ref_string, ref_string.size() * sizeof(char), hipMemcpyHostToDevice ));
        return ref_string.size();
}

void freeHash(void) {
	free (hash_table);
	free (coordinate);
	HANDLE_ERROR( hipUnbindTexture (tex_hash_table) );
	HANDLE_ERROR( hipUnbindTexture (tex_coordinate) );
	HANDLE_ERROR( hipFree (dev_hash_table) );
	HANDLE_ERROR( hipFree (dev_coordinate) );
}

void freeRef(void) {
	HANDLE_ERROR( hipUnbindTexture (tex_ref_string) );
	HANDLE_ERROR( hipFree (dev_ref_string) );
}

__global__ void loadHashTest(int *tmp_coordinate, int *tmp_hash_table, long long coord_num) {
	for(int i = 0; i < INDEX_NUM; i++) {
		tmp_hash_table[i] = tex1Dfetch(tex_hash_table, i);
	}
	for(int i = 0; i < coord_num; i++) {
		tmp_coordinate[i] = tex1Dfetch(tex_coordinate, i);
	}
}

__global__ void loadRefTest(char *tmp_ref_string, long long ref_num) {
	for(int i = 0; i < ref_num; i++) {
		tmp_ref_string[i] = tex1Dfetch(tex_ref_string, i);
	}
}

int ref_test(void) {
	char* tmp_ref_string;
	long long ref_num = loadRef("toy_ref_0");
	fprintf(stdout, "reference number: %lli  \n", ref_num);
	char* str_ref_string = (char*)malloc(sizeof(char)*ref_num);
	HANDLE_ERROR(hipMalloc((void**)&tmp_ref_string, ref_num*sizeof(char)));
	loadRefTest <<<1, 1>>> (tmp_ref_string, ref_num);
	HANDLE_ERROR(hipMemcpy(str_ref_string, tmp_ref_string, ref_num*sizeof(char), hipMemcpyDeviceToHost ));
	bool error_ref_string = false;
	for(int i = 0; i<ref_num; i++){
		if(ref_string[i] != str_ref_string[i])
			error_ref_string = true;
	}
	if (error_ref_string == true)
		fprintf(stdout, "Ref String ERROR!! \n");
	else
		fprintf(stdout, "Ref String PASSED! \n");

	HANDLE_ERROR( hipFree(tmp_ref_string) );
	freeRef();
	return 0;
}

int hash_test(void) {
	int* tmp_coordinate;
	int* tmp_hash_table;
	long long coord_num = loadHash("toy_hash_0");
	fprintf(stdout, "coordinate number: %lli  \n", coord_num);
	int* str_hash_table = (int*)malloc(sizeof(int)*INDEX_NUM);
	int* str_coordinate = (int*)malloc(sizeof(int)*coord_num);
	HANDLE_ERROR(hipMalloc((void**)&tmp_hash_table, INDEX_NUM*sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&tmp_coordinate, coord_num*sizeof(int)));
	loadHashTest <<<1, 1>>> (tmp_coordinate, tmp_hash_table, coord_num);
	HANDLE_ERROR(hipMemcpy(str_hash_table, tmp_hash_table, INDEX_NUM*sizeof(int), hipMemcpyDeviceToHost ));
	HANDLE_ERROR(hipMemcpy(str_coordinate, tmp_coordinate, coord_num*sizeof(int), hipMemcpyDeviceToHost ));
	bool error_hash_table = false;
	bool error_coordinate = false;
	for(int i = 0; i<INDEX_NUM; i++){
		if(hash_table[i] != str_hash_table[i]) error_hash_table = true;
	}
	for(int i = 0; i<coord_num; i++){
		if(coordinate[i] != str_coordinate[i]) error_coordinate = true;
	}
	if (error_hash_table == true)   fprintf(stdout, "Hash table ERROR!! \n");
	else			    	fprintf(stdout, "Hash table PASSED! \n");
	if (error_coordinate == true)   fprintf(stdout, " Coordinate ERROR!! \n");
	else			    	fprintf(stdout, " Coordinate PASSED! \n");

	HANDLE_ERROR( hipFree( tmp_hash_table ) );
	HANDLE_ERROR( hipFree( tmp_coordinate ) );
	freeHash();
	return 0;
}

