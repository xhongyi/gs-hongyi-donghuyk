#include "hip/hip_runtime.h"
/*
 * hash_transfer.cpp
 *
 *  Created on: Oct 8, 2011
 *      Author: mac
 */
#include <cstdlib>
#include <sstream>
#include <iostream>
#include <assert.h>
#include "common.h"
#include "hash_transfer.h"

int* hash_table;
int* coordinate;
string ref_string;
int* dev_coordinate;
int* dev_hash_table;
char* dev_ref_string;

void HashFree(int ** hash_table, int ** coordinate);

void getHashTablePtr(int ** ptr) {
	*ptr = hash_table;
}

void getCoordinatePtr(int ** ptr) {
	*ptr = coordinate;
}

long long loadHash(string hash_name) {
	long long coord_num = hashReconstructor(&hash_table, &coordinate, hash_name.c_str());
	hipMalloc((void**)&dev_hash_table, INDEX_NUM*sizeof(int));
	hipMalloc((void**)&dev_coordinate, coord_num*sizeof(int));
	hipMemcpy(dev_coordinate, coordinate, coord_num*sizeof(int), hipMemcpyHostToDevice );
	hipMemcpy(dev_hash_table, hash_table, INDEX_NUM*sizeof(int), hipMemcpyHostToDevice );
	return coord_num;
}

int loadRef(string ref_name) {
	char tmp_ref_name[50];
	strcpy(tmp_ref_name, ref_name.c_str());
	refLoader(ref_string, tmp_ref_name);
	char *tmp_ref_string = (char*)malloc( ref_string.size() * sizeof(char));
	strcpy(tmp_ref_string, ref_string.c_str());
	hipMalloc((void**)&dev_ref_string, ref_string.size() * sizeof(char));
	hipMemcpy(dev_ref_string, tmp_ref_string, ref_string.size() * sizeof(char), hipMemcpyHostToDevice );
	free(tmp_ref_string);
	return ref_string.size();
}

void freeHash(void) {
	free(hash_table);
	free(coordinate);
	hipFree (dev_hash_table);
	hipFree (dev_coordinate);
}

void freeRef(void) {
	hipFree (dev_ref_string);
}

//void HashFree(int ** index_db, int ** coordinate_db){
//	free(*index_db);
//	free(*coordinate_db);
//}

/*
__global__ void loadHashTest(int *tmp_coordinate, int *tmp_hash_table, long long coord_num, int* dev_hash_table, int* dev_coordinate) {
	for(int i = 0; i < INDEX_NUM; i++) {
		tmp_hash_table[i] = dev_hash_table[i];
	}
	for(int i = 0; i < coord_num; i++) {
		tmp_coordinate[i] = dev_coordinate[i];
	}
}

__global__ void loadRefTest(char *tmp_ref_string, long long ref_num, char *dev_ref_string) {
	for(int i = 0; i < ref_num; i++) {
		tmp_ref_string[i] = dev_ref_string[i];
	}
}

int ref_test(void) {
	char* tmp_ref_string;
	long long ref_num = loadRef("toy_ref_0");
	fprintf(stdout, "reference number: %lli  \n", ref_num);
	char* str_ref_string = (char*)malloc(sizeof(char)*ref_num);
	hipMalloc((void**)&tmp_ref_string, ref_num*sizeof(char));
	loadRefTest <<<1, 1>>> (tmp_ref_string, ref_num, dev_ref_string);
	hipMemcpy(str_ref_string, tmp_ref_string, ref_num*sizeof(char), hipMemcpyDeviceToHost );
	bool error_ref_string = false;
	for(int i = 0; i<ref_num; i++){
		if(ref_string[i] != str_ref_string[i])
			error_ref_string = true;
	}
	if (error_ref_string == true)
		fprintf(stdout, "Ref String ERROR!! \n");
	else
		fprintf(stdout, "Ref String PASSED! \n");

	hipFree(tmp_ref_string);
	freeRef();
	return 0;
}

int hash_test(void) {
	int* tmp_coordinate;
	int* tmp_hash_table;
	long long coord_num = loadHash("toy_hash_0");
	fprintf(stdout, "coordinate number: %lli  \n", coord_num);
	int* str_hash_table = (int*)malloc(sizeof(int)*INDEX_NUM);
	int* str_coordinate = (int*)malloc(sizeof(int)*coord_num);
	hipMalloc((void**)&tmp_hash_table, INDEX_NUM*sizeof(int));
	hipMalloc((void**)&tmp_coordinate, coord_num*sizeof(int));
	loadHashTest <<<1, 1>>> (tmp_coordinate, tmp_hash_table, coord_num, dev_hash_table, dev_coordinate);
	hipMemcpy(str_hash_table, tmp_hash_table, INDEX_NUM*sizeof(int), hipMemcpyDeviceToHost );
	hipMemcpy(str_coordinate, tmp_coordinate, coord_num*sizeof(int), hipMemcpyDeviceToHost );
	bool error_hash_table = false;
	bool error_coordinate = false;
	for(int i = 0; i<INDEX_NUM; i++){
		if(hash_table[i] != str_hash_table[i]) {
			error_hash_table = true;
			cout << "i:" << i << " ref:" << hash_table[i] << " str:" << str_hash_table[i] << endl;
			exit(1);
		}
	}
	for(int i = 0; i<coord_num; i++){
		if(coordinate[i] != str_coordinate[i]) error_coordinate = true;
	}
	if (error_hash_table == true)
		fprintf(stdout, "Hash table ERROR!! \n");
	else
		fprintf(stdout, "Hash table PASSED! \n");
	if (error_coordinate == true)
		fprintf(stdout, "Coordinate ERROR!! \n");
	else
		fprintf(stdout, "Coordinate PASSED! \n");

	hipFree( tmp_hash_table );
	hipFree( tmp_coordinate );
	freeHash();
	return 0;
}
*/
