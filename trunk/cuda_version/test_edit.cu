#include "hip/hip_runtime.h"
/*
 * test_search.cpp
 *
 *  Created on: Oct 19, 2011
 *	  Author: mac
 */
#include <iostream>
#include "universal_header.h"

using namespace std;


#include "hash_test.cu"


__global__ void cuda_editDistanceCal (char * dev_test_read, char * dev_ref_read, ED_result * result) {
	ED_path path[20];
	/*
	 * The trick is in the third agrument. If I pick the forward and backward breaking position at the same position,
	 * it going all correct.
	 * However, if I changed it like some starts at key 0, some key 1, some key 2 like what I did here,
	 * the warp will break down. Actually, it will break down in a way that the key 0 ones will go into 1 warp
	 * key 1 ones will go into 1 warp and key 2 ones will go into 1 warp.
	 * Feel free to tweak the %number.
	 */
	result[threadIdx.x] = editDistanceCal(dev_test_read, dev_ref_read, 2, path, 6, 5, 5);
}

int test_cuda(void) {

	char test_read[READ_LENGTH];
	strcpy(
			test_read,
//			"GGGTGGTAGGTGCAGAGACGGGAGGGGCAGAGCCGCAGGCACAGCCAAGAGGGCTGAAGAAATGGTAGAACGGAGCAGCTGGTGATGTGTGGGCCCACCGGCCCCAGG");
			"CACGTTTTATTTTATTTTTTGAGATGGAGTCTCACTCTGTCACCCAGGCTGGAGTGTGGTGGCGCAATCTTTGCTCACTGCAACCTCCGCCTCCCGGGTTCAAGCGAT");

	//Getting the sort key.

	char ref_read[READ_LENGTH];
	strcpy(
			ref_read,
			//"GGGTGGTAGGTGCAGAGACGGGAGGGGCAGAGCCGCAGGCACAGCCAAGAGGGCTGAAGAAATGGTAGAACGGAGCAGCTGGTGATGTGTGGGCCCACCGGCCCCAGG");
			"TTACTATATATATATTTTTTGAGATAGGGTCTCACTCTGTCACCCAGGCTGGAGTGCCCTGACATGATCTTGGCTCACTGCAACCTCCACCTCCCGGGTTCAAGCGAT");

	ED_result result[1];

	char * dev_test_read;
	char * dev_ref_read;

	ED_result * dev_result;

	hipMalloc( (void**) &dev_test_read, READ_LENGTH * sizeof(char) );
	hipMalloc( (void**) &dev_ref_read, READ_LENGTH * sizeof(char) );

	hipMemcpy(dev_test_read, test_read, READ_LENGTH * sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(dev_ref_read, ref_read, READ_LENGTH * sizeof(char), hipMemcpyHostToDevice);

	hipMalloc( (void**) &dev_result, 1 * sizeof(ED_result) );

	cuda_editDistanceCal <<<1, 1>>> (dev_test_read, dev_ref_read, dev_result);

	hipMemcpy(result, dev_result, 1 * sizeof(ED_result), hipMemcpyDeviceToHost);

	for (int i = 0; i < 1; i++) {
		cout << "i: " << i << endl;
		if (result[i].correct) {
			cout << "correct! " << endl;
			cout << "result[" << i << "].diff_num: " << result[i].diff_num << endl;
			for (int j = 0; j < result[i].diff_num; i++) {
			 	switch (result[i].error[j].diff) {
			 	case MISMATCH:
			 		cout << "Mismatch at " << result[i].error[j].location << endl;
					cout << "Error: " << result[i].error[j].diff_char << endl;
			 		break;
				case INSERTION:
			 		cout << "Insertion at " << result[i].error[j].location << endl;
			 		cout << "Error: " << result[i].error[j].diff_char << endl;
			 		break;
			 	case DELETION:
			 		cout << "Deletion at " << result[i].error[j].location << endl;
			 		cout << "Error: " << result[i].error[j].diff_char << endl;
			 		break;
			 	}
			}
		}
		else
			cout << "Incorrect!!!" << endl;
	}

	hipFree(dev_test_read);
	hipFree(dev_ref_read);
	hipFree(dev_result);
	return 0;
}

int main() {
	test_cuda();
	return 0;
}
