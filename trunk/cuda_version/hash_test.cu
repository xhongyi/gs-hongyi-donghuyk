#include "hip/hip_runtime.h"
/*
 * test_search.cpp
 *
 *  Created on: Oct 19, 2011
 *	  Author: mac
 */
#include <iostream>
#include "universal_header.h"

using namespace std;

__global__ void loadHashTest(int *tmp_coordinate, int *tmp_hash_table, long long coord_num, int* dev_hash_table, int* dev_coordinate) {
	for(int i = 0; i < INDEX_NUM; i++) {
		tmp_hash_table[i] = dev_hash_table[i];
	}
	for(int i = 0; i < coord_num; i++) {
		tmp_coordinate[i] = dev_coordinate[i];
	}
}

__global__ void loadRefTest(char *tmp_ref_string, long long ref_num, char *dev_ref_string) {
	for(int i = 0; i < ref_num; i++) {
		tmp_ref_string[i] = dev_ref_string[i];
	}
}

int ref_test(int ref_num) {
	char* tmp_ref_string;
//	long long ref_num = loadRef("toy_ref_0");
	fprintf(stdout, "reference number: %i  \n", ref_num);
	char* str_ref_string = (char*)malloc(sizeof(char)*ref_num);
	hipMalloc((void**)&tmp_ref_string, ref_num*sizeof(char));
	loadRefTest <<<1, 1>>> (tmp_ref_string, ref_num, dev_ref_string);
	hipMemcpy(str_ref_string, tmp_ref_string, ref_num*sizeof(char), hipMemcpyDeviceToHost );
	bool error_ref_string = false;
	for(int i = 0; i<ref_num; i++){
		if(ref_string[i] != str_ref_string[i])
			error_ref_string = true;
	}
	if (error_ref_string == true)
		fprintf(stdout, "Ref String ERROR!! \n");
	else
		fprintf(stdout, "Ref String PASSED! \n");

	free(str_ref_string);
	//hipFree(tmp_ref_string);
	//freeRef();
	return 0;
}

int hash_test(int coord_num) {
	int* tmp_coordinate;
	int* tmp_hash_table;
//	long long coord_num = loadHash("toy_hash_0");
	fprintf(stdout, "coordinate number: %i  \n", coord_num);
	int* str_hash_table = (int*)malloc(sizeof(int)*INDEX_NUM);
	int* str_coordinate = (int*)malloc(sizeof(int)*coord_num);
	hipMalloc((void**)&tmp_hash_table, INDEX_NUM*sizeof(int));
	hipMalloc((void**)&tmp_coordinate, coord_num*sizeof(int));
	loadHashTest <<<1, 1>>> (tmp_coordinate, tmp_hash_table, coord_num, dev_hash_table, dev_coordinate);
	hipMemcpy(str_hash_table, tmp_hash_table, INDEX_NUM*sizeof(int), hipMemcpyDeviceToHost );
	hipMemcpy(str_coordinate, tmp_coordinate, coord_num*sizeof(int), hipMemcpyDeviceToHost );
	bool error_hash_table = false;
	bool error_coordinate = false;
	for(int i = 0; i<INDEX_NUM; i++){
		if(hash_table[i] != str_hash_table[i]) {
			error_hash_table = true;
			cout << "i:" << i << " ref:" << hash_table[i] << " str:" << str_hash_table[i] << endl;
			exit(1);
		}
	}
	for(int i = 0; i<coord_num; i++){
		if(coordinate[i] != str_coordinate[i]) error_coordinate = true;
	}
	if (error_hash_table == true)
		fprintf(stdout, "Hash table ERROR!! \n");
	else
		fprintf(stdout, "Hash table PASSED! \n");
	if (error_coordinate == true)
		fprintf(stdout, "Coordinate ERROR!! \n");
	else
		fprintf(stdout, "Coordinate PASSED! \n");
	free(str_hash_table);
	free(str_coordinate);
	hipFree( tmp_hash_table );
	hipFree( tmp_coordinate );
	//freeHash();
	return 0;
}
