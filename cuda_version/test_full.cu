//#define DEBUG
#include <iostream>
#include <iostream>
#include <deque>
#include <list>
#include <map>
#include <cstdio>
#include <fstream>
#include <string>
#include <time.h>
#include "universal_header.h"

#define FRAGMENT_LENGTH 108

using namespace std;

void test_full(string hash_file_name, string ref_file_name, string output_file_name, string result_input_name) {
	set_max_indel_num(3);
	set_max_diff_num(3);
	ifstream ref_file;
	ifstream input_file;
	ofstream store_file;

//	final_result filter_result;
	int monitor_counter = 0; 		// for operation monitoring
	long long monitor_counter2 = 0; // for operation monitoring
	long long accumulate_time = 0;

	int fragment_set  = MAX_FRAGMENT_SET_NUM;
	int fragment_size = MAX_FRAGMENT_SIZE;
	int thread_size   = MAX_THREAD_NUM;
	GPU_fragment test_fragment[fragment_size];
	GPU_fragment* dev_fragment;
	final_result* dev_result;
	final_result *test_result = (final_result*) malloc(sizeof(final_result)*fragment_size);
	hipMalloc((void**) &dev_fragment, sizeof(GPU_fragment)*fragment_size);
	hipMalloc((void**) &dev_result,   sizeof(final_result)*fragment_size);

	// get fragment from reference file

	for (int j = 0 ; j < 1 ; j++) {
	//for (int j = 0 ; j < MAX_CONTIG_FILE ; j++) {
		map<int, int> binary_search;
		map<int, int> distribution;
		map<int, int> correct_count;
		char * file_ref	= (char*) malloc(sizeof(char)*50);
		char * file_hash= (char*) malloc(sizeof(char)*50);
		char * file_store= (char*) malloc(sizeof(char)*50);
	   		sprintf(file_ref ,  "%s%i", (char*) ref_file_name.c_str(), j);
	   		sprintf(file_hash,  "%s%i", (char*) hash_file_name.c_str(), j);
	   		sprintf(file_store, "%s%i_%i_%i_%i", (char*) output_file_name.c_str(), j, 
				fragment_set, fragment_size, thread_size);
		cout << "*** ref_file_name  :" << file_ref  << endl;
		cout << "*** hash_file_name :" << file_hash << endl;
		cout << "*** store_file_name:" << file_store << endl;
		// store file 
		store_file.open(file_store);

		// reference file load at string
		string ref;
		cout << "*** Status : Start ref strings" << endl;
		int ref_num = loadRef(file_ref);
		cout << "*** Status : End   ref strings" << endl;

		// hash table load 
		cout << "*** Status : Start load hash table" << endl;
		long long coord_num = loadHash(file_hash);
		cout << "*** Status : End   load hash table" << endl;

		int total_spilled_num = 0;

		ref_file.open(file_ref);
		if (!ref_file.is_open()) {
			cout << " Error File Open : " << file_ref << endl;
			break;
		}
		time_t start_time;
		time(&start_time);
		input_file.open(result_input_name.c_str());
		do {
			// input fragment fetch from result_input
			for (int i = 0 ; i < fragment_size ; i ++ ){
				input_file >> test_fragment[i].fragment;
				if(!input_file.good()){
					fragment_size = i;
					break;
				}
			}
			// Getting the sort key.
			for (int k = 0; k < fragment_size; k++) {
				key_struct sort_input[KEY_NUMBER];
				for (int i = 0; i < KEY_NUMBER; i++) {
					char key[KEY_LENGTH];
					for (int j = 0; j < KEY_LENGTH; j++) {
						key[j] = test_fragment[k].fragment[j + KEY_LENGTH * i];
					}
					int key_hash = hashVal(key);
					int key_entry = hash_table[key_hash];
					int key_entry_size = coordinate[key_entry];
					sort_input[i].order = 0;
					sort_input[i].key_number = i;
					sort_input[i].key_entry = key_entry;
					sort_input[i].key_entry_size = key_entry_size;
				}
				sortPrefilter(test_fragment[k].sorted_keys, sort_input);
//				for (int i = 0; i < KEY_NUMBER; i++) {
//					cout << " fragment_set: "   << k
//						 << " key_number:"	  << test_fragment[k].sorted_keys[i].key_number
//						 << " key_entry: "	  << test_fragment[k].sorted_keys[i].key_entry
//						 << " key_entry_size: " << test_fragment[k].sorted_keys[i].key_entry_size
//						 << " base: "		   << test_fragment[k].sorted_keys[i].base << endl;
//				}
//				cout << "****************************************************************************" << endl;
			}
			hipMemcpy(dev_fragment, &test_fragment, sizeof(GPU_fragment)*fragment_size, hipMemcpyHostToDevice);
			searchFragment <<<fragment_set, thread_size>>> (dev_fragment, fragment_size, 
						dev_ref_string, dev_hash_table, dev_coordinate, 3, 3, dev_result);
			hipMemcpy(test_result, dev_result, sizeof(final_result)*fragment_size, hipMemcpyDeviceToHost);
			for (int j = 0; j < fragment_size; j++) {
//				cout << "****************************************************************************" << endl;
//				cout << "Spilled	  : " << test_result[j].spilled << endl;
//				cout << "Result Size  : " << test_result[j].size << endl;
//				cout << "****************************************************************************" << endl;
//				cout << "Fragment	 : " << test_result[j].fragment << endl;
	
//				cout << "Fragment	 : ";
//				for (int i = 0; i < READ_LENGTH; i++){
//					cout << test_result[j].fragment[i];
//				}
//				cout << endl << "* Size  : " << test_result[j].size << endl;
				int tmp_size = test_result[j].size;
				if(tmp_size > 100) {
					tmp_size = 100;
				}
				if(test_result[j].spilled == true) {
					total_spilled_num = total_spilled_num + 1;
				}
//				for (int i = 0; i < tmp_size; i++) {
//					cout << "* Result coor  : " << test_result[j].coor_results[i].coordiante;
//					cout << "   Result diff  : " << test_result[j].coor_results[i].diff_num << endl;
//				}
//				cout << "****************************************************************************" << endl;
	            correct_count[tmp_size]++;
			}
			monitor_counter = monitor_counter + fragment_size;
			monitor_counter2 = monitor_counter2 + fragment_size;
			if (monitor_counter >= 10000) {
				fprintf(stdout, "hash distribution count: %lld \n", monitor_counter2);
					monitor_counter = 0;
				}
			if (monitor_counter2 >= 1000000) {
					monitor_counter2 = 0;
			}
		}while(input_file.good());
		fragment_size = MAX_FRAGMENT_SIZE;
		hipFree(dev_fragment);
		hipFree(dev_result);
		freeHash();
		freeRef();
	
		time_t end_time;
		time(&end_time);
		cout << endl;
		accumulate_time = accumulate_time + difftime(end_time, start_time);
		ref_file.close();
		input_file.close();
	
		long long total_fragment_num2 = 0;
		long long total_pass_num = 0;
		store_file << endl << "Number of Edit-distance Passed" << endl;
		for (map<int, int>::iterator p = correct_count.begin(); p != correct_count.end(); p++) {
			store_file << "index :" << p->first << "	num :" << p->second << endl;
			total_fragment_num2 = total_fragment_num2 + p->second;
			total_pass_num = total_pass_num + p->first * p->second;
		}
		store_file << endl;
		store_file << "---------------------------------------------" << endl;
		store_file << "total_fragment_num : " << total_fragment_num2 << endl;
		store_file << "total_pass_num____ : " << total_pass_num << endl;
		store_file << "total_spilled_num_ : " << total_spilled_num << endl;
		store_file << "Start_time________ : " << ctime(&start_time);
		store_file << "End_time__________ : " << ctime(&end_time);
		store_file << "TIme Diff_________ : " << difftime(end_time,start_time) << endl;
		store_file << "Accumulated Time__ : " << accumulate_time << endl;
		store_file << "---------------------------------------------" << endl;
	
		cout << "---------------------------------------------" << endl;
		cout << "total_fragment_num : " << total_fragment_num2 << endl;
		cout << "total_pass_num____ : " << total_pass_num << endl;
		cout << "total_spilled_num_ : " << total_spilled_num << endl;
		cout << "Start_time________ : " << ctime(&start_time);
		cout << "End_time__________ : " << ctime(&end_time);
		cout << "TIme Diff_________ : " << difftime(end_time,start_time) << endl;
		cout << "Accumulated Time__ : " << accumulate_time << endl;
		cout << "---------------------------------------------" << endl;
		store_file.close();
	}
	cout << "Accumulated Time : " << accumulate_time << endl;
}

int main() {
	char * file_ref		= (char*) malloc(sizeof(char)*50);	
	char * file_input	= (char*) malloc(sizeof(char)*50);	
	char * file_hash	= (char*) malloc(sizeof(char)*50);
	char * file_dist	= (char*) malloc(sizeof(char)*50);
	sprintf(file_ref,  "%s", "result_ref_");
	sprintf(file_hash, "%s", "result_hash_");
	sprintf(file_input, "%s", "result_input");

	fprintf(stdout,"Mode : Cheapest Select\n");
	sprintf(file_dist, "%s", "result_dist_cheap_");
	test_full(file_hash, file_ref, file_dist, file_input);

	free(file_ref);
	free(file_input);
	free(file_hash);
	free(file_dist);
	return 0;
}

