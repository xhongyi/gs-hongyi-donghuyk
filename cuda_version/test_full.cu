//#define DEBUG
#include <iostream>
#include <iostream>
#include <deque>
#include <list>
#include <map>
#include <cstdio>
#include <fstream>
#include <string>
#include <time.h>
#include "universal_header.h"

#define FRAGMENT_LENGTH 108

using namespace std;

void test_full(string hash_file_name, string ref_file_name, string output_file_name, string result_input_name) {
	set_max_indel_num(3);
	set_max_diff_num(3);
	ifstream ref_file;
	ifstream input_file;
	ofstream store_file;

//	final_result filter_result;
	long long monitor_counter = 0; 		// for operation monitoring
	long long monitor_counter2 = 0; // for operation monitoring

	long long prefilter_time = 0;
	long long memcpy_input_time = 0;
	long long cuda_time = 0;
	long long memcpy_output_time = 0;
	long long analysis_time = 0;
	long long total_time = 0;

	long long acc_prefilter_time = 0;
	long long acc_memcpy_input_time = 0;
	long long acc_cuda_time = 0;
	long long acc_memcpy_output_time = 0;
	long long acc_analysis_time = 0;
	long long acc_total_time = 0;

	int fragment_set  = MAX_FRAGMENT_SET_NUM;
	int fragment_size = MAX_FRAGMENT_SIZE;
	int thread_size   = MAX_THREAD_NUM;
	GPU_fragment test_fragment[fragment_size];
	GPU_fragment* dev_fragment;
	final_result* dev_result;
	final_result *test_result = (final_result*) malloc(sizeof(final_result)*fragment_size);
	hipMalloc((void**) &dev_fragment, sizeof(GPU_fragment)*fragment_size);
	hipMalloc((void**) &dev_result,   sizeof(final_result)*fragment_size);

	// get fragment from reference file

	for (int j = 0 ; j < 1 ; j++) {
	//for (int j = 0 ; j < MAX_CONTIG_FILE ; j++) {
		map<int, int> binary_search;
		map<int, int> distribution;
		map<int, int> correct_count;
		char * file_ref	= (char*) malloc(sizeof(char)*50);
		char * file_hash= (char*) malloc(sizeof(char)*50);
		char * file_store= (char*) malloc(sizeof(char)*50);
	   		sprintf(file_ref ,  "%s%i", (char*) ref_file_name.c_str(), j);
	   		sprintf(file_hash,  "%s%i", (char*) hash_file_name.c_str(), j);
	   		sprintf(file_store, "%s%i_%i_%i_%i", (char*) output_file_name.c_str(), j, 
				fragment_set, fragment_size, thread_size);
		cout << "*** ref_file_name  :" << file_ref  << endl;
		cout << "*** hash_file_name :" << file_hash << endl;
		cout << "*** store_file_name:" << file_store << endl;
		// store file 
		store_file.open(file_store);

		// reference file load at string
		string ref;
		cout << "*** Status : Start ref strings" << endl;
		int ref_num = loadRef(file_ref);
		cout << "*** Status : End   ref strings" << endl;

		// hash table load 
		cout << "*** Status : Start load hash table" << endl;
		long long coord_num = loadHash(file_hash);
		cout << "*** Status : End   load hash table" << endl;

		int total_spilled_num = 0;

		ref_file.open(file_ref);
		if (!ref_file.is_open()) {
			cout << " Error File Open : " << file_ref << endl;
			break;
		}
		input_file.open(result_input_name.c_str());
		do {
			time_t start_time;
			time(&start_time);
			// input fragment fetch from result_input
			for (int i = 0 ; i < fragment_size ; i ++ ){
				input_file >> test_fragment[i].fragment;
				if(!input_file.good()){
					fragment_size = i;
					break;
				}
			}
			// Getting the sort key.
			for (int k = 0; k < fragment_size; k++) {
				key_struct sort_input[KEY_NUMBER];
				for (int i = 0; i < KEY_NUMBER; i++) {
					char key[KEY_LENGTH];
					for (int j = 0; j < KEY_LENGTH; j++) {
						key[j] = test_fragment[k].fragment[j + KEY_LENGTH * i];
					}
					int key_hash = hashVal(key);
					int key_entry = hash_table[key_hash];
					int key_entry_size = coordinate[key_entry];
					sort_input[i].order = 0;
					sort_input[i].key_number = i;
					sort_input[i].key_entry = key_entry;
					sort_input[i].key_entry_size = key_entry_size;
				}
				sortPrefilter(test_fragment[k].sorted_keys, sort_input);
//				for (int i = 0; i < KEY_NUMBER; i++) {
//					cout << " fragment_set: "   << k
//						 << " key_number:"	  << test_fragment[k].sorted_keys[i].key_number
//						 << " key_entry: "	  << test_fragment[k].sorted_keys[i].key_entry
//						 << " key_entry_size: " << test_fragment[k].sorted_keys[i].key_entry_size
//						 << " base: "		   << test_fragment[k].sorted_keys[i].base << endl;
//				}
//				cout << "****************************************************************************" << endl;
			}
			time_t start_memcpy_input_time;
			time(&start_memcpy_input_time);
			hipMemcpy(dev_fragment, &test_fragment, sizeof(GPU_fragment)*fragment_size, hipMemcpyHostToDevice);
			time_t start_cuda_time;
			time(&start_cuda_time);
			searchFragment <<<fragment_set, thread_size>>> (dev_fragment, fragment_size, 
						dev_ref_string, dev_hash_table, dev_coordinate, 3, 3, dev_result);
			time_t start_memcpy_output_time;
			time(&start_memcpy_output_time);
			hipMemcpy(test_result, dev_result, sizeof(final_result)*fragment_size, hipMemcpyDeviceToHost);
			time_t start_analysis_time;
			time(&start_analysis_time);
			for (int j = 0; j < fragment_size; j++) {
//				cout << "****************************************************************************" << endl;
//				cout << "Spilled	  : " << test_result[j].spilled << endl;
//				cout << "Result Size  : " << test_result[j].size << endl;
//				cout << "****************************************************************************" << endl;
//				cout << "Fragment	 : " << test_result[j].fragment << endl;
	
//				cout << "Fragment	 : ";
//				for (int i = 0; i < READ_LENGTH; i++){
//					cout << test_result[j].fragment[i];
//				}
//				cout << endl << "* Size  : " << test_result[j].size << endl;
				int tmp_size = test_result[j].size;
				if(tmp_size > 100) {
					tmp_size = 100;
				}
				if(test_result[j].spilled == true) {
					total_spilled_num = total_spilled_num + 1;
				}
//				for (int i = 0; i < tmp_size; i++) {
//					cout << "* Result coor  : " << test_result[j].coor_results[i].coordiante;
//					cout << "   Result diff  : " << test_result[j].coor_results[i].diff_num << endl;
//				}
//				cout << "****************************************************************************" << endl;
	            correct_count[tmp_size]++;
			}
			monitor_counter = monitor_counter + fragment_size;
			monitor_counter2 = monitor_counter2 + fragment_size;
			if (monitor_counter >= 10000) {
				fprintf(stdout, "hash distribution count: %lld \n", monitor_counter2);
					monitor_counter = 0;
				}
			if (monitor_counter2 >= 1000000) {
					monitor_counter2 = 0;
			}
			time_t end_time;
			time(&end_time);
 			prefilter_time 		= prefilter_time + difftime(start_memcpy_input_time, start_time);
			memcpy_input_time 	= memcpy_input_time + difftime(start_cuda_time, start_memcpy_input_time);
 			cuda_time 			= cuda_time + difftime(start_memcpy_output_time, start_cuda_time);
			memcpy_output_time 	= memcpy_output_time + difftime(start_analysis_time, start_memcpy_output_time);
 			analysis_time 		= analysis_time + difftime(end_time, start_analysis_time);
 			total_time 			= total_time + difftime(end_time, start_time);
		}while(input_file.good());
		fragment_size = MAX_FRAGMENT_SIZE;
		hipFree(dev_fragment);
		hipFree(dev_result);
		freeHash();
		freeRef();
	
 		acc_prefilter_time 		= acc_prefilter_time + prefilter_time;
		acc_memcpy_input_time 	= acc_memcpy_input_time + memcpy_input_time;
 		acc_cuda_time 			= acc_cuda_time + cuda_time;
		acc_memcpy_output_time 	= acc_memcpy_output_time + memcpy_output_time;
 		acc_analysis_time 		= acc_analysis_time + analysis_time;
		acc_total_time 			= acc_total_time + total_time;
		ref_file.close();
		input_file.close();
	
		long long total_fragment_num2 = 0;
		long long total_pass_num = 0;
		store_file << endl << "Number of Edit-distance Passed" << endl;
		for (map<int, int>::iterator p = correct_count.begin(); p != correct_count.end(); p++) {
			store_file << "index :" << p->first << "	num :" << p->second << endl;
			total_fragment_num2 = total_fragment_num2 + p->second;
			total_pass_num = total_pass_num + p->first * p->second;
		}
		store_file << endl;
		store_file << "---------------------------------------------" << endl;
		store_file << "total_fragment_num : " << total_fragment_num2 << endl;
		store_file << "total_pass_num____ : " << total_pass_num << endl;
		store_file << "total_spilled_num_ : " << total_spilled_num << endl;
		store_file << "Acc_prefilter_time : " << acc_prefilter_time << endl;
		store_file << "Acc_memcpy_input__ : " << acc_memcpy_input_time << endl;
		store_file << "Acc_cuda__________ : " << acc_cuda_time << endl;
		store_file << "Acc_memcpy_output_ : " << acc_memcpy_output_time << endl;
		store_file << "Acc_analysis_time_ : " << acc_analysis_time << endl;
		store_file << "Accumulated Time__ : " << acc_total_time << endl;
		store_file << "---------------------------------------------" << endl;
	
		cout << "---------------------------------------------" << endl;
		cout << "total_fragment_num : " << total_fragment_num2 << endl;
		cout << "total_pass_num____ : " << total_pass_num << endl;
		cout << "total_spilled_num_ : " << total_spilled_num << endl;
		cout << "Acc_prefilter_time : " << acc_prefilter_time << endl;
		cout << "Acc_memcpy_input__ : " << acc_memcpy_input_time << endl;
		cout << "Acc_cuda__________ : " << acc_cuda_time << endl;
		cout << "Acc_memcpy_output_ : " << acc_memcpy_output_time << endl;
		cout << "Acc_analysis_time_ : " << acc_analysis_time << endl;
		cout << "Accumulated Time__ : " << acc_total_time << endl;
		cout << "---------------------------------------------" << endl;
		store_file.close();
	}
	cout << "Accumulated Time : " << acc_total_time << endl;
}

int main() {
	char * file_ref		= (char*) malloc(sizeof(char)*50);	
	char * file_input	= (char*) malloc(sizeof(char)*50);	
	char * file_hash	= (char*) malloc(sizeof(char)*50);
	char * file_dist	= (char*) malloc(sizeof(char)*50);
	sprintf(file_ref,  "%s", "result_ref_");
	sprintf(file_hash, "%s", "result_hash_");
	sprintf(file_input, "%s", "result_input");

	fprintf(stdout,"Mode : Cheapest Select\n");
	sprintf(file_dist, "%s", "result_dist_cheap_");
	test_full(file_hash, file_ref, file_dist, file_input);

	free(file_ref);
	free(file_input);
	free(file_hash);
	free(file_dist);
	return 0;
}

			time_t start_time;
			time_t start_memcpy_input_time;
			time_t start_cuda_time;
			time_t start_memcpy_output_time;
			time_t start_analysis_time;
			time_t end_time;

