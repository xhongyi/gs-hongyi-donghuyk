//#define DEBUG
#include <iostream>
#include <iostream>
#include <deque>
#include <list>
#include <map>
#include <cstdio>
#include <fstream>
#include <string>
#include <time.h>
#include "universal_header.h"

#define FRAGMENT_LENGTH 108

using namespace std;

void test_full(string hash_file_name, string ref_file_name, string output_file_name, string result_input_name) {
	set_max_indel_num(3);
	set_max_diff_num(3);
	ifstream ref_file;
	ifstream input_file;
//	ofstream store_file;
    FILE * pFileW;

//	final_result filter_result;
	long long monitor_counter = 0; 		// for operation monitoring
	long long monitor_counter2 = 0; // for operation monitoring

	float prefilter_time = 0;
	float memcpy_input_time = 0;
	float cuda_time = 0;
	float memcpy_output_time = 0;
	float analysis_time = 0;
	float total_time = 0;

	float acc_prefilter_time = 0;
	float acc_memcpy_input_time = 0;
	float acc_cuda_time = 0;
	float acc_memcpy_output_time = 0;
	float acc_analysis_time = 0;
	float acc_total_time = 0;

	int fragment_set  = MAX_FRAGMENT_SET_NUM;
	int fragment_size = MAX_FRAGMENT_SIZE;
	int thread_size   = MAX_THREAD_NUM;
	GPU_fragment test_fragment[fragment_size];
	GPU_fragment* dev_fragment;
	final_result* dev_result;
	final_result *test_result = (final_result*) malloc(sizeof(final_result)*fragment_size);
	hipMalloc((void**) &dev_fragment, sizeof(GPU_fragment)*fragment_size);
	hipMalloc((void**) &dev_result,   sizeof(final_result)*fragment_size);

	// get fragment from reference file

	for (int j = 0 ; j < 1 ; j++) {
	//for (int j = 0 ; j < MAX_CONTIG_FILE ; j++) {
		map<int, int> binary_search;
		map<int, int> distribution;
		map<int, int> correct_count;
		char * file_ref	= (char*) malloc(sizeof(char)*50);
		char * file_hash= (char*) malloc(sizeof(char)*50);
		char * file_store= (char*) malloc(sizeof(char)*50);
	   		sprintf(file_ref ,  "%s%i", (char*) ref_file_name.c_str(), j);
	   		sprintf(file_hash,  "%s%i", (char*) hash_file_name.c_str(), j);
	   		sprintf(file_store, "%s%i_%i_%i_%i", (char*) output_file_name.c_str(), j, 
				fragment_set, fragment_size, thread_size);
		cout << "*** ref_file_name  :" << file_ref  << endl;
		cout << "*** hash_file_name :" << file_hash << endl;
		cout << "*** store_file_name:" << file_store << endl;
		// store file 
//		store_file.open(file_store);
		pFileW = fopen(file_store, "w");

		// reference file load at string
		string ref;
		cout << "*** Status : Start ref strings" << endl;
		int ref_num = loadRef(file_ref);
		cout << "*** Status : End   ref strings" << endl;

		// hash table load 
		cout << "*** Status : Start load hash table" << endl;
		long long coord_num = loadHash(file_hash);
		cout << "*** Status : End   load hash table" << endl;

		int total_spilled_num = 0;

		ref_file.open(file_ref);
		if (!ref_file.is_open()) {
			cout << " Error File Open : " << file_ref << endl;
			break;
		}
		input_file.open(result_input_name.c_str());
		do {
			hipEvent_t start_prefilter_time, stop_prefilter_time;
			hipEventCreate(&start_prefilter_time);
			hipEventRecord(start_prefilter_time, 0);
			// input fragment fetch from result_input
			for (int i = 0 ; i < fragment_size ; i ++ ){
				input_file >> test_fragment[i].fragment;
				if(!input_file.good()){
					fragment_size = i;
					break;
				}
			}
			// Getting the sort key.
			for (int k = 0; k < fragment_size; k++) {
				key_struct sort_input[KEY_NUMBER];
				for (int i = 0; i < KEY_NUMBER; i++) {
					char key[KEY_LENGTH];
					for (int j = 0; j < KEY_LENGTH; j++) {
						key[j] = test_fragment[k].fragment[j + KEY_LENGTH * i];
					}
					int key_hash = hashVal(key);
					int key_entry = hash_table[key_hash];
					int key_entry_size = coordinate[key_entry];
					sort_input[i].order = 0;
					sort_input[i].key_number = i;
					sort_input[i].key_entry = key_entry;
					sort_input[i].key_entry_size = key_entry_size;
				}
				sortPrefilter(test_fragment[k].sorted_keys, sort_input);
//				for (int i = 0; i < KEY_NUMBER; i++) {
//					cout << " fragment_set: "   << k
//						 << " key_number:"	  << test_fragment[k].sorted_keys[i].key_number
//						 << " key_entry: "	  << test_fragment[k].sorted_keys[i].key_entry
//						 << " key_entry_size: " << test_fragment[k].sorted_keys[i].key_entry_size
//						 << " base: "		   << test_fragment[k].sorted_keys[i].base << endl;
//				}
//				cout << "****************************************************************************" << endl;
			}
			hipEventCreate(&stop_prefilter_time);
			hipEventRecord(stop_prefilter_time, 0);
			hipEventSynchronize(stop_prefilter_time);
			float tmp_prefilter_time;
			hipEventElapsedTime(&tmp_prefilter_time, start_prefilter_time, stop_prefilter_time);
 			prefilter_time = prefilter_time + tmp_prefilter_time;
			hipEventDestroy(start_prefilter_time);
			hipEventDestroy(stop_prefilter_time);

			hipEvent_t start_memcpy_input_time, stop_memcpy_input_time;
			hipEventCreate(&start_memcpy_input_time);
			hipEventRecord(start_memcpy_input_time,0);
			hipMemcpy(dev_fragment, &test_fragment, sizeof(GPU_fragment)*fragment_size, hipMemcpyHostToDevice);
			hipEventCreate(&stop_memcpy_input_time);
			hipEventRecord(stop_memcpy_input_time, 0);
			hipEventSynchronize(stop_memcpy_input_time);
			float tmp_memcpy_input_time;
			hipEventElapsedTime(&tmp_memcpy_input_time, start_memcpy_input_time, stop_memcpy_input_time);
			memcpy_input_time = memcpy_input_time + tmp_memcpy_input_time;
			hipEventDestroy(start_memcpy_input_time);
			hipEventDestroy(stop_memcpy_input_time);

			hipEvent_t start_cuda_time, stop_cuda_time;
			hipEventCreate(&start_cuda_time);
			hipEventRecord(start_cuda_time,0);
			searchFragment <<<fragment_set, thread_size>>> (dev_fragment, fragment_size, 
						dev_ref_string, dev_hash_table, dev_coordinate, 3, 3, dev_result);
			hipEventCreate(&stop_cuda_time);
			hipEventRecord(stop_cuda_time, 0);
			hipEventSynchronize(stop_cuda_time);
			float tmp_cuda_time;
			hipEventElapsedTime(&tmp_cuda_time, start_cuda_time, stop_cuda_time);
			cuda_time = cuda_time + tmp_cuda_time;
			hipEventDestroy(start_cuda_time);
			hipEventDestroy(stop_cuda_time);

			hipEvent_t start_memcpy_output_time, stop_memcpy_output_time;
			hipEventCreate(&start_memcpy_output_time);
			hipEventRecord(start_memcpy_output_time,0);
			hipMemcpy(test_result, dev_result, sizeof(final_result)*fragment_size, hipMemcpyDeviceToHost);
			hipEventCreate(&stop_memcpy_output_time);
			hipEventRecord(stop_memcpy_output_time, 0);
			hipEventSynchronize(stop_memcpy_output_time);
			float tmp_memcpy_output_time;
			hipEventElapsedTime(&tmp_memcpy_output_time, start_memcpy_output_time, stop_memcpy_output_time);
			memcpy_output_time = memcpy_output_time + tmp_memcpy_output_time;
			hipEventDestroy(start_memcpy_output_time);
			hipEventDestroy(stop_memcpy_output_time);

			hipEvent_t start_analysis_time, stop_analysis_time;
			hipEventCreate(&start_analysis_time);
			hipEventRecord(start_analysis_time,0);
			for (int j = 0; j < fragment_size; j++) {
//				cout << "****************************************************************************" << endl;
//				cout << "Spilled	  : " << test_result[j].spilled << endl;
//				cout << "Result Size  : " << test_result[j].size << endl;
//				cout << "****************************************************************************" << endl;
//				cout << "Fragment	 : " << test_result[j].fragment << endl;
	
//				cout << "Fragment	 : ";
//				for (int i = 0; i < READ_LENGTH; i++){
//					cout << test_result[j].fragment[i];
//				}
//				cout << endl << "* Size  : " << test_result[j].size << endl;
				int tmp_size = test_result[j].size;
				if(tmp_size > 100) {
					tmp_size = 100;
				}
				if(test_result[j].spilled == true) {
					total_spilled_num = total_spilled_num + 1;
				}
//				for (int i = 0; i < tmp_size; i++) {
//					cout << "* Result coor  : " << test_result[j].coor_results[i].coordiante;
//					cout << "   Result diff  : " << test_result[j].coor_results[i].diff_num << endl;
//				}
//				cout << "****************************************************************************" << endl;
	            correct_count[tmp_size]++;
			}
			monitor_counter = monitor_counter + fragment_size;
			monitor_counter2 = monitor_counter2 + fragment_size;
			if (monitor_counter >= 10000) {
				fprintf(stdout, "hash distribution count: %lld \n", monitor_counter2);
					monitor_counter = 0;
				}
			if (monitor_counter2 >= 1000000) {
					monitor_counter2 = 0;
			}
			hipEventCreate(&stop_analysis_time);
			hipEventRecord(stop_analysis_time, 0);
			hipEventSynchronize(stop_analysis_time);
			float tmp_analysis_time;
			hipEventElapsedTime(&tmp_analysis_time, start_analysis_time, stop_analysis_time);
			analysis_time = analysis_time + tmp_analysis_time;
			hipEventDestroy(start_analysis_time);
			hipEventDestroy(stop_analysis_time);
			



		}while(input_file.good());
		fragment_size = MAX_FRAGMENT_SIZE;
		hipFree(dev_fragment);
		hipFree(dev_result);
		freeHash();
		freeRef();
 		total_time = total_time + prefilter_time + memcpy_input_time + cuda_time 
				+ memcpy_output_time + analysis_time;
	
 		acc_prefilter_time 		= acc_prefilter_time 	+ prefilter_time;
		acc_memcpy_input_time 	= acc_memcpy_input_time + memcpy_input_time;
 		acc_cuda_time 			= acc_cuda_time 		+ cuda_time;
		acc_memcpy_output_time 	= acc_memcpy_output_time + memcpy_output_time;
 		acc_analysis_time 		= acc_analysis_time + analysis_time;
		acc_total_time 			= acc_total_time + total_time;
		ref_file.close();
		input_file.close();
	
		long long total_fragment_num2 = 0;
		long long total_pass_num = 0;
		fprintf(pFileW,"Number of Edit-distance Passed\n");
		for (map<int, int>::iterator p = correct_count.begin(); p != correct_count.end(); p++) {
			fprintf(pFileW,"index : %i	num : %i\n", p->first, p->second);
			total_fragment_num2 = total_fragment_num2 + p->second;
			total_pass_num = total_pass_num + p->first * p->second;
		}
		fprintf(pFileW,"\n");
	
		fprintf(pFileW,"---------------------------------------------\n");
		fprintf(pFileW,"total_fragment_num : %lliea\n", total_fragment_num2);
		fprintf(pFileW,"total_pass_num____ : %lliea\n", total_pass_num);
		fprintf(pFileW,"total_spilled_num_ : %iea\n", total_spilled_num);
		fprintf(pFileW,"---------------------------------------------\n");
		fprintf(pFileW,"Acc_prefilter_time : %3.1fms\n", acc_prefilter_time);
		fprintf(pFileW,"Acc_memcpy_input__ : %3.1fms\n", acc_memcpy_input_time);
		fprintf(pFileW,"Acc_cuda__________ : %3.1fms\n", acc_cuda_time);
		fprintf(pFileW,"Acc_memcpy_output_ : %3.1fms\n", acc_memcpy_output_time);
		fprintf(pFileW,"Acc_analysis_time_ : %3.1fms\n", acc_analysis_time);
		fprintf(pFileW,"Accumulated Time__ : %3.1fms\n", acc_total_time);
		fprintf(pFileW,"---------------------------------------------\n");

		printf("---------------------------------------------\n");
		printf("total_fragment_num : %lliea\n", total_fragment_num2);
		printf("total_pass_num____ : %lliea\n", total_pass_num);
		printf("total_spilled_num_ : %iea\n", total_spilled_num);
		printf("---------------------------------------------\n");
		printf("Acc_prefilter_time : %3.1fms\n", acc_prefilter_time);
		printf("Acc_memcpy_input__ : %3.1fms\n", acc_memcpy_input_time);
		printf("Acc_cuda__________ : %3.1fms\n", acc_cuda_time);
		printf("Acc_memcpy_output_ : %3.1fms\n", acc_memcpy_output_time);
		printf("Acc_analysis_time_ : %3.1fms\n", acc_analysis_time);
		printf("Accumulated Time__ : %3.1fms\n", acc_total_time);
		printf("---------------------------------------------\n");

		fclose(pFileW);
	}
	cout << "Accumulated Time : " << acc_total_time << endl;
}

int main() {
	char * file_ref		= (char*) malloc(sizeof(char)*50);	
	char * file_input	= (char*) malloc(sizeof(char)*50);	
	char * file_hash	= (char*) malloc(sizeof(char)*50);
	char * file_dist	= (char*) malloc(sizeof(char)*50);
	sprintf(file_ref,  "%s", "result_ref_");
	sprintf(file_hash, "%s", "result_hash_");
	sprintf(file_input, "%s", "result_input");

	fprintf(stdout,"Mode : Cheapest Select\n");
	sprintf(file_dist, "%s", "result_dist_cheap_");
	test_full(file_hash, file_ref, file_dist, file_input);

	free(file_ref);
	free(file_input);
	free(file_hash);
	free(file_dist);
	return 0;
}
