#include "hip/hip_runtime.h"
/*
 * edit_distance.cpp
 *
 *  Created on: Oct 29, 2011
 *      Author: mac
 */

/*
 *
 */
#define		_MAIN_PATH_		0
#define		_DELETE_PATH_	1
#define		_INSERT_PATH_	2
#define		_UN_FILLED_		1000

#include "edit_distance.h"
#include <algorithm>
#include <cstdlib>
//#include <cstring>
#include <iostream>
#include <assert.h>
/*
 *          reference string
 * 		  a p p l i c a t i o n
 *      0 1 2
 * t  a 1 0 1 2
 * e  p 2 1 0 1 2    <----delete_path
 * s  p   2 1 0 1 2
 * t  p     2 1 1 2 3
 *    l       2 1 2 3
 * s  o     ^   2 2 3 <------main_path
 * t  c     |     3 2
 * i  a     |         2
 * n  t     |           2
 * g  i     |             2
 *    o     |               2
 *	        |
 *	    insert_path
 *
 *
 *          reference string
 *        a p p l i c a t i o n
 *      0 1 2 3
 * t  a 1 0 1 2 3
 * e  p 2 1 0 1 2 3  <----bondary_path
 * s  p 3 2 1 0 1 2 3
 * t  p   3 2 1 1 2 3 4
 *    l     3 2 1 2 3 4 5
 * s  o     ^ 3 2 2 3 4 5 <----delete_path
 * t  c     |   3 3 2 3 4
 * i  a     |       3 2 3 4
 * n  i     |         3 3 3 4
 * g  o     |             4 3 4
 *    n     |             ^ 4 3 <------main_path
 *	        |             |
 *	   bondary_path  insert_path
 *      |
 *      | | | 3 3 3 3 4 5
 *      | | 2 2 2 2 3 4 5
 *      | 1 1 1 1 2 3 4 4 4 4 4
 *      0 0 0 0 1 2 3 3 3 3 3 3
 *      1 1 1 1 1 2 2 2 3 4 4 |
 *      2 2 2 2 2 3 3 3   | | |
 *      3 3 3 3 3       | | | |
 *
 */
__device__ void initializePath(ED_path* path, int main_lane, int max_indel_num);

__device__ ED_result editDistanceCalFWD(char* test_read, char* ref_read,
		int key_num, ED_path* path, int main_lane, int max_indel_num,
		int max_diff_num);

__device__ ED_result editDistanceCalBWD(char* test_read, char* ref_read,
		int key_num, ED_path* path, int main_lane, int max_indel_num,
		int max_diff_num);

__device__ void initializeFWDFront(int key_num, ED_path* path, int main_lane,
		int max_indel_num);

__device__ void initializeBWDFront(int key_num, ED_path* path, int main_lane,
		int max_indel_num);

// initializePath only fills the path elements now.
__device__ void initializePath(ED_path* path, int main_lane, int max_indel_num) {
	for (int i = 0; i < max_indel_num * 2 + 3; i++) {
		for (int j = 0; j <= READ_LENGTH; j++) {
			path[i].path_cost[j] = _UN_FILLED_;
		}
	}
}

__device__ void initializeFWDFront(int key_num, ED_path* path, int main_lane,
		int max_indel_num) {
	for (int i = 0; i < max_indel_num * 2 + 3; i++)
		//For the insertion lanes, the front point is shift right.
		path[i].front_idx = (i < main_lane) ? key_num * KEY_LENGTH + main_lane
				- i : key_num * KEY_LENGTH;
}

__device__ void initializeBWDFront(int key_num, ED_path* path, int main_lane,
		int max_indel_num) {
	for (int i = 0; i < max_indel_num * 2 + 3; i++)
		//For the insertion lanes, the front point is shift right.
		path[i].front_idx = (i > main_lane) ? key_num * KEY_LENGTH + main_lane
				- i : key_num * KEY_LENGTH;
}

__device__ ED_result editDistanceCal(char* test_read, char* ref_read,
		int key_num, ED_path* path, int main_lane, int max_indel_num,
		int max_diff_num) {

	DEBUG_PRINT1("Inside ED test 1\n");

	//Initialize path
	ED_result result;
	ED_result FWD_result, BWD_result;
	initializePath(path, main_lane, max_indel_num);
	/*----------------------------------------------------------------------
	for (int j = 0; j <= READ_LENGTH; j++) {
		if (j == (threadIdx.x % 4) + threadIdx.x * 10) 
			path[main_lane].path_cost[j] = threadIdx.x;
		else
			path[main_lane].path_cost[j] = _UN_FILLED_;
	}
	----------------------------------------------------------------------*/
	DEBUG_PRINT2("FWDeditDistanceCal started  threadId: %i\n", threadIdx.x); //Idealy, We should first see all threads printing a since they are at the same warp
/*	
	for (int i = 0; i < 2 * threadIdx.x; i++) {
		printf ("i: %i, threadID: %i\n", i, threadIdx.x);
		int product = 1;
		if (threadIdx.x % 2 == 1) {
			for (int j = 1; j < threadIdx.x; j++) {
				product *= j;
			}
		}
		printf("product: %i, threadID: %i\n", product, threadIdx.x);
	}

	printf("should sync! threadId: %i\n", threadIdx.x);

	for (int i = 0; i < 2 * threadIdx.x; i++) {
		printf ("i: %i, threadID: %i\n", i, threadIdx.x);
		int product = 1;
		if (threadIdx.x % 2 == 1) {
			for (int j = 1; j < threadIdx.x; j++) {
				product *= j;
			}
		}
		printf("product: %i, threadID: %i\n", product, threadIdx.x);
	}
*/

	int cur_lane = main_lane;

	printf("going into while loop now!!  threadId: %i\n", threadIdx.x);
	
	initializeFWDFront(key_num, path, main_lane, max_indel_num);
	
	path[cur_lane].path_cost[path[cur_lane].front_idx] = 0;
	int cur_dist = 0;
	
	while (true) {

		//First Pick a path.
		while (path[cur_lane].path_cost[path[cur_lane].front_idx] != cur_dist) {
			if (cur_lane == max_indel_num * 2 + 1) { //Next is boundary
				if (cur_dist >= max_diff_num) { //Check if we exceeds the max diff tolarence
					//TODO: BREAK AND END.
					result.correct = false;
					break;
				} else {
					//Go to next tier
					cur_lane = 1;
					cur_dist++;
				}
			} else {
				cur_lane++;
			}
		}

		if (path[cur_lane].front_idx >= READ_LENGTH - KEY_LENGTH * key_num) {
			if (path[cur_lane].front_idx == READ_LENGTH) {
				printf("Slide ending!!  threadId: %i\n", threadIdx.x);
				result.correct = true;
				break;
			}
		}
		printf ("path[cur_lane].front_idx: %i, threadID: %i\n", path[cur_lane].front_idx, threadIdx.x);
		path[cur_lane].front_idx++;
	}
	
	if (result.correct == false)
		return result;

	printf("should sync! threadId: %i\n", threadIdx.x);


	initializeFWDFront(key_num, path, cur_lane, max_indel_num);
	while (true) {
		if (path[cur_lane].front_idx >= READ_LENGTH - KEY_LENGTH * key_num) {
			if (path[cur_lane].front_idx == READ_LENGTH) {
				printf("Slide ending!!  threadId: %i\n", threadIdx.x);
				result.correct = true;
				break;
			}
		}
		printf ("path[cur_lane].front_idx: %i, threadID: %i\n", path[cur_lane].front_idx, threadIdx.x);
		path[cur_lane].front_idx++;
	}

	FWD_result = editDistanceCalFWD(test_read, ref_read, key_num, path,
			main_lane, max_indel_num, max_diff_num);


	__syncthreads(); //Force all threads synchronize.

	printf("BWDeditDistanceCal started  threadId: %i\n", threadIdx.x); //We should see all threads print b. However, it is not doing that.
	//BWD_result = editDistanceCalBWD(test_read, ref_read, key_num, path,
	//		main_lane, max_indel_num, max_diff_num);
	DEBUG_PRINT2("2 Directions have finished  threadId: %i\n", threadIdx.x);

	result.diff_num = FWD_result.diff_num;// + BWD_result.diff_num;

	//This is the result index used to probe the result error queue.
	int error_idx = 0;

	if (FWD_result.correct /*&& BWD_result.correct*/ && result.diff_num
			<= max_diff_num) {

		result.correct = true;
/*
		for (int i = 0; i < BWD_result.diff_num; i++) {
			result.error[error_idx] = BWD_result.error[i];
			error_idx++;
		}
*/
		for (int i = FWD_result.diff_num - 1; i >= 0; i--) {
			result.error[error_idx] = FWD_result.error[i];
			error_idx++;
		}
	}
	else
		result.correct = false;

	result.correct = true;
	result.diff_num = 0;
	return result;
}
/*
__device__ ED_result editDistanceCalFWD(char* test_read, char* ref_read,
		int key_num, ED_path* path, int main_lane, int max_indel_num,
		int max_diff_num) {
	//Return result;
	ED_result result;

	//Initialize the Front of each lane
	initializeFWDFront(key_num, path, main_lane, max_indel_num);

	//Current distance pointer set to 0
	int cur_dist = 0;
	//Start at the main lane
	int cur_lane = main_lane;
	//Set the first cost of the main lane to 0
	path[cur_lane].path_cost[path[cur_lane].front_idx] = 0;

	bool ED_finished = false;

	//String pointers. Used in comparing characters
	int test_idx;
	int ref_idx;

	//Do string Comparison.
	while (true) {
		//First Pick a path.
		while (path[cur_lane].path_cost[path[cur_lane].front_idx] != cur_dist) {
			if (cur_lane == max_indel_num * 2 + 1) { //Next is boundary
				if (cur_dist >= max_diff_num) { //Check if we exceeds the max diff tolarence
					//TODO: BREAK AND END.
					ED_finished = true;
					result.correct = false;
					break;
				} else {
					//Go to next tier
					cur_lane = 1;
					cur_dist++;
				}
			} else {
				cur_lane++;
			}
		}

		//Indicate if the lane should be stopped
		int slide_stop = 0;

		//Slide down the lane.
		*
		 * If the 2 string does not differ much, which we assume will be the common case after passing the filtering,
		 * then they would loop like 20~100 times. That's why we want to strip out the divergent portion and put it in
		 * a different path picking loop.
		 *
		int m = 0;
//		if (!ED_finished) {
			//for (int m = 0; m <= 10 * READ_LENGTH - key_num * KEY_LENGTH; m++) { 
			while (true) {
				printf("counter m is: %i\n", m);
				printf("sliding  threadId: %i\n", threadIdx.x);
				printf("path[cur_lane].front: path[cur_lane].front_idx: %i  threadId: %i\n", path[cur_lane].front_idx, threadIdx.x);
				if (cur_lane != main_lane)
					printf("*****cur_lane changed!!! it is: %i, main_lane is: %i\n", cur_lane, main_lane);
				//Conservative test, speed up common case
				if (path[cur_lane].front_idx >= READ_LENGTH - max_indel_num) {

					if ((cur_lane <= main_lane && path[cur_lane].front_idx
							== READ_LENGTH) //Insertion lane
							|| (cur_lane > main_lane && path[cur_lane].front_idx //Deletion Lane
									== READ_LENGTH + main_lane - cur_lane)) {
						printf("Slide ending!!  threadId: %i\n", threadIdx.x);
						ED_finished = true;
						result.correct = true;
						break;
					}
				}
*
				//Compute string idx:
				test_idx = path[cur_lane].front_idx + cur_lane - main_lane;
				ref_idx = path[cur_lane].front_idx;

				//test if can slide down
				if (!(path[cur_lane].path_cost[path[cur_lane].front_idx + 1]
						== cur_dist //If can just slide
						|| test_read[test_idx] == ref_read[ref_idx]))
					slide_stop = 1;

				//Check neighbor lanes and update them. Modify -> increment -> modify
				//Check lower neighbor
				if (path[cur_lane + 1].path_cost[path[cur_lane].front_idx]
						> cur_dist + slide_stop + 1)
					path[cur_lane + 1].path_cost[path[cur_lane].front_idx]
							= cur_dist + 1;
*
				//Increment
				path[cur_lane].front_idx++;
*
				//Check itself
				if (path[cur_lane].path_cost[path[cur_lane].front_idx] > cur_dist
						+ slide_stop)
					path[cur_lane].path_cost[path[cur_lane].front_idx] = cur_dist
							+ slide_stop;
				//Check upper (right) neighbor
				if (path[cur_lane - 1].path_cost[path[cur_lane].front_idx]
						> cur_dist + slide_stop + 1)
					path[cur_lane - 1].path_cost[path[cur_lane].front_idx]
							= cur_dist + 1;

				//stop if can't slide anymore
				if (slide_stop == 1)
					break;
*
			m++;
			}
			
//		}
		
		if (ED_finished)
			break;
		
	}
	
	DEBUG_PRINT2("inside editCalFWD, after path cost generation a  threadId: %i\n", threadIdx.x);
	DEBUG_PRINT2("inside editCalFWD, after path cost generation b  threadId: %i\n", threadIdx.x);

	//Tracing back period
	*
	 * After you got the matrix, and all distance number, you have to go back and find out which path is the
	 * "least divergent path" and also find out where are those insertions and deletions and mismatches.
	 * The code here should not divergent much since each element has to compare it's left, up and up-left neighbor.
	 * The only divergent part is when it see an insertion or mismatch, they will have to go inside the "if block"
	 * and update next iteration position. Otherwise, it's just 3 if tests. Since the "differences" between 2 strings
	 * should be fairly small (at most 5 out of 108 comparisons). I think it's acceptiable.
	 
	if (result.correct == false)
		return result;
	else { //If pass the test, trace back

		//char temp_result[30]; //Temp string. Used for appending.
		int cur_idx = (cur_lane <= main_lane) ? READ_LENGTH : READ_LENGTH
				+ main_lane - cur_lane;

		result.diff_num = path[cur_lane].path_cost[cur_idx];

		cur_dist = path[cur_lane].path_cost[cur_idx];

		int error_ptr = 0;

		while (cur_lane != main_lane || cur_idx != key_num * KEY_LENGTH) {

			//If we should have an insertion
			if (cur_idx == key_num * KEY_LENGTH
					|| path[cur_lane - 1].path_cost[cur_idx]
							< path[cur_lane].path_cost[cur_idx - 1]) {

				result.error[error_ptr].diff = INSERTION;
				result.error[error_ptr].location = cur_idx + cur_lane
						- main_lane - 1;
				result.error[error_ptr].diff_char
						= test_read[result.error[error_ptr].location];

				error_ptr++;
				cur_lane--;
				continue;
			}

			//If we should have a deletion
			if (path[cur_lane + 1].path_cost[cur_idx - 1]
					< path[cur_lane].path_cost[cur_idx - 1]) {

				result.error[error_ptr].diff = DELETION;
				result.error[error_ptr].location = cur_idx + cur_lane
						- main_lane - 1;
				result.error[error_ptr].diff_char = ref_read[cur_idx];

				error_ptr++;
				cur_lane++;
				cur_idx--;
				continue;
			}

			//Check if we have a mismatch
			if (path[cur_lane].path_cost[cur_idx - 1]
					< path[cur_lane].path_cost[cur_idx]) {

				result.error[error_ptr].diff = MISMATCH;
				result.error[error_ptr].location = cur_idx + cur_lane
						- main_lane - 1;
				result.error[error_ptr].diff_char
						= test_read[result.error[error_ptr].location];

				error_ptr++;
				cur_idx--;
				continue;
			}

			//Move to the next element
			cur_idx--;
		}
	}
	*
	DEBUG_PRINT2("inside editCalFWD, after tracing back generation a  threadId: %i\n", threadIdx.x);
	DEBUG_PRINT2("inside editCalFWD, after tracing back generation b  threadId: %i\n", threadIdx.x);
	return result;
}
*/

/*
 * Going Forward, starting from the key_num * KEY_LENGTH position and going forward until end.
 */
__device__ ED_result editDistanceCalFWD(char* test_read, char* ref_read,
		int key_num, ED_path* path, int main_lane, int max_indel_num,
		int max_diff_num) {
	//Return result;
	ED_result result;

	//Initialize the Front of each lane
	initializeFWDFront(key_num, path, main_lane, max_indel_num);

	//Current distance pointer set to 0
	int cur_dist = 0;
	//Start at the main lane
	int cur_lane = main_lane;
	//Set the first cost of the main lane to 0
	path[cur_lane].path_cost[path[cur_lane].front_idx] = 0;

	bool ED_finished = false;

	//String pointers. Used in comparing characters
	int test_idx;
	int ref_idx;

	//Do string Comparison.
	while (true) {
		//First Pick a path.
		while (path[cur_lane].path_cost[path[cur_lane].front_idx] != cur_dist) {
			if (cur_lane == max_indel_num * 2 + 1) { //Next is boundary
				if (cur_dist >= max_diff_num) { //Check if we exceeds the max diff tolarence
					//TODO: BREAK AND END.
					ED_finished = true;
					result.correct = false;
					break;
				} else {
					//Go to next tier
					cur_lane = 1;
					cur_dist++;
				}
			} else {
				cur_lane++;
			}
		}

		//Indicate if the lane should be stopped
		int slide_stop = 0;

		//Slide down the lane.
		for (int m = 0; m <= READ_LENGTH - key_num * KEY_LENGTH; m++) { 
		//while (true) {
			printf("counter m is: %i\n", m);
			printf("sliding  threadId: %i\n", threadIdx.x);
			printf("path[cur_lane].front: path[cur_lane].front_idx: %i  threadId: %i\n", path[cur_lane].front_idx, threadIdx.x);
			//Conservative test, speed up common case
			if (path[cur_lane].front_idx >= READ_LENGTH - max_indel_num) {
			
				if ((cur_lane <= main_lane && path[cur_lane].front_idx
						== READ_LENGTH) //Insertion lane
						|| (cur_lane > main_lane && path[cur_lane].front_idx //Deletion Lane
								== READ_LENGTH + main_lane - cur_lane)) {
					printf("Slide ending!!  threadId: %i\n", threadIdx.x);
					ED_finished = true;
					result.correct = true;
					break;
				}
			}

			//Compute string idx:
			test_idx = path[cur_lane].front_idx + cur_lane - main_lane;
			ref_idx = path[cur_lane].front_idx;

			//test if can slide down
			if (!(path[cur_lane].path_cost[path[cur_lane].front_idx + 1]
					== cur_dist //If can just slide
					|| test_read[test_idx] == ref_read[ref_idx]))
				slide_stop = 1;

			//Check neighbor lanes and update them. Modify -> increment -> modify
			//Check lower neighbor
			if (path[cur_lane + 1].path_cost[path[cur_lane].front_idx]
					> cur_dist + slide_stop + 1)
				path[cur_lane + 1].path_cost[path[cur_lane].front_idx]
						= cur_dist + 1;
			//Increment
			path[cur_lane].front_idx++;
			//Check itself
			if (path[cur_lane].path_cost[path[cur_lane].front_idx] > cur_dist
					+ slide_stop)
				path[cur_lane].path_cost[path[cur_lane].front_idx] = cur_dist
						+ slide_stop;
			//Check upper (right) neighbor
			if (path[cur_lane - 1].path_cost[path[cur_lane].front_idx]
					> cur_dist + slide_stop + 1)
				path[cur_lane - 1].path_cost[path[cur_lane].front_idx]
						= cur_dist + 1;

			//stop if can't slide anymore
			if (slide_stop == 1)
				break;
		}
		
		if (ED_finished)
			break;
		
	}
	
	DEBUG_PRINT2("inside editCalFWD, after path cost generation a  threadId: %i\n", threadIdx.x);
	DEBUG_PRINT2("inside editCalFWD, after path cost generation b  threadId: %i\n", threadIdx.x);

	//Tracing back period

	if (result.correct == false)
		return result;
	else { //If pass the test, trace back

		//char temp_result[30]; //Temp string. Used for appending.
		int cur_idx = (cur_lane <= main_lane) ? READ_LENGTH : READ_LENGTH
				+ main_lane - cur_lane;

		result.diff_num = path[cur_lane].path_cost[cur_idx];

		cur_dist = path[cur_lane].path_cost[cur_idx];

		int error_ptr = 0;

		while (cur_lane != main_lane || cur_idx != key_num * KEY_LENGTH) {

			//If we should have an insertion
			if (cur_idx == key_num * KEY_LENGTH
					|| path[cur_lane - 1].path_cost[cur_idx]
							< path[cur_lane].path_cost[cur_idx - 1]) {

				result.error[error_ptr].diff = INSERTION;
				result.error[error_ptr].location = cur_idx + cur_lane
						- main_lane - 1;
				result.error[error_ptr].diff_char
						= test_read[result.error[error_ptr].location];

				error_ptr++;
				cur_lane--;
				continue;
			}

			//If we should have a deletion
			if (path[cur_lane + 1].path_cost[cur_idx - 1]
					< path[cur_lane].path_cost[cur_idx - 1]) {

				result.error[error_ptr].diff = DELETION;
				result.error[error_ptr].location = cur_idx + cur_lane
						- main_lane - 1;
				result.error[error_ptr].diff_char = ref_read[cur_idx];

				error_ptr++;
				cur_lane++;
				cur_idx--;
				continue;
			}

			//Check if we have a mismatch
			if (path[cur_lane].path_cost[cur_idx - 1]
					< path[cur_lane].path_cost[cur_idx]) {

				result.error[error_ptr].diff = MISMATCH;
				result.error[error_ptr].location = cur_idx + cur_lane
						- main_lane - 1;
				result.error[error_ptr].diff_char
						= test_read[result.error[error_ptr].location];

				error_ptr++;
				cur_idx--;
				continue;
			}

			//Move to the next element
			cur_idx--;
		}
	}
	DEBUG_PRINT2("inside editCalFWD, after tracing back generation a  threadId: %i\n", threadIdx.x);
	DEBUG_PRINT2("inside editCalFWD, after tracing back generation b  threadId: %i\n", threadIdx.x);
	return result;
}

/*
 * BWD is backward. It's pretty much the same with forward. The difference is just sliding direction. It's going
 * backward. The stuctures are all the same. No comments here.
 */
__device__ ED_result editDistanceCalBWD(char* test_read, char* ref_read,
		int key_num, ED_path* path, int main_lane, int max_indel_num,
		int max_diff_num) {
	//Return result;
	ED_result result;

	//Initialize the Front of each lane
	initializeBWDFront(key_num, path, main_lane, max_indel_num);

	//Current distance pointer set to 0
	int cur_dist = 0;
	//Start at the main lane
	int cur_lane = main_lane;
	//Set the first cost of the main lane to 0
	path[cur_lane].path_cost[path[cur_lane].front_idx] = 0;

	bool ED_finished = false;

	//String pointers. Used in comparing characters
	int test_idx;
	int ref_idx;

	//Pick a lane to go through
	while (!ED_finished) {
		//First Pick a path.
		while (path[cur_lane].path_cost[path[cur_lane].front_idx] != cur_dist) {
			if (cur_lane == max_indel_num * 2 + 1) { //Next is boundary
				if (cur_dist >= max_diff_num) { //Check if we exceeds the max diff tolarence
					//TODO: BREAK AND END.
					ED_finished = true;
					result.correct = false;
					break;
				} else {
					//Go to next tier
					cur_lane = 1;
					cur_dist++;
				}
			} else {
				cur_lane++;
			}
		}

		//Indicate if the lane should be stopped
		int slide_stop = 0;

		//Slide down the lane.
		while (!ED_finished) {
			//Conservative test, speed up common case
			if (path[cur_lane].front_idx <= max_indel_num) {
				//Test if it's the last element
				if ((cur_lane >= main_lane && path[cur_lane].front_idx == 0) //Insertion lane
						|| (cur_lane < main_lane && path[cur_lane].front_idx //Deletion Lane
								== main_lane - cur_lane)) {
					ED_finished = true;
					result.correct = true;
					break;
				}
			}

			//Compute string idx:
			test_idx = path[cur_lane].front_idx + cur_lane - main_lane - 1;
			ref_idx = path[cur_lane].front_idx - 1;

			//test if can slide down
			if (!(path[cur_lane].path_cost[path[cur_lane].front_idx - 1]
					== cur_dist //If can just slide
					|| test_read[test_idx] == ref_read[ref_idx]))
				slide_stop = 1;

			//Check neighbor lanes and update them. Modify -> decrement -> modify
			//Check upper neighbor
			if (path[cur_lane - 1].path_cost[path[cur_lane].front_idx]
					> cur_dist + slide_stop + 1)
				path[cur_lane - 1].path_cost[path[cur_lane].front_idx]
						= cur_dist + 1;
			//Decrement
			path[cur_lane].front_idx--;
			//Check itself
			if (path[cur_lane].path_cost[path[cur_lane].front_idx] > cur_dist
					+ slide_stop)
				path[cur_lane].path_cost[path[cur_lane].front_idx] = cur_dist
						+ slide_stop;
			//Check lower (left) neighbor
			if (path[cur_lane + 1].path_cost[path[cur_lane].front_idx]
					> cur_dist + slide_stop + 1)
				path[cur_lane + 1].path_cost[path[cur_lane].front_idx]
						= cur_dist + 1;

			//stop if can't slide anymore
			if (slide_stop == 1) {
				break;
			}
		}
	}
	
	DEBUG_PRINT2("inside editCalBWD, after path cost generation a  threadId: %i\n", threadIdx.x);
	DEBUG_PRINT2("inside editCalBWD, after path cost generation b  threadId: %i\n", threadIdx.x);

	//Tracing back period
	if (result.correct == false)
		return result;
	else { //If pass the test, trace back

		//char temp_result[30]; //Temp string. Used for appending.
		int cur_idx = (cur_lane >= main_lane) ? 0 : main_lane - cur_lane;

		result.diff_num = path[cur_lane].path_cost[cur_idx];

		cur_dist = path[cur_lane].path_cost[cur_idx];

		int error_ptr = 0;

		while (cur_lane != main_lane || cur_idx != key_num * KEY_LENGTH) {
			//If we should have an insertion
			if (cur_idx == key_num * KEY_LENGTH
					|| path[cur_lane + 1].path_cost[cur_idx]
							< path[cur_lane].path_cost[cur_idx + 1]) {

				result.error[error_ptr].diff = INSERTION;
				result.error[error_ptr].location = cur_idx + cur_lane
						- main_lane;
				result.error[error_ptr].diff_char
						= test_read[result.error[error_ptr].location];

				error_ptr++;
				cur_lane++;
				continue;
			}

			//If we should have a deletion
			if (path[cur_lane - 1].path_cost[cur_idx + 1]
					< path[cur_lane].path_cost[cur_idx + 1]) {

				result.error[error_ptr].diff = DELETION;
				result.error[error_ptr].location = cur_idx + cur_lane
						- main_lane;
				result.error[error_ptr].diff_char = ref_read[cur_idx];

				error_ptr++;
				cur_lane--;
				cur_idx++;
				continue;
			}

			//Check if we have a mismatch
			if (path[cur_lane].path_cost[cur_idx + 1]
					< path[cur_lane].path_cost[cur_idx]) {

				result.error[error_ptr].diff = MISMATCH;
				result.error[error_ptr].location = cur_idx + cur_lane
						- main_lane;
				result.error[error_ptr].diff_char
						= test_read[result.error[error_ptr].location];

				error_ptr++;
				cur_idx++;
				continue;
			}

			//Move to the next element
			cur_idx++;
		}
	}
	DEBUG_PRINT2("inside editCalBWD, after tracing back generation a  threadId: %i\n", threadIdx.x);
	DEBUG_PRINT2("inside editCalBWD, after tracing back generation b  threadId: %i\n", threadIdx.x);
	return result;
}

