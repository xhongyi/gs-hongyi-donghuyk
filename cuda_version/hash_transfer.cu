/*
 * hash_transfer.cpp
 *
 *  Created on: Oct 8, 2011
 *      Author: mac
 */
#include <cstdlib>
#include <sstream>
#include <iostream>
#include <assert.h>
#include "hip/hip_runtime.h"
#include "common.h"
#include "hash_transfer.h"

int* hash_table;
int* coordinate;
string ref_string;
int* dev_coordinate;
int* dev_hash_table;
char* dev_ref_string;
//texture<int> tex_hash_table;
//texture<int> tex_coordinate;
//texture<char> tex_ref_string;

void getHashTablePtr(int ** ptr) {
	*ptr = hash_table;
}

void getCoordinatePtr(int ** ptr) {
	*ptr = coordinate;
}

long long loadHash(string hash_name) {
	long long coord_num = hashReconstructor(&hash_table, &coordinate, hash_name.c_str());
	HANDLE_ERROR(hipMalloc((void**)&dev_hash_table, INDEX_NUM*sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_coordinate, coord_num*sizeof(int)));
	HANDLE_ERROR(hipBindTexture(NULL, tex_hash_table, dev_hash_table, INDEX_NUM*sizeof(int)));
	HANDLE_ERROR(hipBindTexture(NULL, tex_coordinate, dev_coordinate, coord_num*sizeof(int)));
	HANDLE_ERROR(hipMemcpy(dev_coordinate, coordinate, coord_num*sizeof(int), hipMemcpyHostToDevice ));
	HANDLE_ERROR(hipMemcpy(dev_hash_table, hash_table, INDEX_NUM*sizeof(int), hipMemcpyHostToDevice ));
	return coord_num;
}

int loadRef(string ref_name) {
        char tmp_ref_name[50];
        strcpy(tmp_ref_name, ref_name.c_str());
        refLoader(ref_string, tmp_ref_name);
	char *tmp_ref_string = (char*)malloc( ref_string.size() * sizeof(char));
        strcpy(tmp_ref_string, ref_string.c_str());
	HANDLE_ERROR(hipMalloc((void**)&dev_ref_string, ref_string.size() * sizeof(char)));
        HANDLE_ERROR(hipBindTexture(NULL, tex_ref_string, dev_ref_string, ref_string.size() * sizeof(char)));
        HANDLE_ERROR(hipMemcpy(dev_ref_string, tmp_ref_string, ref_string.size() * sizeof(char), hipMemcpyHostToDevice ));
        return ref_string.size();
}

void freeHash(void) {
	free (hash_table);
	free (coordinate);
	HANDLE_ERROR( hipUnbindTexture (tex_hash_table) );
	HANDLE_ERROR( hipUnbindTexture (tex_coordinate) );
	HANDLE_ERROR( hipFree (dev_hash_table) );
	HANDLE_ERROR( hipFree (dev_coordinate) );
}

void freeRef(void) {
	HANDLE_ERROR( hipUnbindTexture (tex_ref_string) );
	HANDLE_ERROR( hipFree (dev_ref_string) );
}
