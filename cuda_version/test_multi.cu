/*
 * test_search.cpp
 *
 *  Created on: Oct 19, 2011
 *	  Author: mac
 */
#include <iostream>
#include "universal_header.h"

using namespace std;


#include "hash_test.cu"


int test_cuda(void) {

	int ref_num = loadRef("toy_ref_0");
	long long coord_num = loadHash("toy_hash_0");

	hash_test(coord_num);
	ref_test(ref_num);
	int fragment_set = 4;
	GPU_fragment test_fragment[fragment_set];
	strcpy(test_fragment[0].fragment, 
	"GGGTGGTAGGTGCAGAGACGGGAGGGGCAGAGCCGCAGGCACAGCCAAGAGGGCTGAAGAAATGGTAGAACGGAGCAGCTGGTGATGTGTGGGCCCACCGGCCCCAGG");
	strcpy(test_fragment[1].fragment, 
	"TGGCCCTGGGAGAGCAGGTGGAAGATCAGGCAGGCCATCGCTGCCACAGAACCCAGTGGATTGGCCTAGGTGGGATCTCTGAGCTCAACAAGCCCTCTCTCTTAGTTT");
	strcpy(test_fragment[2].fragment, 
	"CCTAACCCAACCCTAACCCTAACCCTAACCCTAACCCTAACCCTAACCCCTAACCCTAACCCTAACCCTAACCCTAACCTAACCCTAACCCTAACCCTAACCCTAACC");
	strcpy(test_fragment[3].fragment, 
	"CCTAACCCTAACCCTAACCCTAACCCTAACCCTAACCCCTAACCCTAACCCTAACCCTAACCCTCGCGGTACCCTCAGCCGGCCCGCCCGCCCGGGTCTGACCTGAGG");

	//Getting the sort key.
	cout << "********** Fragment Input **************************************************" << endl;
	for (int k = 0; k < fragment_set; k++) {
		key_struct sort_input[KEY_NUMBER];
		for (int i = 0; i < KEY_NUMBER; i++) {
			char key[KEY_LENGTH];
			for (int j = 0; j < KEY_LENGTH; j++) {
				key[j] = test_fragment[k].fragment[j + KEY_LENGTH * i];
			}
			int key_hash = hashVal(key);
			int key_entry = hash_table[key_hash];
			int key_entry_size = coordinate[key_entry];
			sort_input[i].order = 0;
			sort_input[i].key_number = i;
			sort_input[i].key_entry = key_entry;
			sort_input[i].key_entry_size = key_entry_size;
		}
		sortPrefilter(test_fragment[k].sorted_keys, sort_input);
		for (int i = 0; i < KEY_NUMBER; i++) {
			cout << " fragment_set: "	<< k
				 << " key_number:"		<< test_fragment[k].sorted_keys[i].key_number 
				 << " key_entry: "		<< test_fragment[k].sorted_keys[i].key_entry 
				 << " key_entry_size: " << test_fragment[k].sorted_keys[i].key_entry_size 
				 << " base: " 			<< test_fragment[k].sorted_keys[i].base << endl;
		}
		cout << "****************************************************************************" << endl;
	}

	GPU_fragment* dev_fragment;
	final_result* dev_result;
	final_result test_result[fragment_set];

	hipMalloc((void**) &dev_fragment, sizeof(GPU_fragment)*fragment_set);
	hipMalloc((void**) &dev_result,   sizeof(final_result)*fragment_set);
	hipMemcpy(dev_fragment, &test_fragment, sizeof(GPU_fragment)*fragment_set, hipMemcpyHostToDevice);
	searchFragment <<<fragment_set, 1>>> (dev_fragment, fragment_set, dev_ref_string, dev_hash_table, 
								dev_coordinate, 5, 5, dev_result);
	hipMemcpy(test_result, dev_result, sizeof(final_result)*fragment_set, hipMemcpyDeviceToHost);
	for (int j = 0; j < fragment_set; j++) {
		cout << "****************************************************************************" << endl;
		cout << "Fragment	  : " << test_result[j].fragment << endl;
		cout << "Spilled	  : " << test_result[j].spilled << endl;
		cout << "Result Size  : " << test_result[j].size << endl;
		cout << "****************************************************************************" << endl;
		for (int i = 0; i < test_result[j].size; i++) {
			cout << "* Result coor  : " << test_result[j].coor_results[i].coordiante;
			cout << "	Result diff  : " << test_result[j].coor_results[i].diff_num << endl;
		}
		cout << "****************************************************************************" << endl;
	}
	hipFree(dev_fragment);
	hipFree(dev_result);
	freeHash();
	freeRef();
	return 0;
}

int main() {
	test_cuda();
	return 0;
}
