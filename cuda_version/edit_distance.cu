#include "hip/hip_runtime.h"
/*
 * edit_distance.cpp
 *
 *  Created on: Oct 29, 2011
 *      Author: mac
 */

/*
 *
 */
#define		_MAIN_PATH_		0
#define		_DELETE_PATH_	1
#define		_INSERT_PATH_	2
#define		_UN_FILLED_		1000

#include "edit_distance.h"
#include <algorithm>
#include <cstdlib>
//#include <cstring>
#include <iostream>
#include <assert.h>
/*
 *          reference string
 * 		  a p p l i c a t i o n
 *      0 1 2
 * t  a 1 0 1 2
 * e  p 2 1 0 1 2    <----delete_path
 * s  p   2 1 0 1 2
 * t  p     2 1 1 2 3
 *    l       2 1 2 3
 * s  o     ^   2 2 3 <------main_path
 * t  c     |     3 2
 * i  a     |         2
 * n  t     |           2
 * g  i     |             2
 *    o     |               2
 *	        |
 *	    insert_path
 *
 *
 *          reference string
 *        a p p l i c a t i o n
 *      0 1 2 3
 * t  a 1 0 1 2 3
 * e  p 2 1 0 1 2 3  <----bondary_path
 * s  p 3 2 1 0 1 2 3
 * t  p   3 2 1 1 2 3 4
 *    l     3 2 1 2 3 4 5
 * s  o     ^ 3 2 2 3 4 5 <----delete_path
 * t  c     |   3 3 2 3 4
 * i  a     |       3 2 3 4
 * n  i     |         3 3 3 4
 * g  o     |             4 3 4
 *    n     |             ^ 4 3 <------main_path
 *	        |             |
 *	   bondary_path  insert_path
 *      |
 *      | | | 3 3 3 3 4 5
 *      | | 2 2 2 2 3 4 5
 *      | 1 1 1 1 2 3 4 4 4 4 4
 *      0 0 0 0 1 2 3 3 3 3 3 3
 *      1 1 1 1 1 2 2 2 3 4 4 |
 *      2 2 2 2 2 3 3 3   | | |
 *      3 3 3 3 3       | | | |
 *
 *
 *
 *
 *
 *
 *
 *
 *
 */
__device__ void initializePath(ED_path* path, int main_lane);

__device__ ED_result editDistanceCalFWD(char* test_read, char* ref_read, int key_num, ED_path* path, int main_lane);

__device__ ED_result editDistanceCalBWD(char* test_read, char* ref_read, int key_num, ED_path* path, int main_lane);

__device__ void initializeFWDFront(int key_num, ED_path* path, int main_lane);

__device__ void initializeBWDFront(int key_num, ED_path* path, int main_lane);

// initializePath only fills the path elements now.
__device__ void initializePath(ED_path* path, int* main_lane) {
	for (int i = 0; i < max_indel_num * 2 + 3; i++) {
		for (int j = 0; j <= READ_LENGTH; j++) {
			path[i].path_cost[j] = _UN_FILLED_;
		}
	}
}

__device__ void initializeFWDFront(int key_num, ED_path* path, int main_lane) {
	for (int i = 0; i < max_indel_num * 2 + 3; i++)
		//For the insertion lanes, the front point is shift right.
		path[i].front_idx = (i < main_lane) ? key_num * KEY_LENGTH + main_lane
				- i : key_num * KEY_LENGTH;
}

__device__ void initializeBWDFront(int key_num, ED_path* path, int main_lane) {
	for (int i = 0; i < max_indel_num * 2 + 3; i++)
		//For the insertion lanes, the front point is shift right.
		path[i].front_idx = (i > main_lane) ? key_num * KEY_LENGTH + main_lane
				- i : key_num * KEY_LENGTH;
}

__device__ ED_result editDistanceCal(char* test_read, char* ref_read,
		int key_num, ED_path* path, int main_lane) {
	//Initialize path
	ED_result result;
	ED_result FWD_result, BWD_result;
	initializePath(path, main_lane);
	FWD_result = editDistanceCalFWD(test_read, ref_read, key_num, path,
			main_lane);
	BWD_result = editDistanceCalBWD(test_read, ref_read, key_num, path,
			main_lane);

	result.diff_num = FWD_result.diff_num + BWD_result.diff_num;

	//This is the result index used to probe the result error queue.
	int error_idx = 0;

	if (FWD_result.correct && BWD_result.correct && result.diff_num
			<= max_diff_num) {

		result.correct = true;

		for (int i = 0; i < BWD_result.diff_num; i++) {
			result.error[error_idx] = BWD_result.error[i];
			error_idx++;
		}

		for (int i = FWD_result.diff_num - 1; i >= 0; i--) {
			result.error[error_idx] = FWD_result.error[i];
			error_idx++;
		}

		//The total diff_num should be equal to the error number just filled.
		assert(error_idx == result.diff_num);

		/*
		 cout << "***" << endl;
		 cout << "Total difference Number: " << result.diff_num << endl;

		 for (int i = 0; i < result.diff_num; i++) {
		 switch (result.error[i].diff) {
		 case MISMATCH:
		 cout << "Mismatch at " << result.error[i].location << endl;
		 cout << "Error: " << result.error[i].diff_char << endl;
		 break;
		 case INSERTION:
		 cout << "Insertion at " << result.error[i].location << endl;
		 cout << "Error: " << result.error[i].diff_char << endl;
		 break;
		 case DELETION:
		 cout << "Deletion at " << result.error[i].location << endl;
		 cout << "Error: " << result.error[i].diff_char << endl;
		 break;
		 }
		 }

		 cout << "###" << endl;
		 */

	} else
		result.correct = false;

	return result;
}

__device__ ED_result editDistanceCalFWD(char* test_read, char* ref_read, int key_num, ED_path* path, int main_lane) {
	//Return result;
	ED_result result;
	//strcpy(result.compare_result, "\0");

	//Initialize the Front of each lane
	initializeFWDFront(key_num, path, main_lane);

	//Current distance pointer set to 0
	int cur_dist = 0;
	//Start at the main lane
	int cur_lane = main_lane;
	//Set the first cost of the main lane to 0
	path[cur_lane].path_cost[path[cur_lane].front_idx] = 0;

	bool ED_finished = false;

	//String pointers. Used in comparing characters
	int test_idx;
	int ref_idx;

	//Pick a lane to go through
	while (!ED_finished) {
		//cout << "Here 0" << endl;
		//First Pick a path.
		while (path[cur_lane].path_cost[path[cur_lane].front_idx] != cur_dist) {
			if (cur_lane == max_indel_num * 2 + 1) { //Next is boundary
				if (cur_dist >= max_diff_num) { //Check if we exceeds the max diff tolarence
					//TODO: BREAK AND END.
					ED_finished = true;
					result.correct = false;
					break;
				} else {
					//Go to next tier
					cur_lane = 1;
					cur_dist++;
				}
			} else {
				cur_lane++;
			}
		}

		//Indicate if the lane should be stopped
		int slide_stop = 0;

		//cout << "cur_dist: " << cur_dist << endl;
		//cout << "cur_lane: " << cur_lane << endl;

		//Slide down the lane.
		while (!ED_finished) {
			//cout << "lane_front_idx: " << path[cur_lane].front_idx << endl;
			//Conservative test, speed up common case
			if (path[cur_lane].front_idx >= READ_LENGTH - max_indel_num) {
				//Test if it's the last element
				if ((cur_lane <= main_lane && path[cur_lane].front_idx
						== READ_LENGTH) //Insertion lane
						|| (cur_lane > main_lane && path[cur_lane].front_idx //Deletion Lane
								== READ_LENGTH + main_lane - cur_lane)) {
					ED_finished = true;
					result.correct = true;
					break;
				}
			}

			//Compute string idx:
			test_idx = path[cur_lane].front_idx + cur_lane - main_lane;
			ref_idx = path[cur_lane].front_idx;

			//test if can slide down
			if (!(path[cur_lane].path_cost[path[cur_lane].front_idx + 1]
					== cur_dist //If can just slide
					|| test_read[test_idx] == ref_read[ref_idx]))
				slide_stop = 1;

			//Check neighbor lanes and update them. Modify -> increment -> modify
			//Check lower neighbor
			if (path[cur_lane + 1].path_cost[path[cur_lane].front_idx]
					> cur_dist + slide_stop + 1)
				path[cur_lane + 1].path_cost[path[cur_lane].front_idx]
						= cur_dist + 1;
			//Increment
			path[cur_lane].front_idx++;
			//Check itself
			if (path[cur_lane].path_cost[path[cur_lane].front_idx] > cur_dist
					+ slide_stop)
				path[cur_lane].path_cost[path[cur_lane].front_idx] = cur_dist
						+ slide_stop;
			//Check upper (right) neighbor
			if (path[cur_lane - 1].path_cost[path[cur_lane].front_idx]
					> cur_dist + slide_stop + 1)
				path[cur_lane - 1].path_cost[path[cur_lane].front_idx]
						= cur_dist + 1;

			//stop if can't slide anymore
			if (slide_stop == 1)
				break;
		}
	}

	//cout << "Path generated" << endl;

	//Tracing back period
	if (result.correct == false)
		return result;
	else { //If pass the test, trace back

		char temp_result[30]; //Temp string. Used for appending.
		int cur_idx = (cur_lane <= main_lane) ? READ_LENGTH : READ_LENGTH
				+ main_lane - cur_lane;

		result.diff_num = path[cur_lane].path_cost[cur_idx];

		cur_dist = path[cur_lane].path_cost[cur_idx];

		int error_ptr = 0;

		//		int same_count = 0;
		//cout << "cur_lane: " << cur_lane << " cur_idx: " << cur_idx << endl;
		while (cur_lane != main_lane || cur_idx != key_num * KEY_LENGTH) {
			//cout << "cur_lane: " << cur_lane << " cur_idx: " << cur_idx << endl;

			//If we should have an insertion
			if (cur_idx == key_num * KEY_LENGTH
					|| path[cur_lane - 1].path_cost[cur_idx]
							< path[cur_lane].path_cost[cur_idx - 1]) {

				result.error[error_ptr].diff = INSERTION;
				result.error[error_ptr].location = cur_idx + cur_lane
						- main_lane - 1;
				result.error[error_ptr].diff_char
						= test_read[result.error[error_ptr].location];
				error_ptr++;
				/*
				 if (same_count != 0) {
				 //cout << "Here m" << endl;
				 strcpy(temp_result, result.compare_result);
				 sprintf(result.compare_result, "^%c%d%s",
				 test_read[cur_idx + cur_lane - main_lane - 1],
				 same_count, temp_result);
				 }
				 else {
				 //cout << "Here n" << endl;
				 strcpy(temp_result, result.compare_result);
				 sprintf(result.compare_result, "^%c%s",
				 test_read[cur_idx + cur_lane - main_lane - 1],
				 temp_result);
				 }
				 */
				cur_lane--;
				//same_count = 0;
				continue;
			}

			//If we should have a deletion
			if (path[cur_lane + 1].path_cost[cur_idx - 1]
					< path[cur_lane].path_cost[cur_idx - 1]) {

				result.error[error_ptr].diff = DELETION;
				result.error[error_ptr].location = cur_idx + cur_lane
						- main_lane - 1;
				result.error[error_ptr].diff_char = ref_read[cur_idx];
				error_ptr++;
				/*
				 if (same_count != 0) {
				 //cout << "Here x" << endl;
				 strcpy(temp_result, result.compare_result);
				 sprintf(result.compare_result, "`%c%d%s",
				 ref_read[cur_idx - 1], same_count,
				 temp_result);
				 } else {
				 //cout << "Here y" << endl;
				 strcpy(temp_result, result.compare_result);
				 sprintf(result.compare_result, "`%c%s", ref_read[cur_idx - 1],
				 temp_result);
				 }
				 */
				cur_lane++;
				cur_idx--;
				//same_count = 0;
				continue;
			}

			//Check if we have a mismatch
			if (path[cur_lane].path_cost[cur_idx - 1]
					< path[cur_lane].path_cost[cur_idx]) {

				result.error[error_ptr].diff = MISMATCH;
				result.error[error_ptr].location = cur_idx + cur_lane
						- main_lane - 1;
				result.error[error_ptr].diff_char
						= test_read[result.error[error_ptr].location];
				error_ptr++;
				/*
				 //cout << "Here1" << endl;
				 if (same_count != 0) {
				 strcpy(temp_result, result.compare_result);
				 sprintf(result.compare_result, "%c%d%s",
				 test_read[cur_idx + cur_lane - main_lane - 1],
				 same_count, temp_result);

				 //cout << "Here2" << endl;
				 } else {
				 strcpy(temp_result, result.compare_result);
				 sprintf(result.compare_result, "%c%s",
				 test_read[cur_idx + cur_lane - main_lane - 1],
				 temp_result);
				 //cout << "Here3" << endl;
				 }
				 */
				cur_idx--;
				//same_count = 0;
				continue;
			}

			//Move to the next element
			cur_idx--;
			//same_count++;
			//cout << "same_count: " << same_count << endl;
		}

		//The 2 number should match. The error iteration should finally meet the total number
		assert (error_ptr == result.diff_num);
		/*
		 if (same_count != 0) { //If we have some same count at the begining
		 //cout << "Here t" << endl;
		 strcpy(temp_result, result.compare_result);
		 sprintf(result.compare_result, "%d%s", same_count,
		 temp_result);
		 }
		 */
	}
	return result;
}

__device__ ED_result editDistanceCalBWD(char* test_read, char* ref_read, int key_num, ED_path* path, int main_lane) {
	//Return result;
	ED_result result;
	//strcpy(result.compare_result, "\0");

	//Initialize the Front of each lane
	initializeBWDFront(key_num, path, main_lane);

	//Current distance pointer set to 0
	int cur_dist = 0;
	//Start at the main lane
	int cur_lane = main_lane;
	//Set the first cost of the main lane to 0
	path[cur_lane].path_cost[path[cur_lane].front_idx] = 0;

	bool ED_finished = false;

	//String pointers. Used in comparing characters
	int test_idx;
	int ref_idx;

	//Pick a lane to go through
	while (!ED_finished) {
		//cout << "Here 0" << endl;
		//First Pick a path.
		while (path[cur_lane].path_cost[path[cur_lane].front_idx] != cur_dist) {
			if (cur_lane == max_indel_num * 2 + 1) { //Next is boundary
				if (cur_dist >= max_diff_num) { //Check if we exceeds the max diff tolarence
					//TODO: BREAK AND END.
					ED_finished = true;
					result.correct = false;
					break;
				} else {
					//Go to next tier
					cur_lane = 1;
					cur_dist++;
				}
			} else {
				cur_lane++;
			}
		}

		//Indicate if the lane should be stopped
		int slide_stop = 0;

		//cout << "cur_dist: " << cur_dist << endl;
		//cout << "cur_lane: " << cur_lane << endl;

		//Slide down the lane.
		while (!ED_finished) {
			//cout << "lane_front_idx: " << path[cur_lane].front_idx << endl;
			//Conservative test, speed up common case
			if (path[cur_lane].front_idx <= max_indel_num) {
				//Test if it's the last element
				if ((cur_lane >= main_lane && path[cur_lane].front_idx == 0) //Insertion lane
						|| (cur_lane < main_lane && path[cur_lane].front_idx //Deletion Lane
								== main_lane - cur_lane)) {
					ED_finished = true;
					result.correct = true;
					break;
				}
			}

			//Compute string idx:
			test_idx = path[cur_lane].front_idx + cur_lane - main_lane - 1;
			ref_idx = path[cur_lane].front_idx - 1;

			//test if can slide down
			if (!(path[cur_lane].path_cost[path[cur_lane].front_idx - 1]
					== cur_dist //If can just slide
					|| test_read[test_idx] == ref_read[ref_idx]))
				slide_stop = 1;

			//Check neighbor lanes and update them. Modify -> decrement -> modify
			//Check upper neighbor
			if (path[cur_lane - 1].path_cost[path[cur_lane].front_idx]
					> cur_dist + slide_stop + 1)
				path[cur_lane - 1].path_cost[path[cur_lane].front_idx]
						= cur_dist + 1;
			//Decrement
			path[cur_lane].front_idx--;
			//Check itself
			if (path[cur_lane].path_cost[path[cur_lane].front_idx] > cur_dist
					+ slide_stop)
				path[cur_lane].path_cost[path[cur_lane].front_idx] = cur_dist
						+ slide_stop;
			//Check lower (left) neighbor
			if (path[cur_lane + 1].path_cost[path[cur_lane].front_idx]
					> cur_dist + slide_stop + 1)
				path[cur_lane + 1].path_cost[path[cur_lane].front_idx]
						= cur_dist + 1;

			//stop if can't slide anymore
			if (slide_stop == 1) {
				break;
			}
		}
	}

	//cout << "Path generated" << endl;

	//Tracing back period
	if (result.correct == false)
		return result;
	else { //If pass the test, trace back

		//char temp_result[30]; //Temp string. Used for appending.
		int cur_idx = (cur_lane >= main_lane) ? 0 : main_lane - cur_lane;

		cout << "cur_lane: " << cur_lane << endl;
		cout << "cur_idx: " << cur_idx << endl;

		result.diff_num = path[cur_lane].path_cost[cur_idx];

		cur_dist = path[cur_lane].path_cost[cur_idx];

		int error_ptr = 0;

		//		int same_count = 0;
		//cout << "cur_lane: " << cur_lane << " cur_idx: " << cur_idx << endl;
		while (cur_lane != main_lane || cur_idx != key_num * KEY_LENGTH) {
			cout << "cur_lane: " << cur_lane << " cur_idx: " << cur_idx
					<< " cur_distance: " << path[cur_lane].path_cost[cur_idx]
					<< endl;

			//If we should have an insertion
			if (cur_idx == key_num * KEY_LENGTH
					|| path[cur_lane + 1].path_cost[cur_idx]
							< path[cur_lane].path_cost[cur_idx + 1]) {

				result.error[error_ptr].diff = INSERTION;
				result.error[error_ptr].location = cur_idx + cur_lane
						- main_lane;
				result.error[error_ptr].diff_char
						= test_read[result.error[error_ptr].location];
				error_ptr++;
				/*
				 if (same_count != 0) {
				 //cout << "Here m" << endl;
				 strcpy(temp_result, result.compare_result);
				 sprintf(result.compare_result, "^%c%d%s",
				 test_read[cur_idx + cur_lane - main_lane - 1],
				 same_count, temp_result);
				 }
				 else {
				 //cout << "Here n" << endl;
				 strcpy(temp_result, result.compare_result);
				 sprintf(result.compare_result, "^%c%s",
				 test_read[cur_idx + cur_lane result_lane - 1],
				 temp_result);
				 }
				 */
				cur_lane++;
				//same_count = 0;
				continue;
			}

			//If we should have a deletion
			if (path[cur_lane - 1].path_cost[cur_idx + 1]
					< path[cur_lane].path_cost[cur_idx + 1]) {

				result.error[error_ptr].diff = DELETION;
				result.error[error_ptr].location = cur_idx + cur_lane
						- main_lane;
				result.error[error_ptr].diff_char = ref_read[cur_idx];
				error_ptr++;
				/*
				 if (same_count != 0) {
				 //cout << "Here x" << endl;
				 strcpy(temp_result, result.compare_result);
				 sprintf(result.compare_result, "`%c%d%s",
				 ref_read[cur_idx - 1], same_count,
				 temp_result);
				 } else {
				 //cout << "Here y" << endl;
				 strcpy(temp_result, result.compare_result);
				 sprintf(result.compare_result, "`%c%s", ref_read[cur_idx - 1],
				 temp_result);
				 }
				 */
				cur_lane--;
				cur_idx++;
				//same_count = 0;
				continue;
			}

			//Check if we have a mismatch
			if (path[cur_lane].path_cost[cur_idx + 1]
					< path[cur_lane].path_cost[cur_idx]) {

				result.error[error_ptr].diff = MISMATCH;
				result.error[error_ptr].location = cur_idx + cur_lane
						- main_lane;
				result.error[error_ptr].diff_char
						= test_read[result.error[error_ptr].location];
				error_ptr++;
				/*
				 //cout << "Here1" << endl;
				 if (same_count != 0) {
				 strcpy(temp_result, result.compare_result);
				 sprintf(result.compare_result, "%c%d%s",
				 test_read[cur_idx + cur_lane - main_lane - 1],
				 same_count, temp_result);

				 //cout << "Here2" << endl;
				 } else {
				 strcpy(temp_result, result.compare_result);
				 sprintf(result.compare_result, "%c%s",
				 test_read[cur_idx + cur_lane - main_lane - 1],
				 temp_result);
				 //cout << "Here3" << endl;
				 }
				 */
				cur_idx++;
				//same_count = 0;
				continue;
			}

			//Move to the next element
			cur_idx++;
			//same_count++;
			//cout << "same_count: " << same_count << endl;
		}

		//The 2 number should match. The error iteration should finally meet the total number
		cout << "error_ptr: " << error_ptr << endl;
		cout << "result.diff_num: " << result.diff_num << endl;
		assert (error_ptr == result.diff_num);
		/*
		 if (same_count != 0) { //If we have some same count at the begining
		 //cout << "Here t" << endl;
		 strcpy(temp_result, result.compare_result);
		 sprintf(result.compare_result, "%d%s", same_count,
		 temp_result);
		 }
		 */
	}
	return result;
}

