#include "hip/hip_runtime.h"
/*
 * edit_distance.cpp
 *
 *  Created on: Oct 29, 2011
 *      Author: mac
 */

/*
 *
 */
#define		_MAIN_PATH_		0
#define		_DELETE_PATH_	1
#define		_INSERT_PATH_	2
#define		_UN_FILLED_		1000

#include "edit_distance.h"
#include <algorithm>
#include <cstdlib>
//#include <cstring>
#include <iostream>
#include <assert.h>
/*
 *          reference string
 * 		  a p p l i c a t i o n
 *      0 1 2
 * t  a 1 0 1 2
 * e  p 2 1 0 1 2    <----delete_path
 * s  p   2 1 0 1 2
 * t  p     2 1 1 2 3
 *    l       2 1 2 3
 * s  o     ^   2 2 3 <------main_path
 * t  c     |     3 2
 * i  a     |         2
 * n  t     |           2
 * g  i     |             2
 *    o     |               2
 *	        |
 *	    insert_path
 *
 *
 *          reference string
 *        a p p l i c a t i o n
 *      0 1 2 3
 * t  a 1 0 1 2 3
 * e  p 2 1 0 1 2 3  <----bondary_path
 * s  p 3 2 1 0 1 2 3
 * t  p   3 2 1 1 2 3 4
 *    l     3 2 1 2 3 4 5
 * s  o     ^ 3 2 2 3 4 5 <----delete_path
 * t  c     |   3 3 2 3 4
 * i  a     |       3 2 3 4
 * n  i     |         3 3 3 4
 * g  o     |             4 3 4
 *    n     |             ^ 4 3 <------main_path
 *	        |             |
 *	   bondary_path  insert_path
 *      |
 *      | | | 3 3 3 3 4 5
 *      | | 2 2 2 2 3 4 5
 *      | 1 1 1 1 2 3 4 4 4 4 4
 *      0 0 0 0 1 2 3 3 3 3 3 3
 *      1 1 1 1 1 2 2 2 3 4 4 |
 *      2 2 2 2 2 3 3 3   | | |
 *      3 3 3 3 3       | | | |
 *
 */
__device__ void initializePath(ED_path* path, int main_lane, int max_indel_num);

__device__ ED_result editDistanceCalFWD(char* test_read, char* ref_read,
		int key_num, ED_path* path, int main_lane, int max_indel_num,
		int max_diff_num);

__device__ ED_result editDistanceCalBWD(char* test_read, char* ref_read,
		int key_num, ED_path* path, int main_lane, int max_indel_num,
		int max_diff_num);

__device__ void initializeFWDFront(int key_num, ED_path* path, int main_lane,
		int max_indel_num);

__device__ void initializeBWDFront(int key_num, ED_path* path, int main_lane,
		int max_indel_num);

// initializePath only fills the path elements now.
__device__ void initializePath(ED_path* path, int main_lane, int max_indel_num) {
	for (int i = 0; i < max_indel_num * 2 + 3; i++) {
		for (int j = 0; j <= READ_LENGTH; j++) {
			path[i].path_cost[j] = _UN_FILLED_;
		}
	}
}

__device__ void initializeFWDFront(int key_num, ED_path* path, int main_lane,
		int max_indel_num) {
	for (int i = 0; i < max_indel_num * 2 + 3; i++)
		//For the insertion lanes, the front point is shift right.
		path[i].front_idx = (i < main_lane) ? key_num * KEY_LENGTH + main_lane
				- i : key_num * KEY_LENGTH;
}

__device__ void initializeBWDFront(int key_num, ED_path* path, int main_lane,
		int max_indel_num) {
	for (int i = 0; i < max_indel_num * 2 + 3; i++)
		//For the insertion lanes, the front point is shift right.
		path[i].front_idx = (i > main_lane) ? key_num * KEY_LENGTH + main_lane
				- i : key_num * KEY_LENGTH;
}

__device__ ED_result editDistanceCal(char* test_read, char* ref_read,
		int key_num, ED_path* path, int main_lane, int max_indel_num,
		int max_diff_num) {

	DEBUG_PRINT1("Inside ED test 1\n");

	//Initialize path
	ED_result result;
	ED_result FWD_result, BWD_result;
	initializePath(path, main_lane, max_indel_num);
	/*----------------------------------------------------------------------
	for (int i = 0; i < max_indel_num * 2 + 3; i++) {
		for (int j = 0; j <= READ_LENGTH; j++) {
			path[i].path_cost[j] = _UN_FILLED_;
		}
	}
	----------------------------------------------------------------------*/
	DEBUG_PRINT2("FWDeditDistanceCal started  threadId: %i\n", threadIdx.x); //Idealy, We should first see all threads printing a since they are at the same warp
	FWD_result = editDistanceCalFWD(test_read, ref_read, key_num, path,
			main_lane, max_indel_num, max_diff_num);

	//__syncthreads(); //Force all threads synchronize.

	DEBUG_PRINT2("BWDeditDistanceCal started  threadId: %i\n", threadIdx.x); //We should see all threads print b. However, it is not doing that.
	BWD_result = editDistanceCalBWD(test_read, ref_read, key_num, path,
			main_lane, max_indel_num, max_diff_num);
	DEBUG_PRINT2("2 Directions have finished  threadId: %i\n", threadIdx.x);

	result.diff_num = FWD_result.diff_num + BWD_result.diff_num;

	//This is the result index used to probe the result error queue.
	int error_idx = 0;

	if (FWD_result.correct && BWD_result.correct && result.diff_num
			<= max_diff_num) {

		result.correct = true;

		for (int i = 0; i < BWD_result.diff_num; i++) {
			result.error[error_idx] = BWD_result.error[i];
			error_idx++;
		}

		for (int i = FWD_result.diff_num - 1; i >= 0; i--) {
			result.error[error_idx] = FWD_result.error[i];
			error_idx++;
		}

	} else
		result.correct = false;

	return result;
}

/*
 * Going Forward, starting from the key_num * KEY_LENGTH position and going forward until end.
 */
__device__ ED_result editDistanceCalFWD(char* test_read, char* ref_read,
		int key_num, ED_path* path, int main_lane, int max_indel_num,
		int max_diff_num) {
	//Return result;
	ED_result result;

	//Initialize the Front of each lane
	initializeFWDFront(key_num, path, main_lane, max_indel_num);

	//Current distance pointer set to 0
	int cur_dist = 0;
	//Start at the main lane
	int cur_lane = main_lane;
	//Set the first cost of the main lane to 0
	path[cur_lane].path_cost[path[cur_lane].front_idx] = 0;

	bool ED_finished = false;

	//String pointers. Used in comparing characters
	int test_idx;
	int ref_idx;

	//Do string Comparison.
	while (!ED_finished) {
		//First Pick a path.
		/*
		 * This Portion is the divergence portion, where each thread will pick a path to go through.
		 * The path pick may be different. But after a path is picked, there will be no difference.
		 * The picked path will be recorded in cur_lane.
		 */
		while (path[cur_lane].path_cost[path[cur_lane].front_idx] != cur_dist) {
			if (cur_lane == max_indel_num * 2 + 1) { //Next is boundary
				if (cur_dist >= max_diff_num) { //Check if we exceeds the max diff tolarence
					//TODO: BREAK AND END.
					ED_finished = true;
					result.correct = false;
					break;
				} else {
					//Go to next tier
					cur_lane = 1;
					cur_dist++;
				}
			} else {
				cur_lane++;
			}
		}

		//Indicate if the lane should be stopped
		int slide_stop = 0;

		//Slide down the lane.
		/*
		 * If the 2 string does not differ much, which we assume will be the common case after passing the filtering,
		 * then they would loop like 20~100 times. That's why we want to strip out the divergent portion and put it in
		 * a different path picking loop.
		 */
		while (!ED_finished) {
			//Conservative test, speed up common case
			if (path[cur_lane].front_idx >= READ_LENGTH - max_indel_num) {
				//Test if it's the last element
				if ((cur_lane <= main_lane && path[cur_lane].front_idx
						== READ_LENGTH) //Insertion lane
						|| (cur_lane > main_lane && path[cur_lane].front_idx //Deletion Lane
								== READ_LENGTH + main_lane - cur_lane)) {
					ED_finished = true;
					result.correct = true;
					break;
				}
			}

			//Compute string idx:
			test_idx = path[cur_lane].front_idx + cur_lane - main_lane;
			ref_idx = path[cur_lane].front_idx;

			//test if can slide down
			if (!(path[cur_lane].path_cost[path[cur_lane].front_idx + 1]
					== cur_dist //If can just slide
					|| test_read[test_idx] == ref_read[ref_idx]))
				slide_stop = 1;

			//Check neighbor lanes and update them. Modify -> increment -> modify
			//Check lower neighbor
			if (path[cur_lane + 1].path_cost[path[cur_lane].front_idx]
					> cur_dist + slide_stop + 1)
				path[cur_lane + 1].path_cost[path[cur_lane].front_idx]
						= cur_dist + 1;
			//Increment
			path[cur_lane].front_idx++;
			//Check itself
			if (path[cur_lane].path_cost[path[cur_lane].front_idx] > cur_dist
					+ slide_stop)
				path[cur_lane].path_cost[path[cur_lane].front_idx] = cur_dist
						+ slide_stop;
			//Check upper (right) neighbor
			if (path[cur_lane - 1].path_cost[path[cur_lane].front_idx]
					> cur_dist + slide_stop + 1)
				path[cur_lane - 1].path_cost[path[cur_lane].front_idx]
						= cur_dist + 1;

			//stop if can't slide anymore
			if (slide_stop == 1)
				break;
		}
	}
	
	DEBUG_PRINT2("inside editCalFWD, after path cost generation a  threadId: %i\n", threadIdx.x);
	DEBUG_PRINT2("inside editCalFWD, after path cost generation b  threadId: %i\n", threadIdx.x);

	//Tracing back period
	/*
	 * After you got the matrix, and all distance number, you have to go back and find out which path is the
	 * "least divergent path" and also find out where are those insertions and deletions and mismatches.
	 * The code here should not divergent much since each element has to compare it's left, up and up-left neighbor.
	 * The only divergent part is when it see an insertion or mismatch, they will have to go inside the "if block"
	 * and update next iteration position. Otherwise, it's just 3 if tests. Since the "differences" between 2 strings
	 * should be fairly small (at most 5 out of 108 comparisons). I think it's acceptiable.
	 */
	if (result.correct == false)
		return result;
	else { //If pass the test, trace back

		//char temp_result[30]; //Temp string. Used for appending.
		int cur_idx = (cur_lane <= main_lane) ? READ_LENGTH : READ_LENGTH
				+ main_lane - cur_lane;

		result.diff_num = path[cur_lane].path_cost[cur_idx];

		cur_dist = path[cur_lane].path_cost[cur_idx];

		int error_ptr = 0;

		while (cur_lane != main_lane || cur_idx != key_num * KEY_LENGTH) {

			//If we should have an insertion
			if (cur_idx == key_num * KEY_LENGTH
					|| path[cur_lane - 1].path_cost[cur_idx]
							< path[cur_lane].path_cost[cur_idx - 1]) {

				result.error[error_ptr].diff = INSERTION;
				result.error[error_ptr].location = cur_idx + cur_lane
						- main_lane - 1;
				result.error[error_ptr].diff_char
						= test_read[result.error[error_ptr].location];

				error_ptr++;
				cur_lane--;
				continue;
			}

			//If we should have a deletion
			if (path[cur_lane + 1].path_cost[cur_idx - 1]
					< path[cur_lane].path_cost[cur_idx - 1]) {

				result.error[error_ptr].diff = DELETION;
				result.error[error_ptr].location = cur_idx + cur_lane
						- main_lane - 1;
				result.error[error_ptr].diff_char = ref_read[cur_idx];

				error_ptr++;
				cur_lane++;
				cur_idx--;
				continue;
			}

			//Check if we have a mismatch
			if (path[cur_lane].path_cost[cur_idx - 1]
					< path[cur_lane].path_cost[cur_idx]) {

				result.error[error_ptr].diff = MISMATCH;
				result.error[error_ptr].location = cur_idx + cur_lane
						- main_lane - 1;
				result.error[error_ptr].diff_char
						= test_read[result.error[error_ptr].location];

				error_ptr++;
				cur_idx--;
				continue;
			}

			//Move to the next element
			cur_idx--;
		}
	}
	DEBUG_PRINT2("inside editCalFWD, after tracing back generation a  threadId: %i\n", threadIdx.x);
	DEBUG_PRINT2("inside editCalFWD, after tracing back generation b  threadId: %i\n", threadIdx.x);
	return result;
}

/*
 * BWD is backward. It's pretty much the same with forward. The difference is just sliding direction. It's going
 * backward. The stuctures are all the same. No comments here.
 */
__device__ ED_result editDistanceCalBWD(char* test_read, char* ref_read,
		int key_num, ED_path* path, int main_lane, int max_indel_num,
		int max_diff_num) {
	//Return result;
	ED_result result;

	//Initialize the Front of each lane
	initializeBWDFront(key_num, path, main_lane, max_indel_num);

	//Current distance pointer set to 0
	int cur_dist = 0;
	//Start at the main lane
	int cur_lane = main_lane;
	//Set the first cost of the main lane to 0
	path[cur_lane].path_cost[path[cur_lane].front_idx] = 0;

	bool ED_finished = false;

	//String pointers. Used in comparing characters
	int test_idx;
	int ref_idx;

	//Pick a lane to go through
	while (!ED_finished) {
		//First Pick a path.
		while (path[cur_lane].path_cost[path[cur_lane].front_idx] != cur_dist) {
			if (cur_lane == max_indel_num * 2 + 1) { //Next is boundary
				if (cur_dist >= max_diff_num) { //Check if we exceeds the max diff tolarence
					//TODO: BREAK AND END.
					ED_finished = true;
					result.correct = false;
					break;
				} else {
					//Go to next tier
					cur_lane = 1;
					cur_dist++;
				}
			} else {
				cur_lane++;
			}
		}

		//Indicate if the lane should be stopped
		int slide_stop = 0;

		//Slide down the lane.
		while (!ED_finished) {
			//Conservative test, speed up common case
			if (path[cur_lane].front_idx <= max_indel_num) {
				//Test if it's the last element
				if ((cur_lane >= main_lane && path[cur_lane].front_idx == 0) //Insertion lane
						|| (cur_lane < main_lane && path[cur_lane].front_idx //Deletion Lane
								== main_lane - cur_lane)) {
					ED_finished = true;
					result.correct = true;
					break;
				}
			}

			//Compute string idx:
			test_idx = path[cur_lane].front_idx + cur_lane - main_lane - 1;
			ref_idx = path[cur_lane].front_idx - 1;

			//test if can slide down
			if (!(path[cur_lane].path_cost[path[cur_lane].front_idx - 1]
					== cur_dist //If can just slide
					|| test_read[test_idx] == ref_read[ref_idx]))
				slide_stop = 1;

			//Check neighbor lanes and update them. Modify -> decrement -> modify
			//Check upper neighbor
			if (path[cur_lane - 1].path_cost[path[cur_lane].front_idx]
					> cur_dist + slide_stop + 1)
				path[cur_lane - 1].path_cost[path[cur_lane].front_idx]
						= cur_dist + 1;
			//Decrement
			path[cur_lane].front_idx--;
			//Check itself
			if (path[cur_lane].path_cost[path[cur_lane].front_idx] > cur_dist
					+ slide_stop)
				path[cur_lane].path_cost[path[cur_lane].front_idx] = cur_dist
						+ slide_stop;
			//Check lower (left) neighbor
			if (path[cur_lane + 1].path_cost[path[cur_lane].front_idx]
					> cur_dist + slide_stop + 1)
				path[cur_lane + 1].path_cost[path[cur_lane].front_idx]
						= cur_dist + 1;

			//stop if can't slide anymore
			if (slide_stop == 1) {
				break;
			}
		}
	}
	
	DEBUG_PRINT2("inside editCalBWD, after path cost generation a  threadId: %i\n", threadIdx.x);
	DEBUG_PRINT2("inside editCalBWD, after path cost generation b  threadId: %i\n", threadIdx.x);

	//Tracing back period
	if (result.correct == false){
DEBUG_PRINT2("1xxx: %i\n", threadIdx.x);
		return result;
	}
	else { //If pass the test, trace back
DEBUG_PRINT2("2xxx: %i\n", threadIdx.x);
		//char temp_result[30]; //Temp string. Used for appending.
		int cur_idx = (cur_lane >= main_lane) ? 0 : main_lane - cur_lane;

		result.diff_num = path[cur_lane].path_cost[cur_idx];

		cur_dist = path[cur_lane].path_cost[cur_idx];

		int error_ptr = 0;
DEBUG_PRINT2("3xxx: %i\n", threadIdx.x);		
for (int i =0; i < READ_LENGTH; i++){
	DEBUG_PRINT2("%c", test_read[i]);
}
	DEBUG_PRINT1("\n");
for (int i =0; i < READ_LENGTH; i++){
	DEBUG_PRINT2("%c", ref_read[i]);
}
	DEBUG_PRINT1("\n");

		while (cur_lane != main_lane || cur_idx != key_num * KEY_LENGTH) {
			DEBUG_PRINT2("4xxx: %i\n", threadIdx.x);
			DEBUG_PRINT2("cur_idx: %i\n", cur_idx);
			DEBUG_PRINT2("cur_idx val : %i\n", key_num * KEY_LENGTH);
			//If we should have an insertion
			if (cur_idx == key_num * KEY_LENGTH || 
				path[cur_lane + 1].path_cost[cur_idx] < path[cur_lane].path_cost[cur_idx + 1]){

				//if (error_ptr < MAX_ERROR_NUM) {	// DHL: block error
					result.error[error_ptr].diff = INSERTION;
					result.error[error_ptr].location = cur_idx + cur_lane - main_lane;
					result.error[error_ptr].diff_char = test_read[result.error[error_ptr].location];
				//}
				if (error_ptr >= MAX_ERROR_NUM) {	// DHL: block error 
					break;
				}

				error_ptr++;
				cur_lane++;
				DEBUG_PRINT4("4xx6: threadIdx %i / error_ptr: %i / cur_lane: %i \n", 
							threadIdx.x, error_ptr, cur_lane);
				continue;
			}

			//If we should have a deletion
			if (path[cur_lane - 1].path_cost[cur_idx + 1]
					< path[cur_lane].path_cost[cur_idx + 1]) {

				result.error[error_ptr].diff = DELETION;
				result.error[error_ptr].location = cur_idx + cur_lane
						- main_lane;
				result.error[error_ptr].diff_char = ref_read[cur_idx];

				error_ptr++;
				cur_lane--;
				cur_idx++;
				continue;
			}

			//Check if we have a mismatch
			if (path[cur_lane].path_cost[cur_idx + 1]
					< path[cur_lane].path_cost[cur_idx]) {
				result.error[error_ptr].diff = MISMATCH;
				result.error[error_ptr].location = cur_idx + cur_lane
						- main_lane;
				result.error[error_ptr].diff_char
						= test_read[result.error[error_ptr].location];
				error_ptr++;
				cur_idx++;
				continue;
			}

			//Move to the next element
			cur_idx++;
		}
	}
	DEBUG_PRINT2("inside editCalBWD, after tracing back generation a  threadId: %i\n", threadIdx.x);
	DEBUG_PRINT2("inside editCalBWD, after tracing back generation b  threadId: %i\n", threadIdx.x);
	return result;
}

