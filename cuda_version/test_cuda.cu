/*
 * test_search.cpp
 *
 *  Created on: Oct 19, 2011
 *	  Author: mac
 */
#include <iostream>
#include "common.h"
#include "hash_transfer.h"

using namespace std;

int test_cuda(void) {
	long long coord_num = loadHash("toy_hash_0");
	GPU_fragment test_fragment;
	test_fragment.fragment = "GGGTGGTAGGTGCAGAGACGGGAGGGGCAGAGCCGCAGGCACAGCCAAGAGGGCTGAAGAAATGGTAGAACGGAGCAGCTGGTGATGTGTGGGCCCACCGGCCCCAGGCTCCTGTCTCCC";

	GPU_fragment* dev_fragment;
	final_result* dev_result;
	final_result* test_result = (final_result*)malloc(sizeof(GPU_fragment));

	hipMalloc((void**)&dev_fragment, sizeof(GPU_fragment));
	hipMalloc((void**)&dev_result,   sizeof(final_result));
	hipMemcpy(dev_fragment, test_fragment, sizeof(GPU_fragment), hipMemcpyHostToDevice );

	searchFragment <<<1, 1>>> (dev_fragment, 1, dev_ref_string, dev_hash_table, dev_coordinate, 5, 5, dev_result);

	hipMemcpy(test_result, dev_result, INDEX_NUM*sizeof(final_result), hipMemcpyDeviceToHost );

	free(str_coordinate);
	hipFree( dev_fragment );
	hipFree( dev_result );
	freeHash();
	freeRef();
	return 0;
}

int main() {
	test_cuda();
	return 0;
}
