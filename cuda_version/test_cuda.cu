/*
 * test_search.cpp
 *
 *  Created on: Oct 19, 2011
 *	  Author: mac
 */
#include <iostream>
#include <cstdlib>
#include <sstream>
#include <stdio.h>

#include "universal_header.h"

using namespace std;

int test_cuda(void) {
	long long coord_num = loadHash("toy_hash_0");
	GPU_fragment test_fragment;
	//test_fragment.fragment = "GGGTGGTAGGTGCAGAGACGGGAGGGGCAGAGCCGCAGGCACAGCCAAGAGGGCTGAAGAAATGGTAGAACGGAGCAGCTGGTGATGTGTGGGCCCACCGGCCCCAGGCTCCTGTCTCCC";
	strcpy(test_fragment.fragment, "GGGTGGTAGGTGCAGAGACGGGAGGGGCAGAGCCGCAGGCACAGCCAAGAGGGCTGAAGAAATGGTAGAACGGAGCAGCTGGTGATGTGTGGGCCCACCGGCCCCAGGCTCCTGTCTCCC");

	GPU_fragment* dev_fragment;
	final_result* dev_result;
	final_result* test_result = (final_result*)malloc(sizeof(GPU_fragment));

	hipMalloc((void**)&dev_fragment, sizeof(GPU_fragment));
	hipMalloc((void**)&dev_result,   sizeof(final_result));
	hipMemcpy(dev_fragment, &test_fragment, sizeof(GPU_fragment), hipMemcpyHostToDevice );

	searchFragment <<<1, 1>>> (dev_fragment, 1, dev_ref_string, dev_hash_table, dev_coordinate, 5, 5, dev_result);

	hipMemcpy(test_result, dev_result, INDEX_NUM*sizeof(final_result), hipMemcpyDeviceToHost );
	cout << "Fragment     : " << (*test_result).fragment << endl;
	cout << "Spilled      : " << (*test_result).spilled << endl;
	cout << "Result Size  : " << (*test_result).size << endl;
	for(int i = 0; i < (*test_result).size; i++) {
		cout << "Result coor  : " << (*test_result).coor_results[i].coordiante << endl;
		cout << "Result diff  : " << (*test_result).coor_results[i].diff_num << endl;
	}

	free(test_result);
	hipFree( dev_fragment );
	hipFree( dev_result );
	freeHash();
	freeRef();
	return 0;
}

int main() {
	test_cuda();
	return 0;
}
