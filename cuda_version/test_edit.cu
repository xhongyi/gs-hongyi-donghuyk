#include "hip/hip_runtime.h"
/*
 * test_search.cpp
 *
 *  Created on: Oct 19, 2011
 *	  Author: mac
 */
#include <iostream>
#include "universal_header.h"

using namespace std;


#include "hash_test.cu"


__global__ void cuda_editDistanceCal (char * dev_test_read, char * dev_ref_read, ED_result * result) {
	ED_path path[20];
	result[threadIdx.x] = editDistanceCal(dev_test_read, dev_ref_read, threadIdx.x% 3, path, 6, 5, 5);
}

int test_cuda(void) {

	char test_read[READ_LENGTH];
	strcpy(
			test_read,
			"GGGTGGTAGGTGCAGAGACGGGAGGGGCAGAGCCGCAGGCACAGCCAAGAGGGCTGAAGAAATGGTAGAACGGAGCAGCTGGTGATGTGTGGGCCCACCGGCCCCAGG");
	//Getting the sort key.

	char ref_read[READ_LENGTH];
	strcpy(
			ref_read,
			"GGGTGGTAGGTGCAGAGACGGGAGGGGCAGAGCCGCAGGCACAGCCAAGAGGGCTGAAGAAATGGTAGAACGGAGCAGCTGGTGATGTGTGGGCCCACCGGCCCCAGG");
	
	ED_result result[10];

	char * dev_test_read;
	char * dev_ref_read;

	ED_result * dev_result;

	hipMalloc( (void**) &dev_test_read, READ_LENGTH * sizeof(char) );
	hipMalloc( (void**) &dev_ref_read, READ_LENGTH * sizeof(char) );

	hipMemcpy(dev_test_read, test_read, READ_LENGTH * sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(dev_ref_read, ref_read, READ_LENGTH * sizeof(char), hipMemcpyHostToDevice);

	hipMalloc( (void**) &dev_result, 10 * sizeof(ED_result) );

	cuda_editDistanceCal <<<1, 10>>> (dev_test_read, dev_ref_read, dev_result);

	hipMemcpy(result, dev_result, 10 * sizeof(ED_result), hipMemcpyDeviceToHost);

	for (int i = 0; i < 10; i++) {
		cout << "i: " << i << endl;
		if (result[i].correct) {
			cout << "correct! " << endl;
			cout << "result[" << i << "].diff_num: " << result[i].diff_num << endl;
			for (int j = 0; j < result[i].diff_num; i++) {
			 	switch (result[i].error[j].diff) {
			 	case MISMATCH:
			 		cout << "Mismatch at " << result[i].error[j].location << endl;
					cout << "Error: " << result[i].error[j].diff_char << endl;
			 		break;
				case INSERTION:
			 		cout << "Insertion at " << result[i].error[j].location << endl;
			 		cout << "Error: " << result[i].error[j].diff_char << endl;
			 		break;
			 	case DELETION:
			 		cout << "Deletion at " << result[i].error[j].location << endl;
			 		cout << "Error: " << result[i].error[j].diff_char << endl;
			 		break;
			 	}
			}
		}
		else
			cout << "Incorrect!!!" << endl;
	}

	hipFree(dev_test_read);
	hipFree(dev_ref_read);
	hipFree(dev_result);
	return 0;
}

int main() {
	test_cuda();
	return 0;
}
