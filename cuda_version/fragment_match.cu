/*
 * fragment_match.cpp
 *
 *  Created on: Oct 8, 2011
 *      Author: mac
 */
#include <cstdlib>
#include <sstream>
#include <iostream>
#include <assert.h>
#include "common.h"
#include "fragment_match.h"
#include "book.h"

int* hash_table;
int* coordinate;
int* dev_coordinate;

void getHashTablePtr(int ** ptr) {
	*ptr = hash_table;
}

void getCoordinatePtr(int ** ptr) {
	*ptr = coordinate;
}

void loadHash(string hash_name) {
	long long coordinate_total_num = hashReconstructor(&hash_table, &coordinate, hash_name.c_str());
	HANDLE_ERROR(hipMalloc((void**)&dev_coordinate, coordinate_total_num*sizeof(int)));
	HANDLE_ERROR(hipMemcpy(dev_coordinate, coordinate, coordinate_total_num*sizeof(int), hipMemcpyHostToDevice ));
}

void freeHash(void) {
	HANDLE_ERROR( hipFree ( dev_coordinate) );
}
