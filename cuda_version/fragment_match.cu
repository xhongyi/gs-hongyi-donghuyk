#include "hip/hip_runtime.h"
/*
 * fragment_match.cpp
 *
 *  Created on: Oct 8, 2011
 *      Author: mac
 */
#include <cstdlib>
#include <sstream>
#include <iostream>
#include <assert.h>
#include "common.h"
#include "edit_distance.h"
#include "fragment_match.h"

bool select_cheapest;

__device__ bool searchKey(int target_coor, int entry_coor, int entry_size,
		int* coordinate, int max_indel_num);

__device__ bool searchKey(int target_coor, int entry_coor, int entry_size,
		int* coordinate, int max_indel_num) {
	DEBUG_PRINT1("1\n");
	if (entry_size == 0)
		return false;
	int lower_bound = entry_coor + 1;
	int upper_bound = entry_coor + entry_size;
	int mid = lower_bound + entry_size / 2;
	while (lower_bound < upper_bound) {
		if (coordinate[mid] == target_coor)
			break;
		else if (coordinate[mid] < target_coor)
			lower_bound = mid + 1;
		else
			upper_bound = mid - 1;
		mid = lower_bound + (upper_bound - lower_bound) / 2;
	}
	DEBUG_PRINT1("2\n");
	DEBUG_PRINT2("mid: %i\n", mid);
	DEBUG_PRINT2("coordinate[mid]: %i\n", coordinate[mid]);
	DEBUG_PRINT2("target_coor: %i\n", target_coor);

	if (coordinate[mid] <= target_coor + max_indel_num && coordinate[mid]
			>= target_coor - max_indel_num) {
		DEBUG_PRINT1("3 true\n");
		return true;
	} else {
		DEBUG_PRINT1("3 false\n");
		return false;
	}
}

bool sortPrefilter(key_struct* sort_result, key_struct* sort_input) {
	for (int i = 0; i < KEY_NUMBER; i++) {
		for (int j = 0; j < KEY_NUMBER; j++) {
			if (sort_input[i].key_entry_size > sort_input[j].key_entry_size) {
				sort_input[i].order = sort_input[i].order + 1;
			}
		}
	}
	int accumulative_base = 0;
	int loop_index = 0;
	for (int i = 0; i < KEY_NUMBER; i++) {
		for (int j = 0; j < KEY_NUMBER; j++) {
			if (sort_input[j].order == i) {
				sort_result[loop_index].key_entry = sort_input[j].key_entry;
				sort_result[loop_index].key_number = sort_input[j].key_number;
				sort_result[loop_index].key_entry_size
						= sort_input[j].key_entry_size;
				sort_result[loop_index].base = accumulative_base;
				accumulative_base += sort_result[loop_index].key_entry_size;
				loop_index = loop_index + 1;
			}
			//if (loop_index == max_diff_num + 1)
			//	return true;
		}
	}
	return true;//false;
}

__global__ void searchFragment(GPU_fragment* fragment, int fragment_size,
		char* ref, int* hash_table, int* coordinate, int max_diff_num,
		int max_indel_num, final_result* result) {
	DEBUG_PRINT1("HELLO!!\n");
	DEBUG_PRINT2("cuda fragment: %c\n", (*fragment).fragment[0]);
	DEBUG_PRINT3("blockIdx.x: %i, threadIdx.x: %i\n", blockIdx.x, threadIdx.x);
	//This will be used in edit_distance Calculation.
	int main_lane = max_indel_num + 1;
	//Each thread will have a path array for edit_distance calculation.
	ED_path path[MAX_ERROR_NUM];

	//Fragment_counter: get cooresponding fragment
	__shared__ int fragment_count;
	__shared__ int size;
	if (threadIdx.x == 0) {
		DEBUG_PRINT1("Hey Man!!! Here is threadIdx.x == 0\n");
		fragment_count = blockIdx.x;
	}

	__syncthreads();

	DEBUG_PRINT3("fragment_count: %i  threadId: %i\n", fragment_count, threadIdx.x);

	while (fragment_count < fragment_size) {
		DEBUG_PRINT2("###Ever started!! threadId: %i\n", threadIdx.x);
		DEBUG_PRINT4("###fragment_count: %i  fragment_size: %i  threadId: %i\n",
				fragment_count, fragment_size, threadIdx.x);
		//get the corresponding key_num and it's coordinate.
		int coor_count = threadIdx.x;
		int cur_key = 0;

		if (threadIdx.x == 0) {
			size = 0;
		}

		__syncthreads();

		//Get the first key
		while (cur_key <= max_diff_num
				&& coor_count
						>= fragment[fragment_count].sorted_keys[cur_key].base
								+ fragment[fragment_count].sorted_keys[cur_key].key_entry_size) {
			DEBUG_PRINT1("&&&cur_key incrementing!!\n");
			cur_key++;
		}

		while (size < MAX_COOR_RESULT_NUM && cur_key <= max_diff_num) {
			DEBUG_PRINT1("Hey doing something!\n");
			//Do adjacency filtering
			int diff_num = 0;
			DEBUG_PRINT2("start diff : diff num : %i\n", diff_num);
			int
					coor_idx =
							fragment[fragment_count].sorted_keys[cur_key].key_entry
									+ 1 + coor_count
									- fragment[fragment_count].sorted_keys[cur_key].base;
			DEBUG_PRINT3(
					"fragment[fragment_count].sorted_keys[cur_key].key_entry: %i  threadId: %i\n",
					fragment[fragment_count].sorted_keys[cur_key].key_entry,
					threadIdx.x);
			DEBUG_PRINT3(
					"fragment[fragment_count].sorted_keys[cur_key].base: %i  threadId: %i\n",
					fragment[fragment_count].sorted_keys[cur_key].base,
					threadIdx.x);
			DEBUG_PRINT3(
					"fragment[fragment_count].sorted_keys[cur_key].key_entry_size: %i  threadId: %i\n",
					fragment[fragment_count].sorted_keys[cur_key].key_entry_size,
					threadIdx.x);
			DEBUG_PRINT6(
					"***fragment_count: %i, cur_key: %i, coor_count: %i, coor_idx: %i  threadId: %i\n",
					fragment_count, cur_key, coor_count, coor_idx, threadIdx.x);

			for (int i = 0; i < KEY_NUMBER; i++) { //for each segment
				if (i - diff_num > KEY_NUMBER - max_diff_num)
					break;
				DEBUG_PRINT2("coor_idx + (fragment[fragment_count].sorted_keys[i].key_number - fragment[fragment_count].sorted_keys[cur_key].key_number) * KEY_LENGTH: %i\n",
							(coordinate[coor_idx] + (fragment[fragment_count].sorted_keys[i].key_number
							- fragment[fragment_count].sorted_keys[cur_key].key_number) * KEY_LENGTH));
				DEBUG_PRINT2("fragment[fragment_count].sorted_keys[i].key_entry: %i\n",
							fragment[fragment_count].sorted_keys[i].key_entry);
				DEBUG_PRINT2("fragment[fragment_count].sorted_keys[i].key_entry_size: %i\n",
							fragment[fragment_count].sorted_keys[i].key_entry_size);
				if (!searchKey(
						coordinate[coor_idx]
								+ (fragment[fragment_count].sorted_keys[i].key_number
										- fragment[fragment_count].sorted_keys[cur_key].key_number)
										* KEY_LENGTH,
						fragment[fragment_count].sorted_keys[i].key_entry,
						fragment[fragment_count].sorted_keys[i].key_entry_size,
						coordinate, max_indel_num)) {
							DEBUG_PRINT2("changing before : diff num : %i\n", diff_num);
							diff_num++;
							DEBUG_PRINT2("changing after : diff num : %i\n", diff_num);
							DEBUG_PRINT1("4\n");
					if (diff_num > max_diff_num)
						break;
				}
			}
			DEBUG_PRINT1("5\n");
			DEBUG_PRINT2("end diff: diff num : %i", diff_num);
			//Edit_distance Calculation
			if (diff_num <= max_diff_num) {
				DEBUG_PRINT2("Hey in ED test now  threadId: %i\n", threadIdx.x);
				DEBUG_PRINT2("cur_key: %i\n", cur_key);
				char ref_str[READ_LENGTH];
				for (int i = 0; i < READ_LENGTH; i++) { //Get reference string
					DEBUG_PRINT2(
							"coordinate: %i\n",
							coordinate[coor_idx]
									- fragment[fragment_count].sorted_keys[cur_key].key_number
											* KEY_LENGTH + i);
					ref_str[i]
							= ref[coordinate[coor_idx]
									- fragment[fragment_count].sorted_keys[cur_key].key_number
											* KEY_LENGTH + i];
					DEBUG_PRINT3("ref_str[%i]: %c\n", i, ref_str[i]);
				}
				DEBUG_PRINT1("7\n");
				DEBUG_PRINT1("7.5\n");

				ED_result
						edit_result =
								editDistanceCal(
										fragment[fragment_count].fragment,
										ref_str,
										fragment[fragment_count].sorted_keys[cur_key].key_number,
										path, main_lane, max_indel_num,
										max_diff_num);

				DEBUG_PRINT1("7.6\n");
				DEBUG_PRINT1("7.7\n");

				__syncthreads();

				DEBUG_PRINT2("1 threadId: %i\n", threadIdx.x);
				DEBUG_PRINT2("2 threadId: %i\n", threadIdx.x);
				if (edit_result.correct) {
					DEBUG_PRINT3("find something!  cur_key: %i  threadId: %i\n", cur_key, threadIdx.x);
					__syncthreads(); // Delete this afterwards!
					int temp_size = size;
					DEBUG_PRINT3("Avoid Duplication temp_size: %i  threadId: %i\n", temp_size, threadIdx.x);
					bool duplicate = false;
					for(int i = 0; i < temp_size; i++){
						if(result[fragment_count].coor_results[i].coordiante == (coordinate[coor_idx]
									- fragment[fragment_count].sorted_keys[cur_key].key_number * KEY_LENGTH)){
							duplicate = true;
						}
					}
					if (duplicate == false) {
						int atomic_size = atomicAdd(&size, 1);
							DEBUG_PRINT4("##### Attomic ADD++ blockId:%i, threadId:%i, size:%i\n",
								blockIdx.x, threadIdx.x, size);
						if (atomic_size < MAX_COOR_RESULT_NUM) {
							DEBUG_PRINT2("***** ATTOMIC ADD happends! threadId: %i\n", threadIdx.x);
							//DEBUG_PRINT6("Attomic ADD blockId:%i, threadId:%i, size:%i, coordinate:%i, diff_num:%i\n",
							DEBUG_PRINT6("***** Attomic ADD blockId:%i, threadId:%i, size:%i, coordinate:%i, diff_num:%i\n",
								blockIdx.x, threadIdx.x, size,
								coordinate[coor_idx]-fragment[fragment_count].sorted_keys[cur_key].key_number*KEY_LENGTH,
								edit_result.diff_num);

							result[fragment_count].coor_results[atomic_size].coordiante
									= coordinate[coor_idx]
											- fragment[fragment_count].sorted_keys[cur_key].key_number
													* KEY_LENGTH;
							result[fragment_count].coor_results[atomic_size].diff_num
									= edit_result.diff_num;
							for (int i = 0; i < edit_result.diff_num; i++)
								result[fragment_count].coor_results[atomic_size].error[i] = edit_result.error[i];
						}
					}
				}
			}
			DEBUG_PRINT1("6\n");
			//move to the next coordinate by incrementing coor_count
			DEBUG_PRINT2("blockDim.x: %i\n", blockDim.x);
			coor_count += blockDim.x;
			//Move to the next key
			while (cur_key <= max_diff_num
					&& coor_count
							>= fragment[fragment_count].sorted_keys[cur_key].base
									+ fragment[fragment_count].sorted_keys[cur_key].key_entry_size) {
				DEBUG_PRINT1("&&&cur_key incrementing!!\n");
				cur_key++;
			}
			DEBUG_PRINT1("still here!!!\n");
		}

		DEBUG_PRINT1("before syncthreads\n");

		__syncthreads();

		DEBUG_PRINT3("print Ids again!! blockIdx.x: %i, threadIdx.x: %i\n",
				blockIdx.x, threadIdx.x);

		DEBUG_PRINT4("Size update blockId:%i, threadId:%i, size:%i\n", blockIdx.x, threadIdx.x, size);
		if (threadIdx.x == 0) {
			DEBUG_PRINT1("I'm Here!!!\n");
			for (int i = 0; i < READ_LENGTH; i++) {
				result[fragment_count].fragment[i]
						= fragment[fragment_count].fragment[i];
				DEBUG_PRINT4("result[%i].fragment[%i]: %c\n", fragment_count, i,
						result[fragment_count].fragment[i]);
			}
			if (size >= MAX_COOR_RESULT_NUM) {
				result[fragment_count].spilled = true;
				result[fragment_count].size = MAX_COOR_RESULT_NUM;
			}
			else {
				result[fragment_count].spilled = false;
				result[fragment_count].size = size;
				DEBUG_PRINT4("##### Size update blockId:%i, threadId:%i, size:%i\n",
								blockIdx.x, threadIdx.x, size);
			}
			fragment_count += gridDim.x;
		}
		__syncthreads();
	}
}

/*
 final_result searchFragment(char* fragment, char* ref, int* hash_table,
 int* coordinate) {
 key_struct sort_input[KEY_NUMBER];
 for (int i = 0; i < KEY_NUMBER; i++) {
 char key[KEY_LENGTH];
 for (int j = 0; j < KEY_NUMBER; j++) {
 key[j] = fragment[j + KEY_LENGTH * i];
 //key = fragment.substr(KEY_LENGTH * i, KEY_LENGTH);
 }
 int key_hash = hashVal(key);
 int key_entry = hash_table[key_hash];
 int key_entry_size = coordinate[key_entry];
 sort_input[i].order = 0;
 sort_input[i].key_number = i;
 sort_input[i].key_entry = key_entry;
 sort_input[i].key_entry_size = key_entry_size;
 }

 key_struct keys_input[KEY_NUMBER];
 sortPrefilter(keys_input, sort_input);
 previous_result.size = 0;
 final_result return_result;
 return_result.total_binary_search = 0;
 return_result.total_edit_perform = 0;
 return_result.total_correct_num = 0;

 for (int k = 0; k < max_diff_num + 1; k++) { //Getting the seed
 for (int i = keys_input[k].key_entry + 1; i <= keys_input[k].key_entry
 + keys_input[k].key_entry_size; i++) { //Going through each entry in the seed coordinate list
 int coor_value = coordinate[i];
 int diff_num = 0;
 if (!searchPrevious(coor_value, keys_input[k].key_number,
 previous_result)) { //Pre-filtering
 return_result.total_binary_search++;
 for (int j = 0; j < KEY_NUMBER; j++) { //for each segment
 if (j - diff_num > KEY_NUMBER - max_diff_num)
 break;
 if (!searchKey(
 coor_value + (keys_input[j].key_number
 - keys_input[k].key_number) * KEY_LENGTH,
 keys_input[j].key_entry,
 keys_input[j].key_entry_size)) {
 diff_num++;
 if (diff_num > max_diff_num)
 break;
 }
 }
 if (diff_num <= max_diff_num) {
 if (previous_result.size <= PREFILTER_SIZE) {
 previous_result.coor[previous_result.size] = coor_value
 - keys_input[k].key_number * KEY_LENGTH; //start_coor;
 previous_result.size++;
 }
 return_result.total_edit_perform++;
 string ref_str(FRAGMENT_LENGTH, 'A');
 ref_str = (*ref).substr(
 coor_value - keys_input[k].key_number * KEY_LENGTH,
 FRAGMENT_LENGTH); //start_coor;

 /////////////////////Just For Testing
 char test_char[READ_LENGTH];
 char ref_char[READ_LENGTH];
 strcpy(test_char, fragment.c_str());
 strcpy(ref_char, ref_str.c_str());
 cout << "ref__read: " << ref_char << endl;
 cout << "test_read: " << test_char << endl;
 cout << "key_num__: " << keys_input[k].key_number << endl;
 /////////////////////Testing END

 ED_result edit_result = editDistanceCal(test_char,
 ref_char, keys_input[k].key_number);
 if (edit_result.correct) {
 return_result.total_correct_num++;
 //cout << "ref_read      : " << ref_str << "  coordinate: "<< (*it_result).coordinate << "  Key_number: "<< (*it_result).key_number;
 //cout << "  result: correct " << endl;
 } else {
 //cout << "ref_read      : " << ref_str << "  coordinate: "<< (*it_result).coordinate << "  Key_number: "<< (*it_result).key_number;
 //cout << "  result: not correct" <<endl;
 }
 }
 }
 }
 }
 return return_result;
 }

 */
